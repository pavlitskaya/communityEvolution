#include "stdafx.h"
#include "include/general_search.h"

__host__ __device__ uint32_t g_binary_search(pair_t* source, pair_t key, uint32_t pair_size, int32_t ind_min, int32_t ind_max, bool& found, bool b_swap){
	if (b_swap && key.second < key.first)thrust::swap(key.first, key.second);

	int32_t ind_mid = 0;
	found = false;

	while (ind_min <= ind_max){
		//printf("ind_min: %lu, ind_max: %lu, ind_mid: %lu \n", ind_min, ind_max, ind_mid);
		ind_mid = ind_min + (ind_max - ind_min)*0.5;
		if (source[ind_mid].first == key.first && source[ind_mid].second == key.second){
			found = true;
			return ind_mid;
		}
		else
			if (source[ind_mid].first > key.first ||
				(source[ind_mid].first == key.first &&
				source[ind_mid].second > key.second)){
			if (ind_mid - 1 < 0)return ind_mid = 0;
			ind_max = ind_mid - 1;
			}
			else
				ind_min = ind_mid + 1;
	}

	//if (!found && ind_mid == pair_size - 1)
		//return pair_size;
	return ind_mid;
}

__host__ __device__ uint32_t g_binary_search(pair_t* source, pair_t key, uint32_t pair_size, bool& found, bool b_swap){
	return g_binary_search(source, key, pair_size, 0, (int32_t)pair_size - 1, found, b_swap);
}

__host__ __device__ bool g_binary_search(pair_t* source, pair_t key, uint32_t pair_size, bool b_swap){
	return g_binary_search(source, key, pair_size, 0, pair_size - 1, b_swap);
}

__host__ __device__ bool g_binary_search(pair_t* source, pair_t key, uint32_t pair_size, uint32_t ind_min, uint32_t ind_max, bool b_swap){
	if (b_swap && key.second < key.first)thrust::swap(key.first, key.second);

	uint32_t ind_mid = 0;
	while (ind_min <= ind_max){
		ind_mid = ind_min + (ind_max - ind_min)*0.5;
		if (source[ind_mid].first == key.first && source[ind_mid].second == key.second){
			return true;
		}
		else
			if (source[ind_mid].first > key.first ||
				(source[ind_mid].first == key.first &&
				source[ind_mid].second > key.second))
				ind_max = ind_mid - 1;
			else
				ind_min = ind_mid + 1;
	}
	return false;
}

__host__ __device__ bool g_binary_search(uint32_t* begin, uint32_t* end, uint32_t key){
	bool found;
	g_binary_search(begin, end, key, 0, (end - 1) - begin, found);
	return found;
}

__host__ __device__ uint32_t g_binary_search(uint32_t* begin, uint32_t* end, uint32_t key, bool& found){
	return g_binary_search(begin, end, key, 0, (end - 1) - begin, found);
}

__host__ __device__ uint32_t g_binary_search(uint32_t* begin, uint32_t* end, uint32_t key, uint32_t ind_min, uint32_t ind_max, bool& found){
	found = false;
	uint32_t ind_mid = 0;
	uint32_t count = 0;
	while (ind_min <= ind_max){
		ind_mid = ind_min + (ind_max - ind_min)*0.5;
		if (*(begin + ind_mid) == key){
			found = true;
			return ind_mid;
		}
		else{
			if (*(begin + ind_mid) > key){
				if (ind_mid == 0)
					return 0;
				ind_max = ind_mid - 1;
			}
			else
				ind_min = ind_mid + 1;
		}
	}
	return ind_mid;
}
