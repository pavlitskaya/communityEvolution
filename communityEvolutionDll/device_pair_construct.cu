#include "stdafx.h"
#include "include/device_pair_construct.h"
#include "include/device_analytic.h"
#include "include/general_arithmetic_structs.h"
#include "include/device_pair_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/display_elements.h"
#include "include/general_pair_structs.h"
#include "include/device_pair.h"

using namespace std;

// takes mirrored
bool generate_pairs(std::vector<uint32_t>& keys, std::vector<uint32_t>& values, std::vector<pair_t>& pairs){
	T_DV<pair_t> d_target;
    T_DV<uint32_t> d_keys(keys.begin(), keys.end());
    T_DV<uint32_t> d_values(values.begin(), values.end());
    generate_pairs(d_keys, d_values, d_target);
	T_HV<pair_t> h_target(d_target.begin(), d_target.end());
	pairs.assign(h_target.begin(), h_target.end());
	return 1;
}

bool generate_unique_pairs(std::vector<uint32_t>& keys, std::vector<uint32_t>& values, std::vector<pair_t>& pairs){
	T_DV<pair_t> d_target;
    T_DV<uint32_t> d_keys(keys.begin(), keys.end());
    T_DV<uint32_t> d_values(values.begin(), values.end());
    generate_pairs(d_keys, d_values, d_target);
	thrust::sort(d_target.begin(), d_target.end());
	d_target.resize(thrust::unique(d_target.begin(), d_target.end()) - d_target.begin());
	T_HV<pair_t> h_target(d_target.begin(), d_target.end());
	pairs.assign(h_target.begin(), h_target.end());
	return 1;
}

bool generate_pairs_deep(std::vector<uint32_t>& keys, std::vector<uint32_t>& values, std::vector<pair_t>& pairs){
	T_DV<pair_t> d_target;
    T_DV<pair_t> d_keys(T_MTI(T_MZIMT(keys.begin(), values.begin()), pair_create()), T_MTI(T_MZIMT(keys.end(), values.end()), pair_create()));
    generate_pairs_deep(d_keys, d_target);
	T_HV<pair_t> h_target(d_target.begin(), d_target.end());
	pairs.assign(h_target.begin(), h_target.end());
	return 1;
}


/* generates degree deep
* parameter: mirrored, sorted
*/
bool generate_pairs_deep(T_DV<pair_t>& d_keys_value, T_DV<pair_t>& d_target){
	T_DV<uint32_t> d_firsts, d_degree;
    T_DV<uint32_t> d_keys(T_MTI(d_keys_value.begin(), first_element()), T_MTI(d_keys_value.end(), first_element()));
    if (!get_degree_mirror(d_keys, d_degree))return 0;
	if (!get_firsts(d_degree, d_firsts))return 0;
	return generate_pairs_deep(d_keys_value.begin(), d_keys_value.end(), d_firsts, d_degree, d_target);
}

/* generates degree deep
* parameter: mirrored, sorted
*/
bool generate_pairs_deep(T_DV<pair_t>::iterator d_keys_value_first, T_DV<pair_t>::iterator d_keys_value_last, T_DV<uint32_t> &d_firsts, T_DV<uint32_t> &d_degree, T_DV<pair_t>& d_target){
	// generate pairs init
	uint32_t max_size = get_max_combination(d_degree.begin(), d_degree.end());
	if (max_size < 1)return 1;

	// generate pairs main
	d_target.assign(max_size, pair_t(0, 0));
	
	uint32_t offset = 0;
	d_firsts.push_back(d_firsts.back() + d_degree.back()); // add last
	for (uint32_t i = 0; i < d_firsts.size() - 1; ++i){
		T_DV<uint32_t> values(
			T_MPI(T_MTI(d_keys_value_first, second_element()), T_MCI<uint32_t>(d_firsts[i])),
			T_MPI(T_MTI(d_keys_value_first, second_element()), T_MCI<uint32_t>(d_firsts[i + 1]))
			);
		uint32_t n_elements = d_degree[i] * (d_degree[i] - 1)*0.5;
		get_pairs(0, n_elements, d_degree[i], values, d_target, offset);
		offset += n_elements;

	}
	d_firsts.erase(d_firsts.end() - 1);

	return 1;
}

/* changes
* parameter: mirrored, sorted
*/
bool generate_pairs(T_DV<pair_t>& d_keys_value, T_DV<pair_t>& d_pairs){
	return generate_pairs(d_keys_value.begin(), d_keys_value.end(), d_pairs);
}

/* changes
* parameter: mirrored, sorted
*/
bool generate_pairs_limit(T_DV<pair_t>& d_keys_value, T_DV<pair_t>& d_pairs, uint32_t& offStart, uint32_t limit, bool& done){
	return generate_pairs_limit(d_keys_value.begin(), d_keys_value.end(), d_pairs, offStart, limit, done);
}

/* changes
* parameter: mirrored, sorted, offset
*/
bool generate_pairs_limit(T_DV<pair_t>::iterator d_keys_value_first, T_DV<pair_t>::iterator d_keys_value_last, T_DV<pair_t>& d_pairs, uint32_t& offStart, uint32_t limit, bool& done){
	d_pairs.clear();
	thrust::sort(d_keys_value_first, d_keys_value_last);
	d_keys_value_last = thrust::unique(d_keys_value_first, d_keys_value_last);
	uint32_t degree = d_keys_value_last - d_keys_value_first;
	uint32_t max_size = degree * (degree - 1) * 0.5;

	// generate pairs init
	if (max_size < 1){
		done = 1;
		return 1;
	}
	// size calculation
	T_DV<uint32_t>sizes(degree);
	thrust::sequence(sizes.begin(), sizes.end(), (int)degree, -1);
	thrust::exclusive_scan(sizes.begin(), sizes.end(), sizes.begin());
	// generate pairs main
	uint32_t off = offStart + 1;
	uint32_t intervals = degree;

	d_pairs.assign(limit, pair_t(0, 0));
	T_DV<pair_t>::iterator new_end = d_pairs.begin();
	//display_vector<uint32_t>(sizes, "sizes");
	while (off < intervals){
		if (sizes[off] - sizes[offStart] > limit){
			break;
		}
		new_end = thrust::copy_if(
			T_MTI(
			T_MZIMT(
			T_MTI(d_keys_value_first, second_element()),
			T_MTI(d_keys_value_first + off, second_element())),
			pair_create_sort()),
			T_MTI(
			T_MZIMT(
			T_MTI(d_keys_value_first + degree - off, second_element()),
			T_MTI(d_keys_value_first + degree, second_element())),
			pair_create_sort()),
			T_MCI<uint32_t>(0),
			new_end,
			equal_to_next(T_DEREF(d_keys_value_first, pair_t), degree, off));
		off += 1;
	}
	offStart = off - 1;
	if (off == intervals)
		done = true;
	d_pairs.erase(thrust::remove(d_pairs.begin(), d_pairs.end(), pair_t(0, 0)), d_pairs.end());
	d_pairs.erase(thrust::remove_if(d_pairs.begin(), d_pairs.end(), is_loop_pair()), d_pairs.end());
	return 1;
}

/* changes
* parameter: mirrored, sorted
*/
bool generate_pairs(T_DV<pair_t>::iterator d_keys_value_first, T_DV<pair_t>::iterator d_keys_value_last, T_DV<pair_t>& d_pairs){
	d_pairs.clear();
	thrust::sort(d_keys_value_first, d_keys_value_last);
	d_keys_value_last = thrust::unique(d_keys_value_first, d_keys_value_last);
	T_DV<uint32_t> d_degree;
    T_DV<uint32_t> d_keys(T_MTI(d_keys_value_first, first_element()), T_MTI(d_keys_value_last, first_element()));
    if (!get_degree_mirror(d_keys, d_degree))return 0;
	// generate pairs init
	uint32_t max_size = get_max_combination(d_degree.begin(), d_degree.end());
	if (max_size < 1)return 1;
	// generate pairs main
	uint32_t size = d_keys_value_last - d_keys_value_first;
	uint32_t off = 1;
	uint32_t intervals = *thrust::max_element(d_degree.begin(), d_degree.end());
	T_CLEAR(d_degree, uint32_t);
	d_pairs.assign(max_size, pair_t(0, 0));
	T_DV<pair_t>::iterator new_end = d_pairs.begin();
	//printf("first: %lu \n", (pair_t(*(d_keys_value_first + size - 1))).first, (pair_t(*(d_keys_value_first + size - 1))).second);
	for (uint32_t i = 0; i < size; ++i){
		//printf("%lu:%lu, ", (pair_t(*(d_keys_value_first + i))).first, (pair_t(*(d_keys_value_first + i))).second);
	}
	//display_direct(T_DEREF(d_keys_value_first, pair_t), T_DEREF(d_keys_value_last, pair_t), "c");
	while (off < intervals){
		T_TRYCATCH(
			new_end = thrust::copy_if(thrust::device,
			T_MTI(
			T_MZIMT(
			T_MTI(d_keys_value_first, second_element()),
			T_MTI(d_keys_value_first + off, second_element())),
			pair_create_sort()),
			T_MTI(
			T_MZIMT(
			T_MTI(d_keys_value_first + size - off, second_element()),
			T_MTI(d_keys_value_first + size, second_element())),
			pair_create_sort()),
			T_MCI<uint32_t>(0),
			new_end,
			equal_to_next(T_DEREF(d_keys_value_first, pair_t), size, off));
			);
		off += 1;
	}
	d_pairs.erase(thrust::remove_if(d_pairs.begin(), d_pairs.end(), is_loop_pair()), d_pairs.end());

	return 1;
}

bool generate_pairs(T_DV<uint32_t>& d_keys, T_DV<uint32_t>& d_values, T_DV<pair_t>& d_pairs){
	return generate_pairs(d_keys.begin(), d_keys.end(), d_values.begin(), d_values.end(), d_pairs);
}

/* takes mirrored
 */
bool generate_pairs(T_DV<uint32_t>::iterator d_keys_first, T_DV<uint32_t>::iterator d_keys_last, T_DV<uint32_t>::iterator d_values_first, T_DV<uint32_t>::iterator d_values_last, T_DV<pair_t>& d_pairs){
    T_DV<pair_t> d_keys(T_MTI(T_MZIMT(d_keys_first, d_values_first),pair_create()), T_MTI(T_MZIMT(d_keys_last, d_values_last),pair_create()));
    return generate_pairs(d_keys, d_pairs);
}

bool pair_create_constant(uint32_t constant, uint32_t start, uint32_t end, T_DV<uint32_t>& values, T_DV<pair_t>::iterator target){
	thrust::transform(
		T_MPI(values.begin(), T_MCI<uint32_t>(start)),
		T_MPI(values.begin(), T_MCI<uint32_t>(end)),
		target,
		pair_create_const(values[constant])
		);
	return 1;
}

__host__ __device__ pair_t get_pair(const uint32_t& x, uint32_t& n){
	uint32_t count = 0;
	uint32_t k = n - 1;
	uint32_t old = 0;
	for (uint32_t j = n - 1; j > 0; j += --k){
		if (x < j){
			break;
		}
		++count;
		old = j;
	}
	return pair_t((uint32_t)count, (uint32_t)x - old + 1 + count);
}

/*
Used to get pairs from from to to in tree n.
n - tree-size
values - relevant value
target - contains relevant pairs afterwards
offset - stores result in target beginning with offset

example: (0, 3, 4,_, 1) stores [_, (0,1),(0,2),(0,3),(1,2)]

*/
bool get_pairs(const uint32_t& from, const uint32_t& to, uint32_t n, T_DV<uint32_t>& values, T_DV<pair_t>& target, uint32_t offset){

	pair_t f = get_pair(from, n);
	pair_t t = get_pair(to, n);
	pair_t tpre = get_pair(to - 1, n);

	// current part
	pair_create_constant(f.first, f.second, n, values, target.begin() + offset);
	uint32_t c = n - f.second;

	// till t.first
	for (uint32_t i = f.first + 1; i < tpre.first; ++i){
		pair_create_constant(i, i + 1, n, values, target.begin() + c + offset);
		c += n - (i + 1);
	}

	if (to > 1)
		pair_create_constant(tpre.first, tpre.first + 1, tpre.second + 1, values, target.begin() + c + offset);

	return 1;
}
