#include "stdafx.h"
#include "include/data_info.h"

pair_t get_device_memory(){
	size_t f, t;
	hipSetDevice(0);
	hipFree(0); 
	hipMemGetInfo(&f, &t);
	return pair_t(f, t);
}

void display_device_memory(){
	pair_t dev_mem = get_device_memory();

	std::locale loc("");
	std::cout.imbue(loc);
	std::cout << "free memory: " << (dev_mem.first / 1024) / 1024 << "MB "
		<< "total memory: " << (dev_mem.second / 1024) / 1024 << "MB" << std::endl;
}

