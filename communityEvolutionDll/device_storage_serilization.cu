#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/device_storage_serilization.h"
#include "include/host_storage_serilization.h"
#include "include/general_pair_structs.h"
#include "include/display_elements.h"

using namespace std;

static uint32_t allocationCount = 0;

bool serialize(T_DV<pair_t>& d_pair, string type, uint32_t offset, uint32_t allocate, uint32_t id){
	pair_t* h_pair = (pair_t*)malloc(sizeof(pair_t)*allocate);
	thrust::copy(d_pair.begin() + offset, d_pair.begin() + offset + allocate, h_pair);
	comevo::Serilization::store(h_pair, allocate, type, id);
	free(h_pair);
	return 1;
}

bool from_device_store(T_DV<uint32_t>& d_val, std::string type, uint32_t offset, uint32_t allocate, uint32_t id){
	T_HV<uint32_t> h_val(d_val.begin() + offset, d_val.begin() + offset + allocate);
	//vector<uint32_t> val(h_val.begin(), h_val.end());
	if (!comevo::Serilization::store(h_val, type, id)) return 0;
	return 1;
}

bool to_device_load(T_DV<uint32_t>& d_val, std::string type, uint32_t id, bool clean){
	T_HV<uint32_t> h_val(0);
	if (!comevo::Serilization::load(h_val, type, id, clean))return 0;
	d_val.assign(h_val.begin(), h_val.end());
	return 1;
}

bool from_device_store(T_DV<pair_t>& d_vec, std::string type, uint32_t offset, uint32_t allocate, uint32_t id){
	T_HV<uint32_t> h_vecFirst(
		T_MTI(d_vec.begin() + offset, first_element()), 
		T_MTI(d_vec.begin() + offset + allocate, first_element()));
	T_HV<uint32_t> h_vecSecond(
		T_MTI(d_vec.begin() + offset, second_element()), 
		T_MTI(d_vec.begin() + offset + allocate, second_element()));
	/*vector<uint32_t> vecFirst(h_vecFirst.begin(), h_vecFirst.end());
	vector<uint32_t> vecSecond(h_vecSecond.begin(), h_vecSecond.end());
	
	if (!comevo::Serilization::store(vecFirst, type + "f", id))return 0;
	if (!comevo::Serilization::store(vecSecond, type + "s", id))return 0;*/
	if (!comevo::Serilization::store(h_vecFirst, type + "f", id))return 0;
	if (!comevo::Serilization::store(h_vecSecond, type + "s", id))return 0;
	return 1;
}

bool to_device_load(T_DV<pair_t>& d_vec, std::string type, uint32_t id, bool clean){
	T_HV<uint32_t> h_vecFirst(0);
	T_HV<uint32_t> h_vecSecond(0);
	if (!comevo::Serilization::load(h_vecFirst, type + "f", id, clean))return 0;
	if (!comevo::Serilization::load(h_vecSecond, type + "s", id, clean))return 0;
	if (h_vecFirst.size() != h_vecSecond.size()){
		cout << "allocation issue count: " << ++allocationCount << endl;
		h_vecFirst.resize(min(h_vecFirst.size(), h_vecSecond.size()));
		h_vecSecond.resize(min(h_vecFirst.size(), h_vecSecond.size()));
	}
	
	d_vec.assign(
		T_MTI(T_MZIMT(h_vecFirst.begin(), h_vecSecond.begin()), pair_create()),
		T_MTI(T_MZIMT(h_vecFirst.end(), h_vecSecond.end()), pair_create()));
	return 1;
}

namespace comevohost{

	bool from_host_store(T_HV<uint32_t>& d_val, std::string type, uint32_t offset, uint32_t allocate, uint32_t id){
		vector<uint32_t> val(d_val.size());
		thrust::copy(d_val.begin() + offset, d_val.begin() + offset + allocate, val.begin());
		if (!comevo::Serilization::store(val, type, id)) return 0;
		return 1;
	}

	bool to_host_load(T_HV<uint32_t>& d_val, std::string type, uint32_t id, bool clean){
		vector<uint32_t> h_val(0);
		if (!comevo::Serilization::load(h_val, type, id, clean))return 0;
		d_val.assign(h_val.begin(), h_val.end());
		return 1;
	}

	bool from_host_store(T_HV<pair_t>& d_vec, std::string type, uint32_t offset, uint32_t allocate, uint32_t id){
		vector<uint32_t> vecFirst(d_vec.size());
		vector<uint32_t> vecSecond(d_vec.size());
		thrust::copy(T_MTI(d_vec.begin() + offset, first_element()), T_MTI(d_vec.begin() + offset + allocate, first_element()), vecFirst.begin());
		thrust::copy(T_MTI(d_vec.begin() + offset, second_element()), T_MTI(d_vec.begin() + offset + allocate, second_element()), vecSecond.begin());
		if (!comevo::Serilization::store(vecFirst, type + "f", id))return 0;
		if (!comevo::Serilization::store(vecSecond, type + "s", id))return 0;
		return 1;
	}

	bool to_host_load(T_HV<pair_t>& d_vec, std::string type, uint32_t id, bool clean){
		vector<uint32_t> vecFirst(0);
		vector<uint32_t> vecSecond(0);
		if (!comevo::Serilization::load(vecFirst, type + "f", id, clean))return 0;
		if (!comevo::Serilization::load(vecSecond, type + "s", id, clean))return 0;
		d_vec.assign(
			T_MTI(T_MZIMT(vecFirst.begin(), vecSecond.begin()), pair_create()),
			T_MTI(T_MZIMT(vecFirst.end(), vecSecond.end()), pair_create()));
		return 1;
	}

}