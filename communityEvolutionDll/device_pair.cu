#include "stdafx.h"
#include "include/device_pair.h"
#include "include/general_pair_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/general_arithmetic_structs.h"
#include "include/device_pair_structs.h"
#include "include/display_elements.h"


bool combine_values(T_DV<pair_t>& d_targetPairs, T_DV<uint32_t>& d_targetVal, T_DV<pair_t>& d_sourcePairs, T_DV<uint32_t>& d_sourceVal){

	T_DV<pair_t> d_mergedPairs(d_targetPairs.size() + d_sourcePairs.size());
	T_DV<uint32_t> d_mergedVals(d_targetPairs.size() + d_sourcePairs.size());
	thrust::merge_by_key(d_targetPairs.begin(), d_targetPairs.end(),
		d_sourcePairs.begin(), d_sourcePairs.end(),
		d_targetVal.begin(), d_sourceVal.begin(),
		d_mergedPairs.begin(), d_mergedVals.begin());

	thrust::pair<T_DV<pair_t>::iterator, T_DV<uint32_t>::iterator> newEnds = thrust::reduce_by_key(d_mergedPairs.begin(), d_mergedPairs.end(),
		d_mergedVals.begin(), d_mergedPairs.begin(), d_mergedVals.begin());
	d_mergedPairs.erase(newEnds.first, d_mergedPairs.end());
	d_mergedVals.erase(newEnds.second, d_mergedVals.end());

	T_DV<uint32_t> d_indices(d_targetPairs.size());
	thrust::lower_bound(d_mergedPairs.begin(), d_mergedPairs.end(), d_targetPairs.begin(), d_targetPairs.end(), d_indices.begin());
	thrust::copy(T_MPI(d_mergedVals.begin(), d_indices.begin()), T_MPI(d_mergedVals.begin(), d_indices.end()), d_targetVal.begin());
	return 1;
}

bool combine_pairs(T_DV<pair_t>& d_targetPairs, T_DV<uint32_t>& d_targetVal, T_DV<pair_t>& d_sourcePairs, T_DV<uint32_t>& d_sourceVal){

	T_DV<pair_t> d_mergedPairs(d_targetPairs.size() + d_sourcePairs.size());
	T_DV<uint32_t> d_mergedVals(d_targetPairs.size() + d_sourcePairs.size());
	thrust::merge_by_key(d_targetPairs.begin(), d_targetPairs.end(),
		d_sourcePairs.begin(), d_sourcePairs.end(), 
		d_targetVal.begin(), d_sourceVal.begin(), 
		d_mergedPairs.begin(), d_mergedVals.begin());

	thrust::pair<T_DV<pair_t>::iterator, T_DV<uint32_t>::iterator> newEnds = thrust::reduce_by_key(d_mergedPairs.begin(), d_mergedPairs.end(),
		d_mergedVals.begin(), d_mergedPairs.begin(), d_mergedVals.begin());
	d_mergedPairs.erase(newEnds.first, d_mergedPairs.end());
	d_mergedVals.erase(newEnds.second, d_mergedVals.end());

	d_targetPairs.swap(d_mergedPairs);
	d_targetVal.swap(d_mergedVals);
	return 1;
}

void pairsToNodes(std::vector<pair_t>& source, std::vector<uint32_t>& target){
	thrust::device_vector<uint32_t> d_target;
    thrust::device_vector<pair_t> d_source = thrust::device_vector<pair_t>(source.begin(), source.end());
    pairsToNodes(d_source, d_target);
	thrust::host_vector<uint32_t> h_target(d_target.begin(), d_target.end());
	target.assign(h_target.begin(), h_target.end());
}

void pairsToNodes(thrust::device_vector<pair_t>& source, thrust::device_vector<uint32_t>& target) {
	target.resize(source.size() * 2);
	
	thrust::transform(
		source.begin(), source.end(), 
		T_MPI(target.begin(), T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0), 
		thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>())),
		first_element());
	thrust::transform(
		source.begin(), source.end(),
		T_MPI(target.begin(), T_MTI(T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0),
		thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>()), set_increase<uint32_t>())),
		second_element());
}

void pairsToUniqueNodes(std::vector<pair_t>& source, std::vector<uint32_t>& target){
	thrust::device_vector<uint32_t> d_target;
    thrust::device_vector<pair_t> d_source = thrust::device_vector<pair_t>(source.begin(), source.end());
    pairsToUniqueNodes(d_source, d_target);
	thrust::host_vector<uint32_t> h_target(d_target.begin(), d_target.end());
	target.assign(h_target.begin(), h_target.end());
}

void pairsToUniqueNodes(thrust::device_vector<pair_t>& source, thrust::device_vector<uint32_t>& target){
	target.resize(source.size() * 2);
	thrust::transform(
		source.begin(), source.end(),
		T_MPI(target.begin(), T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0),
		thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>())),
		first_element());
	thrust::transform(
		source.begin(), source.end(),
		T_MPI(target.begin(), T_MTI(T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0),
		thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>()), set_increase<uint32_t>())),
		second_element());
	thrust::sort(target.begin(), target.end());
	target.resize(thrust::unique(target.begin(), target.end()) - target.begin());
}

void mirror_pairs(T_DV<pair_t>& d_source, T_DV<pair_t>& d_target){
	d_target.assign(d_source.begin(), d_source.end());
	d_target.resize(d_target.size() * 2);
	thrust::transform(d_source.begin(), d_source.end(), d_target.begin() + d_source.size(), pair_create_inverse());
	thrust::sort(d_target.begin(), d_target.end()); 
}

void mirror_pairs_inplace(T_DV<pair_t>& d_source_target){
	uint32_t old_size = d_source_target.size();
	d_source_target.resize(old_size * 2);
	thrust::transform(d_source_target.begin(), d_source_target.begin() + old_size, d_source_target.begin() + old_size, pair_create_inverse());
	thrust::sort(d_source_target.begin(), d_source_target.end());
}
