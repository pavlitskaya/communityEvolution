#include "stdafx.h"
#include "include/host_convert.h"
#include "include/display_elements.h"
#include "include/general_pair.h"
#include "include/device_pair.h"
#include "include/general_structs.h"
#include "include/general_comparsion_structs.h"

using namespace std;

namespace comevohost{

	bool translate_snapshot_to_vector(snapshot_t& snap, T_HV<uint32_t>& h_snapVec){
		for (snapshot_t::iterator it = snap.begin(); it != snap.end(); ++it){
			h_snapVec.insert(h_snapVec.end(), (*it).begin(), (*it).end());
		}
		return 1;
	}

	//bool translate_snapshot_to_matrix(T_HV<uint32_t>& h_preSnapVec, T_HV<uint32_t>& h_sizes, T_HV<uint32_t>& h_relevant, T_HV<bool>& h_preMatrix){
	//	uint32_t offset = 0;
	//	T_HV<bool> h_found(0);
	//	T_HV<uint32_t> h_indices(0);
	//	h_found.resize(h_preSnapVec.size());
	//	uint32_t n = h_relevant.size();
	//	// target: snap in matrix

	//	h_preMatrix.assign(h_sizes.size() * n, 0);

	//	for (uint32_t i = 0; i < h_sizes.size(); ++i){
	//		for (uint32_t j = 0; j < h_sizes[i]; ++j){
	//			h_found[j] = thrust::binary_search(h_relevant.begin(), h_relevant.end(),
	//				h_preSnapVec[offset + j]);
	//		}
	//		h_indices.resize(thrust::count(h_found.begin(), h_found.end(), 1));
	//		T_HV<uint32_t> h_counter(n);
	//		thrust::sequence(h_counter.begin(), h_counter.end());
	//		thrust::copy_if(h_counter.begin(), h_counter.end(), h_found.begin(), h_indices.begin(), is_one<bool>());

	//		thrust::transform(T_MPI(h_preMatrix.begin() + i*n, h_indices.begin()),
	//			T_MPI(h_preMatrix.begin() + i*n, h_indices.end()),
	//			T_MPI(h_preMatrix.begin() + i*n, h_indices.begin()),
	//			set_one<bool>());
	//		offset += h_sizes[i];

	//	}

	//	return 1;
	//}

	bool translate_snapshot_to_matrix(T_HV<uint32_t>& h_preSnapVec, T_HV<uint32_t>& h_sizes, T_HV<uint32_t>& h_relevant, T_HV<bool>& h_preMatrix){
		uint32_t offset = 0;
		T_HV<bool> h_found(0);
		T_HV<uint32_t> h_indices(0);
		h_found.resize(h_relevant.size());
		uint32_t n = h_relevant.size();
		// target: snap in matrix

		// h_sizes.size() show how many communities are in the given matrix, so that every community will be compared with h_relevant
		h_preMatrix.assign(h_sizes.size() * n, 0);

		for (uint32_t i = 0; i < h_sizes.size(); ++i){
			thrust::binary_search(
				h_preSnapVec.begin() + offset,
				h_preSnapVec.begin() + offset + h_sizes[i],
				h_relevant.begin(),
				h_relevant.end(),
				h_found.begin());
			h_indices.resize(thrust::count(h_found.begin(), h_found.end(), 1));
			thrust::copy_if(T_MCI<uint32_t>(0), T_MCI<uint32_t>(n), h_found.begin(), h_indices.begin(), thrust::identity<bool>());

			/*for (uint32_t j = 0; j < h_sizes[i]; ++j){
			h_found[j] = thrust::binary_search(h_relevant.begin(), h_relevant.end(),
			h_preSnapVec[offset + j]);
			}
			h_indices.resize(thrust::count(h_found.begin(), h_found.end(), 1));
			T_HV<uint32_t> h_counter(n);
			thrust::sequence(h_counter.begin(), h_counter.end());
			thrust::copy_if(h_counter.begin(), h_counter.end(), h_found.begin(), h_indices.begin(), is_one<bool>());*/

			thrust::transform(T_MPI(h_preMatrix.begin() + i*n, h_indices.begin()),
				T_MPI(h_preMatrix.begin() + i*n, h_indices.end()),
				T_MPI(h_preMatrix.begin() + i*n, h_indices.begin()),
				set_one<bool>());
			offset += h_sizes[i];

		}

		return 1;
	}

	struct fill_com_vec{
		uint32_t *comSizes, *comVec, *comFirsts;

		__host__ __device__
			fill_com_vec(uint32_t* comSizes, uint32_t* comVec, uint32_t* comFirsts) :
			comSizes(comSizes), comVec(comVec), comFirsts(comFirsts) {}

		__host__ __device__
			void operator()(uint32_t i){
			uint32_t size = comSizes[i];
			for (uint32_t j = 0; j < comSizes[i]; ++j){
				comVec[comFirsts[i] + j] = i;
			}
		}
	};



	bool translate_scom_to_vector(T_HV<uint32_t>& h_comSizes, T_HV<uint32_t>& h_comVec){
		h_comVec.assign(thrust::reduce(h_comSizes.begin(), h_comSizes.end()), 0);
		T_HV<uint32_t>h_comFirsts(h_comSizes.size());
		thrust::exclusive_scan(h_comSizes.begin(), h_comSizes.end(), h_comFirsts.begin());
		T_HV<uint32_t> h_counter(h_comSizes.size());
		thrust::sequence(h_counter.begin(), h_counter.end());
		thrust::for_each(h_counter.begin(), h_counter.end(),
			fill_com_vec(RAWD(h_comSizes), RAWD(h_comVec), RAWD(h_comFirsts)));
		return 1;
	}
}