#include "stdafx.h"
#include "include/host_pair.h"
#include "include/general_pair_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/general_arithmetic_structs.h"
#include "include/device_pair_structs.h"
#include "include/display_elements.h"

namespace comevohost{

	bool combine_values(T_HV<pair_t>& d_targetPairs, T_HV<uint32_t>& d_targetVal, T_HV<pair_t>& d_sourcePairs, T_HV<uint32_t>& d_sourceVal){

		T_HV<pair_t> d_mergedPairs(d_targetPairs.size() + d_sourcePairs.size());
		T_HV<uint32_t> d_mergedVals(d_targetPairs.size() + d_sourcePairs.size());
		thrust::merge_by_key(d_targetPairs.begin(), d_targetPairs.end(),
			d_sourcePairs.begin(), d_sourcePairs.end(),
			d_targetVal.begin(), d_sourceVal.begin(),
			d_mergedPairs.begin(), d_mergedVals.begin());

		thrust::pair<T_HV<pair_t>::iterator, T_HV<uint32_t>::iterator> newEnds = thrust::reduce_by_key(d_mergedPairs.begin(), d_mergedPairs.end(),
			d_mergedVals.begin(), d_mergedPairs.begin(), d_mergedVals.begin());
		d_mergedPairs.erase(newEnds.first, d_mergedPairs.end());
		d_mergedVals.erase(newEnds.second, d_mergedVals.end());

		//thrust::set_intersection_by_key()
		T_HV<pair_t> d_indices(T_MTI(T_MZIMT(d_mergedVals.begin(), d_mergedVals.begin()), pair_create()), T_MTI(T_MZIMT(d_mergedVals.end(), d_mergedVals.end()), pair_create()));

		//thrust::pair<T_HV<pair_t>::iterator, T_HV<uint32_t>::iterator> newEnds = thrust::set_intersection_by_key(d_mergedPairs.begin(), d_mergedPairs.end(), d_targetPairs.begin(), d_targetPairs.end(), d_mergedVals.begin(), d_targetPairs.begin(), d_targetVal.begin());
		thrust::pair<T_HV<pair_t>::iterator, T_HV<pair_t>::iterator> newEnds2 = thrust::set_intersection_by_key(d_mergedPairs.begin(), d_mergedPairs.end(), d_targetPairs.begin(), d_targetPairs.end(), d_indices.begin(), d_targetPairs.begin(), d_indices.begin());
		d_indices.erase(newEnds2.second, d_indices.end());
		thrust::copy(T_MTI(d_indices.begin(), first_element()), T_MTI(d_indices.end(), first_element()), d_targetVal.begin());
		return 1;
	}

	bool combine_pairs(T_HV<pair_t>& d_targetPairs, T_HV<uint32_t>& d_targetVal, T_HV<pair_t>& d_sourcePairs, T_HV<uint32_t>& d_sourceVal){

		T_HV<pair_t> d_mergedPairs(d_targetPairs.size() + d_sourcePairs.size());
		T_HV<uint32_t> d_mergedVals(d_targetPairs.size() + d_sourcePairs.size());
		thrust::merge_by_key(d_targetPairs.begin(), d_targetPairs.end(),
			d_sourcePairs.begin(), d_sourcePairs.end(),
			d_targetVal.begin(), d_sourceVal.begin(),
			d_mergedPairs.begin(), d_mergedVals.begin());

		thrust::pair<T_HV<pair_t>::iterator, T_HV<uint32_t>::iterator> newEnds = thrust::reduce_by_key(d_mergedPairs.begin(), d_mergedPairs.end(),
			d_mergedVals.begin(), d_mergedPairs.begin(), d_mergedVals.begin());
		d_mergedPairs.erase(newEnds.first, d_mergedPairs.end());
		d_mergedVals.erase(newEnds.second, d_mergedVals.end());

		d_targetPairs.swap(d_mergedPairs);
		d_targetVal.swap(d_mergedVals);
		return 1;
	}

	void pairsToNodes(std::vector<pair_t>& source, std::vector<uint32_t>& target){
		T_HV<uint32_t> d_target;
        T_HV<pair_t> d_source(source.begin(), source.end());
        pairsToNodes(d_source, d_target);
		thrust::host_vector<uint32_t> h_target(d_target.begin(), d_target.end());
		target.assign(h_target.begin(), h_target.end());
	}

	void pairsToNodes(T_HV<pair_t>& source, T_HV<uint32_t>& target){
		target.resize(source.size() * 2);

		thrust::transform(
			source.begin(), source.end(),
			T_MPI(target.begin(), T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0),
			thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>())),
			first_element());
		thrust::transform(
			source.begin(), source.end(),
			T_MPI(target.begin(), T_MTI(T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0),
			thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>()), set_increase<uint32_t>())),
			second_element());
	}

	void pairsToUniqueNodes(std::vector<pair_t>& source, std::vector<uint32_t>& target){
		T_HV<uint32_t> d_target;
        T_HV<pair_t> d_source(source.begin(), source.end());
        pairsToUniqueNodes(d_source, d_target);
		thrust::host_vector<uint32_t> h_target(d_target.begin(), d_target.end());
		target.assign(h_target.begin(), h_target.end());
	}

	void pairsToUniqueNodes(T_HV<pair_t>& source, T_HV<uint32_t>& target){
		target.resize(source.size() * 2);
		thrust::transform(
			source.begin(), source.end(),
			T_MPI(target.begin(), T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0),
			thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>())),
			first_element());
		thrust::transform(
			source.begin(), source.end(),
			T_MPI(target.begin(), T_MTI(T_MTI(T_MZIMT(thrust::make_counting_iterator<uint32_t>(0),
			thrust::make_constant_iterator<uint32_t>(2)), zip_mul<uint32_t>()), set_increase<uint32_t>())),
			second_element());
		thrust::sort(target.begin(), target.end());
		target.resize(thrust::unique(target.begin(), target.end()) - target.begin());
	}

	void mirror_pairs(T_HV<pair_t>& d_source, T_HV<pair_t>& d_target){
		d_target.assign(d_source.begin(), d_source.end());
		d_target.resize(d_target.size() * 2);
		thrust::transform(d_source.begin(), d_source.end(), d_target.begin() + d_source.size(), pair_create_inverse());
		thrust::sort(d_target.begin(), d_target.end());
	}

	void mirror_pairs_inplace(T_HV<pair_t>& d_source_target){
		uint32_t old_size = d_source_target.size();
		d_source_target.resize(old_size * 2);
		thrust::transform(d_source_target.begin(), d_source_target.begin() + old_size, d_source_target.begin() + old_size, pair_create_inverse());
		thrust::sort(d_source_target.begin(), d_source_target.end());
	}
}
