#include "stdafx.h"
#include "include/host_algorithm_clique.h"
#include "include/host_analytic.h"
#include "include/general_comparsion_structs.h"

using namespace std;

namespace comevohost{

	bool generate_clique(T_HV<pair_t>& pairs, uint32_t minimumClique, T_HV<uint32_t>& cliques, T_HV<uint32_t>& cliquesFirst){

		T_HV<uint32_t> degree, firsts, nodes;
		get_degree(pairs, degree);
		get_firsts(degree, firsts);
		get_nodes(pairs, firsts, nodes);
		uint32_t n = nodes.size();
		T_HV<uint32_t> nodeId(T_MCI<uint32_t>(0), T_MCI<uint32_t>(nodes.size()));
		
		display_vector<uint32_t>(degree, "degree");
		display_vector<uint32_t>(nodeId, "nodeId");

		thrust::sort_by_key(degree.begin(), degree.end(), nodeId.begin(), thrust::greater<uint32_t>());

		display_vector<uint32_t>(degree, "degree");
		display_vector<uint32_t>(nodeId, "nodeId");

		//uint32_t id = thrust::find(T_MZIMT(T_MCI<uint32_t>(n), degree.begin()), T_MZIMT(T_MCI<uint32_t>(0), degree.end()), is_smaller_equal_zip<uint32_t>()) - T_MZIMT(T_MCI<uint32_t>(n), degree.begin());
		
		/*
get_degree(pairs, degree)
get_nodes(pairs, nodes)
sort_by_key(degree, descend, nodes)
ids()
id = find(T_MCI, T_MCI, degree.begin(), is_greater())
rel_nodes(T_MPI(nodes, ids(0)), T_MPI(nodes, ids(id)));
*/

		return 1;
	}

	bool generate_communities(T_HV<uint32_t>& nodes, T_HV<uint32_t>& cliques, T_HV<uint32_t>& cliquesFirst, T_HV<uint32_t>& communities, T_HV<uint32_t>& communitiesFirst){
		return 1;
	}

	bool algorithm_clique(comevo::Source &source, comevo::Source &target, uint32_t minimumClique){

		//if (!generate_clique())return 0;

		//if (!generate_communities())return 0;

		return 1;
	}
}