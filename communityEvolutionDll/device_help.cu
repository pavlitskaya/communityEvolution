#include "stdafx.h"
#include "include/device_help.h"
#include "include/general_pair_structs.h"

namespace comevo{
	template <typename T> bool back_inserter(T_HV<T>& h_vec, T value){
		h_vec.push_back(value);
		return 1;
	}
	template bool back_inserter<uint32_t>(T_HV<uint32_t>&, uint32_t);
	template bool back_inserter<pair_t>(T_HV<pair_t>&, pair_t);

	bool fill_pair_vector(std::vector<uint32_t>& vecFirst, std::vector<uint32_t>& vecSecond, T_HV<pair_t>& h_vec){
		h_vec.resize(vecFirst.size());
		thrust::copy(T_MTI(T_MZIMT(vecFirst.begin(), vecSecond.begin()), pair_create()), T_MTI(T_MZIMT(vecFirst.end(), vecSecond.end()), pair_create())
			, h_vec.begin());
		return 1;
	}

	template <typename T> bool fill_vector(std::vector<T>& vec, T_HV<T>& h_vec){
		h_vec.resize(vec.size());
		thrust::copy(vec.begin(), vec.end(), h_vec.begin());
		return 1;
	}
	template bool fill_vector<uint32_t>(std::vector<uint32_t>& vec, T_HV<uint32_t>& h_vec);
	template bool fill_vector<pair_t>(std::vector<pair_t>& vec, T_HV<pair_t>& h_vec);

	template <typename T> bool allocate_vector(T_HV<T>& h_vec, uint32_t size){
		h_vec.resize(size);
		return 1;
	}
	template bool allocate_vector<uint32_t>(T_HV<uint32_t>& h_vec, uint32_t size);
	template bool allocate_vector<pair_t>(T_HV<pair_t>& h_vec, uint32_t size);
	

}