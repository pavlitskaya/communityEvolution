#include "display_elements.cu"
#include "include/display_elements.h"

//template void display_direct<uint32_t>(uint32_t*, uint32_t*, char*);
template void display_direct<uint32_t, uint32_t>(pair_t*, pair_t*, char*);

template void display_single<bool>(bool const &);
template void display_single<uint32_t>(uint32_t const &);
template void display_single<uint16_t>(uint16_t const &);

template void display_vector<uint32_t, uint32_t>(vector<thrust::pair<uint32_t, uint32_t> > const &);
template void display_vector<uint16_t, uint16_t>(vector<thrust::pair<uint16_t, uint16_t> > const &);
template void display_vector<time_t>(vector<time_t> const &vec);
template void display_vector<uint32_t>(vector<uint32_t> const &vec);
template void display_vector<uint16_t>(vector<uint16_t> const &vec);
template void display_vector<int>(vector<int> const &vec);
template void display_vector<uint32_t, uint32_t>(T_DV<thrust::pair<uint32_t, uint32_t> > const &);
template void display_vector<uint16_t, uint16_t>(T_DV<thrust::pair<uint16_t, uint16_t> > const &);
template void display_vector<int, int>(T_DV<thrust::pair<int, int> > const &);
template void display_vector<uint32_t>(T_DV<uint32_t> const &vec);
template void display_vector<uint16_t>(T_DV<uint16_t> const &vec);
template void display_vector<int>(T_DV<int> const &vec);
template void display_vector<bool, bool>(vector<thrust::pair<bool, bool> > const &);
template void display_vector<bool>(vector<bool> const &vec);
template void display_vector<bool, bool>(T_DV<thrust::pair<bool, bool> > const &);
template void display_vector<bool>(T_DV<bool> const &vec);
template void display_vector<uint32_t, uint32_t>(T_HV<thrust::pair<uint32_t, uint32_t> > const &);
template void display_vector<uint16_t, uint16_t>(T_HV<thrust::pair<uint16_t, uint16_t> > const &);
template void display_vector<int, int>(T_HV<thrust::pair<int, int> > const &);
template void display_vector<uint32_t>(T_HV<uint32_t> const &vec);
template void display_vector<uint16_t>(T_HV<uint16_t> const &vec);
template void display_vector<int>(T_HV<int> const &vec);
template void display_vector<float>(T_HV<float> const &vec);
template void display_vector<bool, bool>(T_HV<thrust::pair<bool, bool> > const &);
template void display_vector<bool>(T_HV<bool> const &vec);

void display_snapshot(snapshot_t& vec, char* name){
	cout << "snap " << name << endl;
	for (int i = 0; i < vec.size(); ++i){
		cout << "com " << i << endl;
		for (int j = 0; j < vec[i].size(); ++j){
			cout << vec[i][j] << " ";
		}
		cout << endl;
	}
	cout << endl;
	cout << "\n";
}

void display_snapshots(vector<snapshot_t>& vec, char* name){
	cout << name << ": " << endl;
	for (int i = 0; i < vec.size(); ++i){
		cout << "snap " << i << endl;
		for (int j = 0; j < vec[i].size(); ++j){
			cout << "com " << j << endl;
			for (int k = 0; k < vec[i][j].size(); ++k){
				cout << vec[i][j][k] << " ";
			}
			cout << endl;
		}
		cout << endl;
	}
	cout << "\n";
	cout << "\n";
}
