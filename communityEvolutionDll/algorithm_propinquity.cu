#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/algorithm_propinquity.h"
#include "include/device_analytic.h"
#include "include/general_pair_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/device_pair.h"
#include "include/data_info.h"
#include "include/display_elements.h"
#include "include/device_pair_construct.h"
#include "include/general_arithmetic_structs.h"
#include "include/host_storage_serilization.h"
#include "include/general_search.h"
#include "include/device_storage_serilization.h"

using namespace std;

#define CPULIMIT 2500000

struct d_bfs : public thrust::unary_function < uint32_t, uint32_t >
{
	pair_t* pairs;
	uint32_t* firsts, *nodes, size, nodes_size;
	bool *queue, *visited, *pre_visited;

	__host__ __device__
		d_bfs(pair_t* pairs, uint32_t size, uint32_t* firsts, uint32_t* nodes, uint32_t nodes_size, bool* visited, bool* pre_visited, bool* queue) :
		pairs(pairs), size(size), firsts(firsts), nodes(nodes), nodes_size(nodes_size), visited(visited), pre_visited(pre_visited), queue(queue){}

	__host__ __device__
		void operator()(const uint32_t &x)
	{
		bool found;
		uint32_t count = 0;
		if (!queue[x])return;
		visited[x] = 1;
		// add neighbours
		uint32_t it_first = firsts[x];
		while (it_first < firsts[x + 1]){
			uint32_t v = pairs[it_first].second;
			uint32_t nmb = g_binary_search(nodes, nodes + nodes_size, v, found);
			// check node, add it, if visited
			if (!visited[nmb] && !pre_visited[nmb]){
				queue[nmb] = 1;
			}
			++it_first;
		}
		queue[x] = 0;
		return;

	}
};

static uint32_t storageCounter = 0;
struct get_indices : public thrust::unary_function < uint32_t, bool >
{
	uint32_t* firsts, *degree, n;
	__host__ __device__
		get_indices(uint32_t* firsts, uint32_t* degree, uint32_t n) : firsts(firsts), degree(degree), n(n){}

	__host__ __device__
		bool operator()(const uint32_t &x){
		for (uint32_t i = 0; i < n; ++i){
			if (firsts[i] + degree[i] > x){
				if (x >= firsts[i])return 1;
				return 0;
			}
		}
		return 0;
	}
};


void setStorageCounter(uint32_t value){
	storageCounter = value;
}

uint32_t getStorageCounter(){
	return storageCounter;
}

bool bfs(T_DV<pair_t>& d_pairs, T_DV<uint8_t>& propinquities, uint32_t minimum, snapshot_t& communities){
	communities.clear();
	if (propinquities.size() == d_pairs.size())
		d_pairs.erase(thrust::remove_if(d_pairs.begin(), d_pairs.end(), propinquities.begin(), is_smaller<uint8_t>(minimum)), d_pairs.end());
	if (d_pairs.empty())return 0;
	T_DV<uint32_t> d_degree;
	get_degree(d_pairs, d_degree);
	T_DV<pair_t> d_pairs_mirror(d_pairs.begin(), d_pairs.end());
	mirror_pairs_inplace(d_pairs_mirror);
    T_DV<uint32_t> number_of_different_elements(T_MTI(d_pairs_mirror.begin(), first_element()), T_MTI(d_pairs_mirror.end(), first_element()));
    uint32_t n = get_number_of_diff_elements(number_of_different_elements);
	T_CLEAR(d_pairs, pair_t);

	T_DV<uint32_t> d_firsts, d_nodes;
	get_firsts(d_degree, d_firsts);
	get_nodes(d_pairs_mirror, d_firsts, d_nodes);
	d_firsts.push_back(d_firsts.back() + d_degree.back()); // add last

	// init
	T_DV < bool > d_queue(n, 0);
	T_DV<bool> d_visited(n, 0);
	T_HV < bool > h_visited(n);
	T_HV<bool> total_visited(n, 0);
	T_DV<bool> d_pre_visited(n);
	thrust::transform_if(d_degree.begin(), d_degree.end(), d_pre_visited.begin(), set_value<uint32_t>(1), is_smaller<uint32_t>(1));
	thrust::copy(d_pre_visited.begin(), d_pre_visited.end(), total_visited.begin());

	while (total_visited.end() != thrust::find(total_visited.begin(), total_visited.end(), 0)){
		uint32_t s = thrust::find(total_visited.begin(), total_visited.end(), 0) - total_visited.begin();
		thrust::fill(d_visited.begin(), d_visited.end(), 0);

		// search from s
		d_queue[s] = 1;

		// fill queue (check and add neighbours)
		uint32_t q_val = 1;
		do{
			T_TRYCATCH(
				thrust::for_each(T_MCI<U32>(0), T_MCI<U32>(n),
				d_bfs(
				RAWD(d_pairs_mirror), d_pairs_mirror.size(),
				RAWD(d_firsts), RAWD(d_nodes), d_nodes.size(),
				RAWD(d_visited), RAWD(d_pre_visited), RAWD(d_queue)
				));
			);
			q_val = thrust::count_if(d_queue.begin(), d_queue.end(), thrust::identity<bool>());
		} while (q_val > 0);

		// set total
		thrust::copy(d_visited.begin(), d_visited.end(), h_visited.begin());
		thrust::transform(total_visited.begin(), total_visited.end(), h_visited.begin(), total_visited.begin(), thrust::logical_or<bool>());
		thrust::copy(total_visited.begin(), total_visited.end(), d_pre_visited.begin());

		// create and add com
		T_DV < uint32_t > d_com(d_visited.size());
			d_com.erase(thrust::copy_if(d_nodes.begin(), d_nodes.end(), d_visited.begin(), d_com.begin(), thrust::identity<bool>()), d_com.end());
		if (d_com.size() > 2){
			T_HV < uint32_t > h_com(d_com.begin(), d_com.end());
			communities.push_back(vector < uint32_t >(h_com.begin(), h_com.end()));
		}
	}
	return 1;
}

/* The idea of this function is to compress stored files to their limit and removing empty files
 */
bool compress_files(){
	uint32_t cpuLimit = CPULIMIT;

	T_DV<pair_t> d_pair(0), d_fillingPair(0); 
	T_DV<uint32_t> d_val(0), d_fillingVal(0);
	uint32_t offset, stCount = 0;
	uint32_t count = 0;
	for (uint32_t i = 0; i < storageCounter; ++i){
		cout << i << endl;
		uint32_t count = 0;
		// load data
		d_pair.clear();
		d_val.clear();
		if (!to_device_load(d_pair, "pvp", i, true))return 0;
		if (!to_device_load(d_val, "pvi", i, true))return 0;
		if (d_pair.empty())continue;

		cout << i << " " << ++count << endl;
		cout << "size: " << d_pair.size() << endl;
		if (d_fillingPair.empty()){
			d_fillingPair = d_pair;
			d_fillingVal = d_val;
			continue;
		}
		// combine data
		combine_pairs(d_fillingPair, d_fillingVal, d_pair, d_val);
		// check size
		while (d_fillingPair.size() > cpuLimit){
			from_device_store(d_fillingPair, "pvp", 0, cpuLimit, stCount);
			from_device_store(d_fillingVal, "pvi", 0, cpuLimit, stCount);
			++stCount;
			cout << i << " " << ++count << endl;
			// reduce
			d_fillingPair.erase(d_fillingPair.begin(), d_fillingPair.begin() + cpuLimit);
			d_fillingVal.erase(d_fillingVal.begin(), d_fillingVal.begin() + cpuLimit);
		}
	}
	// store the rest
	if (!from_device_store(d_fillingPair, "pvp", 0, d_fillingPair.size(), stCount))return 0;
	if (!from_device_store(d_fillingVal, "pvi", 0, d_fillingVal.size(), stCount)) return 0;
	++stCount;
	storageCounter = stCount;
	return 1;
}

bool cummulate_pairs(T_DV<pair_t>& d_pairs, uint32_t offset, T_DV<uint32_t>& d_cn){
	d_cn.resize(d_pairs.size(), 0);
	if (d_pairs.empty())return 1;
	T_DV<bool> d_found(d_pairs.size());

	for (int i = offset; i < storageCounter; ++i){

		// load and init
		T_DV<pair_t>d_pPair;
		T_DV<uint32_t>d_pVal;
		if (!to_device_load(d_pPair, "pvp", i, true))return 0;
		if (!to_device_load(d_pVal, "pvi", i, true))return 0;
		if (d_pPair.size() == 0 || d_pPair.size() != d_pVal.size()){
			if (!from_device_store(d_pPair, "pvp", 0, 0, i))return 0;
			if (!from_device_store(d_pVal, "pvi", 0, 0, i))return 0;
			continue;
		}
		combine_values(d_pairs, d_cn, d_pPair, d_pVal);
		
		T_DV<bool> d_found(d_pPair.size(), 0);
		//T_DV<bool> d_found(_max(d_pPair.size(), d_pairs.size()));
		//thrust::binary_search(d_pPair.begin(), d_pPair.end(), d_pairs.begin(), d_pairs.end(), d_found.begin());
		thrust::binary_search(d_pairs.begin(), d_pairs.end(), d_pPair.begin(), d_pPair.end(), d_found.begin());
		d_pPair.erase(thrust::remove_if(d_pPair.begin(), d_pPair.end(), d_found.begin(), thrust::identity<bool>()), d_pPair.end());
		d_pVal.erase(thrust::remove_if(d_pVal.begin(), d_pVal.end(), d_found.begin(), thrust::identity<bool>()), d_pVal.end());

		// store
		if (!from_device_store(d_pPair, "pvp", 0, d_pPair.size(), i))return 0;
		if (!from_device_store(d_pVal, "pvi", 0, d_pVal.size(), i))return 0;
	}
	return 1;
}

bool set_new_pairs(T_DV<pair_t>& d_pairs, uint32_t beta, T_DV<uint32_t>& d_propinquity){

	for (int i = 0; i < storageCounter; ++i){
		T_DV<pair_t>d_pPair;
		T_DV<uint32_t>d_pVal;
		if (!to_device_load(d_pPair, "pvp", i, true))return 0;
		if (!to_device_load(d_pVal, "pvi", i, true))return 0;
		if (d_pPair.size() == 0)continue;
		cummulate_pairs(d_pPair, i + 1, d_pVal);

		// 1. count relevant nodes
		uint32_t n_new_nodes = thrust::count_if(d_pVal.begin(), d_pVal.end(), is_greater<uint32_t>(beta - 1));
		uint32_t oldSize = d_propinquity.size();
		d_propinquity.resize(d_propinquity.size() + n_new_nodes);

		uint32_t old_size = d_pairs.size();
		d_pairs.resize(d_pairs.size() + n_new_nodes);

		// add relevant nodes
		thrust::copy_if(
			d_pPair.begin(),
			d_pPair.end(),
			d_pVal.begin(),
			d_pairs.begin() + old_size,
			is_greater<uint32_t>(beta - 1));
		thrust::copy_if(d_pVal.begin(), d_pVal.end(), d_propinquity.begin() + oldSize, is_greater<uint32_t>(beta - 1));

		thrust::sort_by_key(d_pairs.begin(), d_pairs.end(), d_propinquity.begin());
	}
	d_pairs.resize(thrust::unique(d_pairs.begin(), d_pairs.end()) - d_pairs.begin());
	return 1;
}

bool handle_increment(T_DV<pair_t>& all_pairs){
	T_DV<uint32_t>::iterator new_end_i;
	T_DV<pair_t>::iterator new_end_p;
	T_HV<pair_t>::iterator h_new_end_p;
	// sort and create map
	T_TRYCATCH(
		thrust::stable_sort(all_pairs.begin(), all_pairs.end()););
	T_DV<pair_t> d_unique_pairs;
	T_DV<uint32_t> d_unique_values;
	get_count(all_pairs, d_unique_pairs, d_unique_values);

	uint32_t offset = 0;
	do{
        uint32_t allocate = min((unsigned long long)CPULIMIT, (unsigned long long)(d_unique_pairs.size() - offset));
		from_device_store(d_unique_pairs, "pvp", offset, allocate, storageCounter);
		from_device_store(d_unique_values, "pvi", offset, allocate, storageCounter);
		
		offset += allocate;
		++storageCounter;
	} while (offset != d_unique_pairs.size());
	
	return 1;
}

// needs mirror
T_DV<pair_t> get_specific_pairs(T_DV<pair_t> &d_pairs, T_DV<uint32_t> &d_firsts, T_DV<uint32_t> &d_degree, T_DV<uint32_t> &d_id){

	thrust::sort(d_id.begin(), d_id.end());
	T_DV<uint32_t> d_degree_red, d_firsts_red;
	d_firsts_red.assign(
		T_MPI(d_firsts.begin(), d_id.begin()),
		T_MPI(d_firsts.begin(), d_id.end()));
	d_degree_red.assign(
		T_MPI(d_degree.begin(), d_id.begin()),
		T_MPI(d_degree.begin(), d_id.end()));
	cout << "id: " << d_id.size() << endl;

	//T_DV<pair_t> d_pairs(h_pairs.begin(), h_pairs.end());

	T_DV<uint32_t> d_indices(thrust::reduce(d_degree_red.begin(), d_degree_red.end()), 0);
	T_TRYCATCH(
		for (uint32_t i = 0; i < d_pairs.size(); i += 50000){
		thrust::copy_if(T_MCI<uint32_t>(i), T_MCI<uint32_t>(min(i + 50000, (unsigned int)d_pairs.size())), d_indices.begin(), get_indices(RAWD(d_firsts_red), RAWD(d_degree_red), d_firsts_red.size()));
		})

	T_CLEAR(d_firsts_red, uint32_t);
	T_CLEAR(d_degree_red, uint32_t);

	//T_DV<uint32_t> h_indices(d_indices.begin(), d_indices.end());
	T_DV<pair_t> d_pairsResult(T_MPI(d_pairs.begin(), d_indices.begin()), T_MPI(d_pairs.begin(), d_indices.end()));
	T_CLEAR(d_indices, uint32_t);
	return d_pairsResult;
}

// needs mirrored
bool couple_increment(T_DV<pair_t> &d_pairs, T_DV<uint32_t> &d_degree, T_DV<uint32_t> &d_firsts, uint32_t limit){

	if (d_pairs.size() == 0) return 1;
	// 3. split in small and big 
	// 3.1 calculate degree_limit
	T_DV<uint32_t> d_smallId(0);
	T_DV<uint32_t> d_idStorage(T_MCI<uint32_t>(0), T_MCI<uint32_t>(d_degree.size()));

	T_DV<uint32_t> d_degree_sorted(d_degree.begin(), d_degree.end());
	thrust::stable_sort_by_key(d_degree_sorted.begin(), d_degree_sorted.end(), d_idStorage.begin());
	T_DV<uint32_t> scancombinations = get_max_combinations_scanned(d_degree_sorted);
	T_CLEAR(d_degree_sorted, uint32_t);

	// space calculaion:
	uint32_t possiblePairs = limit; //  availableMemory;
	uint32_t possiblePairsBig = limit * 1000; //  availableMemory

	// calculate small ones until 90% done
	uint32_t limitId, doneId = 0, handled = 0, run = 0;

	// else not possible
	uint32_t limitPairs = possiblePairs;
	bool small = true;
	cout << "in the end: " << scancombinations.back() << " ids: " << d_idStorage.size() << endl;
	if (scancombinations[0] < possiblePairsBig){
		do{
			limitId = thrust::upper_bound(scancombinations.begin(), scancombinations.end(), handled + limitPairs) - scancombinations.begin();

			if (limitId == doneId){
				return 1;
				if (doneId < scancombinations.size()) 
					cout << scancombinations[limitId] - handled << endl;
				cout << scancombinations.back() - handled << endl;
				scancombinations[limitId];
				
				limitPairs = possiblePairsBig;
				small = false;
				cout << "switch to big" << endl;
				continue;
			}
			++run;
			uint32_t cur_size = scancombinations[limitId - 1] - handled;
			cout << "handling this time run(" << run << "): " << cur_size << " doing Id: " << limitId << "todo: " << scancombinations.back() - handled << endl;
			if (cur_size != 0){
				d_smallId.assign(d_idStorage.begin() + doneId, d_idStorage.begin() + limitId);

				// get pairs and work on it

				T_DV<pair_t> d_pairs2 = get_specific_pairs(d_pairs, d_firsts, d_degree, d_smallId);
				T_CLEAR(d_smallId, uint32_t);
				T_DV<pair_t>d_target(0);
				if (small){
					cout << "dpairs: " << d_pairs2.size() << endl;
					if (!generate_pairs(d_pairs2, d_target))return 0;
					if (!handle_increment(d_target))return 0;
				}
				else{
					bool done = false;
					uint32_t off = 0;
					while (!done){
						if (!generate_pairs_limit(d_pairs2, d_target, off, limit, done))return 0;
						if (!handle_increment(d_target))return 0;
					}
				}
				T_CLEAR(d_pairs2, pair_t);
			}
			doneId = limitId;
			handled = scancombinations[limitId - 1];
		} while (limitId / d_idStorage.size() < 1);
	}
	else{
		cerr << "problem with size!" << endl;
	}
	return 1;
}

bool calculate_propinquity(T_HV<pair_t>& h_pairs){
	storageCounter = 0;
	T_HV<uint32_t> h_values(h_pairs.size(), 1);
	comevohost::from_host_store(h_pairs, "pvp", 0, h_pairs.size(), storageCounter);
	comevohost::from_host_store(h_values, "pvi", 0, h_values.size(), storageCounter);
	
	h_values.clear();
	++storageCounter;

	T_DV<uint32_t> d_degree, d_firsts;
	T_DV<pair_t>d_pairs_mirror(h_pairs.begin(), h_pairs.end());
	mirror_pairs_inplace(d_pairs_mirror);
	//T_HV<pair_t> h_pairs_mirror(d_pairs_mirror.begin(), d_pairs_mirror.end());

    T_DV<uint32_t> degree_mirror(T_MTI(d_pairs_mirror.begin(), first_element()), T_MTI(d_pairs_mirror.end(), first_element()));
    get_degree_mirror(degree_mirror, d_degree);
	get_firsts(d_degree, d_firsts);
	//T_CLEAR(d_pairs_mirror, pair_t);

	// 2. get limit (device_info)
	pair_t p = get_device_memory();
	uint32_t availableMemory = p.first * 0.7;
	uint32_t n = d_degree.size();
	uint32_t m = d_pairs_mirror.size();

	uint32_t A = (availableMemory/8 - (4 * n + 4 * m)) / 2;
	uint32_t B = (availableMemory/8 - (4 * n + 2 * m)) / 5;
	uint32_t C = min(A, B);
	printf("avl: %d A: %d B: %d, C: %d \n", availableMemory, A, B, C);
	couple_increment(d_pairs_mirror, d_degree, d_firsts, C);
	
	T_DV<pair_t>d_intersection;
    T_DV<pair_t> pairs(h_pairs.begin(), h_pairs.end());
    T_DV<pair_t> pairs_mirror(d_pairs_mirror.begin(), d_pairs_mirror.end());
    get_intersection(pairs, pairs_mirror, d_intersection, d_degree);
	if (d_intersection.size() > 0){
		d_pairs_mirror.assign(d_intersection.begin(), d_intersection.end());
		T_CLEAR(d_intersection, pair_t);
		get_firsts(d_degree, d_firsts);

		couple_increment(d_pairs_mirror, d_degree, d_firsts, C);
	}
	T_CLEAR(d_intersection, pair_t);


	return 1;
}

bool update_graph(T_HV<pair_t>& h_pairs, Threshold &threshold, T_HV<uint32_t>& h_propinquity){
	// A: consider limits
	T_DV<pair_t> d_pairs;
	T_DV<uint32_t> d_degree, d_firsts;
	d_pairs.assign(h_pairs.begin(), h_pairs.end());

	// remove if lower than alpha, add if higher than beta
	// 1. find and remove existing ones, 
	T_DV<uint32_t> d_cn;
	cummulate_pairs(d_pairs, 0, d_cn);

	// 2. if val low alpha remove
	d_pairs.erase(thrust::remove_if(d_pairs.begin(), d_pairs.end(), d_cn.begin(), is_smaller<uint32_t>(threshold.alpha)), d_pairs.end());
	d_cn.erase(thrust::remove_if(d_cn.begin(), d_cn.end(), is_smaller<uint32_t>(threshold.alpha)), d_cn.end());
	// sumup all other pairs
	set_new_pairs(d_pairs, threshold.beta, d_cn);
	h_propinquity.assign(d_cn.begin(), d_cn.end());
	h_pairs.assign(d_pairs.begin(), d_pairs.end());

	return 1;
}

bool algorithm_propinquity(comevo::Source &source, comevo::Source &target, uint32_t from, uint32_t to, Threshold &threshold, uint32_t bfsMinimum, uint32_t maxIterations, uint32_t maxSnap){
	U32 propinquityLimit = maxIterations;
	U32 nSnaps = source.get_n().size(); // number of Snaps
	if (maxSnap != 0)nSnaps = maxSnap;
	
	vector<snapshot_t> snaps;
	for (U32 snapId = 0; snapId < nSnaps; ++snapId){
		cout << endl;
		cout << "snap: " << snapId << endl;
		storageCounter = 0;
		snapshot_t communities(0);
		// get edges
		// A: consider limits
		uint32_t nPairs = source.get_m(snapId);
		if (nPairs == 0){
			snaps.push_back(communities);
			continue;
		}

		// 1. get degree
		T_HV<pair_t> h_pairs;
		T_HV<uint32_t> h_propinquity(0);
		if (to != 0)h_pairs = source.get_edges(snapId, from, to);
		if (to == 0)h_pairs = source.get_edges(snapId);

		uint32_t oldSize;
		// calc prop
		for (U32 run = 0; run < propinquityLimit; ++run){
			cout << "iteration: " << run << endl;
			if (h_pairs.empty())break;
			//display_vector<uint32_t, uint32_t>(h_pairs, "h_pairs");

			if (!calculate_propinquity(h_pairs))return 0;
			//cout << "stc: " << storageCounter << endl;
			// compress
			//if(!compress_files())return 0;
			// update
			oldSize = h_pairs.size();
			if (!update_graph(h_pairs, threshold, h_propinquity))return 0;
			storageCounter = 0;
			cout << "from " << oldSize << " to " << h_pairs.size() << endl;

			if (run > 2 && oldSize == h_pairs.size())break;
			//display_vector<uint32_t>(h_propinquity, "h_propinquity");
		}
		communities.clear();
        if (!h_pairs.empty()) {
            T_DV<pair_t> pairs(h_pairs.begin(), h_pairs.end());
            T_DV<uint8_t> propinqueties(0);
            bfs(pairs, propinqueties, bfsMinimum, communities);
        }
		snaps.push_back(communities);
	}
	vector<pairs_t> vecPairs;
	if(!target.set_source(vecPairs, snaps, SNAPS))return 0;


	return 1;
}
