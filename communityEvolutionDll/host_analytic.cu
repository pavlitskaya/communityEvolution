#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/host_analytic.h"
#include "include/data_source.h"
#include "include/general_comparsion_structs.h"
#include "include/general_arithmetic_structs.h"
#include "include/display_elements.h"
#include "include/general_pair_structs.h"
#include "include/host_pair.h"
#include "include/device_analytic_structs.h"
#include "include/general_search.h"
#include "include/host_convert.h"


using namespace std;

namespace comevohost{
	uint32_t get_number_of_diff_elements(T_HV<uint32_t>& h_source){
		return get_number_of_diff_elements(h_source.begin(), h_source.end());
	}

	uint32_t get_number_of_diff_elements(T_HV<uint32_t>::iterator first, T_HV<uint32_t>::iterator last){
		if (last - first < 2)return (last - first);
		return thrust::count_if(
			T_MTI(T_MZIMT(first, first + 1), is_equal_to()),
			T_MTI(T_MZIMT(last - 1, last), is_equal_to()),
			thrust::logical_not<bool>()) + 1;
	}

	uint32_t get_number_of_diff_elements(T_HV<pair_t>& h_source){
		return get_number_of_diff_elements(h_source.begin(), h_source.end());
	}

	uint32_t get_number_of_diff_elements(T_HV<pair_t>::iterator h_source_first, T_HV<pair_t>::iterator h_source_last){
		if (h_source_last - h_source_first < 2)return (h_source_last - h_source_first);
		return thrust::count_if(
			T_MTI(T_MZIMT(h_source_first, h_source_first + 1), is_equal_to_pair()),
			T_MTI(T_MZIMT(h_source_last - 1, h_source_last), is_equal_to_pair()),
			thrust::logical_not<bool>()) + 1;
	}

	bool get_count(T_HV<pair_t>::iterator h_source_first, T_HV<pair_t>::iterator h_source_last, T_HV<pair_t>& h_unique_pairs, T_HV<uint32_t>& h_unique_count){
		uint32_t n = get_number_of_diff_elements(h_source_first, h_source_last);
		h_unique_pairs.assign(n, pair_t(0, 0));
		h_unique_pairs.erase(thrust::unique_copy(h_source_first, h_source_last, h_unique_pairs.begin()), h_unique_pairs.end());
		h_unique_count.assign(n, 0);
		thrust::reduce_by_key(h_source_first, h_source_last, thrust::make_constant_iterator<uint32_t>(1), thrust::make_discard_iterator(), h_unique_count.begin());
		return 1;
	}

	bool get_count(T_HV<pair_t>& h_source, T_HV<pair_t>& h_unique_pairs, T_HV<uint32_t>& h_unique_count){
		return get_count(h_source.begin(), h_source.end(), h_unique_pairs, h_unique_count);
	}

	// needs mirrored
	bool get_degree_mirror(T_HV<uint32_t>& h_source, T_HV<uint32_t>& h_degree){
		return get_degree_mirror(h_source.begin(), h_source.end(), h_degree);
	}

	bool get_degree_mirror(T_HV<pair_t>& h_source, T_HV<uint32_t>& h_degree){
        T_HV<uint32_t> degree_mirror(T_MTI(h_source.begin(), first_element()), T_MTI(h_source.end(), first_element()));
        return get_degree_mirror(degree_mirror, h_degree);
	}

	bool get_degree_mirror(T_HV<uint32_t>::iterator h_source_first, T_HV<uint32_t>::iterator h_source_last, T_HV<uint32_t>& h_degree){
		uint32_t n = get_number_of_diff_elements(h_source_first, h_source_last);
		h_degree.assign(n, 0);
		thrust::reduce_by_key(h_source_first, h_source_last, thrust::make_constant_iterator<uint32_t>(1), thrust::make_discard_iterator(), h_degree.begin());
		return 1;
	}

	// needs one way source
	bool get_degree(T_HV<pair_t>& h_source, T_HV<uint32_t>& h_degree){
		T_HV<uint32_t> h_nodes;
		pairsToNodes(h_source, h_nodes);
		thrust::sort(h_nodes.begin(), h_nodes.end());
		get_degree_mirror(h_nodes, h_degree);
		return 1;
	}

	bool get_nodes(T_HV<pair_t>& h_pairs, T_HV<uint32_t>& h_firsts, T_HV<uint32_t>& h_nodes){
		h_nodes.resize(h_firsts.size());
		thrust::copy(T_MTI(T_MPI(h_pairs.begin(), h_firsts.begin()), first_element()), T_MTI(T_MPI(h_pairs.begin(), h_firsts.end()), first_element()), h_nodes.begin());
		return 1;
	}

	bool get_firsts(T_HV<uint32_t>& h_degree, T_HV<uint32_t>& h_firsts){
		h_firsts.resize(h_degree.size());
		thrust::exclusive_scan(h_degree.begin(), h_degree.end(), h_firsts.begin());
		return 1;
	}

	bool get_lasts(T_HV<uint32_t>& h_degree, T_HV<uint32_t>& h_lasts){
		h_lasts.resize(h_degree.size());
		thrust::inclusive_scan(h_degree.begin(), h_degree.end(), h_lasts.begin());
		return 1;
	}

	T_HV<uint32_t> get_max_combinations(T_HV<uint32_t>& h_degree){
		return T_HV<uint32_t>(
			T_MTI(T_MTI(T_MZIMT(h_degree.begin(), T_MTI(h_degree.begin(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)),
			T_MTI(T_MTI(T_MZIMT(h_degree.end(), T_MTI(h_degree.end(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)));
	}

	uint32_t get_max_combination(T_HV<uint32_t>& h_degree){
		return get_max_combination(h_degree.begin(), h_degree.end());
	}

	uint32_t get_max_combination(T_HV<uint32_t>::iterator h_degree_first, T_HV<uint32_t>::iterator h_degree_last){
		return thrust::reduce(
			T_MTI(T_MZIMT(h_degree_first, T_MTI(h_degree_first, set_decrease<uint32_t>())), zip_mul<uint32_t>()),
			T_MTI(T_MZIMT(h_degree_last, T_MTI(h_degree_last, set_decrease<uint32_t>())), zip_mul<uint32_t>()))*0.5;
	}

	T_HV<uint32_t> get_max_combinations_scanned(T_HV<uint32_t>& h_degree){
		T_HV<uint32_t> result(h_degree.size());
		thrust::inclusive_scan(T_MTI(T_MTI(T_MZIMT(h_degree.begin(), T_MTI(h_degree.begin(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)),
			T_MTI(T_MTI(T_MZIMT(h_degree.end(), T_MTI(h_degree.end(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)), result.begin());
		return result;
	}

	bool get_intersection(T_HV<pair_t>& h_pairs, T_HV<pair_t>& h_pairs_mirror, T_HV<pair_t>& h_target_mirror, T_HV<uint32_t>& h_target_degree){
		T_HV<uint32_t> h_target_firsts;

		T_HV<uint32_t> h_degree, h_firsts, h_nodes;
		get_degree_mirror(h_pairs_mirror, h_degree);
		get_firsts(h_degree, h_firsts);
		get_nodes(h_pairs_mirror, h_firsts, h_nodes);
		h_firsts.push_back(h_firsts.back() + h_degree.back()); // add last

		h_target_degree.assign(h_pairs.size(), 0);
		thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(h_pairs.size()),
			countCommonNeighbours(RAWD(h_pairs), RAWD(h_pairs_mirror), RAWD(h_nodes), h_nodes.size(), RAWD(h_firsts), h_pairs_mirror.size(), RAWD(h_target_mirror), RAWD(h_target_firsts), RAWD(h_target_degree)));
		h_target_firsts.resize(h_target_degree.size());
		thrust::exclusive_scan(h_target_degree.begin(), h_target_degree.end(), h_target_firsts.begin());

		h_target_mirror.assign(thrust::reduce(h_target_degree.begin(), h_target_degree.end()), pair_t(0, 0));
		if (h_target_mirror.size() > 0){
			thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(h_pairs.size()),
				setCommonNeighbours(RAWD(h_pairs), RAWD(h_pairs_mirror), RAWD(h_nodes), h_nodes.size(), RAWD(h_firsts), h_pairs_mirror.size(), RAWD(h_target_mirror), RAWD(h_target_firsts), RAWD(h_target_degree)));
		}
		h_target_degree.erase(thrust::remove(h_target_degree.begin(), h_target_degree.end(), 0), h_target_degree.end());

		return 1;
	}

	struct increase_community{
		uint32_t* snapVec, *comVec, size, *target_ii, *target_ij;

		__host__ __device__
			increase_community(uint32_t* snapVec, uint32_t* comVec, uint32_t size, uint32_t* target_ii, uint32_t* target_ij) :
			snapVec(snapVec), comVec(comVec), size(size), target_ii(target_ii), target_ij(target_ij) {}

		__host__ __device__
			void operator()(pair_t p){

			// task 1 find common communities
			bool found;
			uint32_t n_first_it = g_binary_search(snapVec, snapVec + size, p.first, found);
			while (n_first_it > 0 && snapVec[n_first_it] == p.first)--n_first_it;
			if (snapVec[n_first_it] != p.first)++n_first_it;

			uint32_t n_seconh_it = g_binary_search(snapVec, snapVec + size, p.second, found);
			while (n_seconh_it > 0 && snapVec[n_seconh_it] == p.second)--n_seconh_it;
			if (snapVec[n_seconh_it] != p.second)++n_seconh_it;

			uint32_t n_first_nd = n_first_it;
			while (n_first_nd < size){
				if (snapVec[n_first_nd] != p.first)
					break;
				target_ij[comVec[n_first_nd]] += 1;
				//atomicAdd((uint32_t*)(target_ij + comVec[n_first_nd]), (uint32_t)1);
				++n_first_nd;
			}

			uint32_t n_seconh_nd = n_seconh_it;
			while (n_seconh_nd < size){
				if (snapVec[n_seconh_nd] != p.second)
					break;
				target_ij[comVec[n_seconh_nd]] += 1;
				//atomicAdd((uint32_t*)(target_ij + comVec[n_seconh_nd]), (uint32_t)1);
				++n_seconh_nd;
			}

			// find common neighbour
			while (n_first_it < size && n_seconh_it < size){
				if (snapVec[n_first_it] != p.first || snapVec[n_seconh_it] != p.second)break;
				if (comVec[n_first_it] == comVec[n_seconh_it]){
					target_ii[comVec[n_first_it]] += 1;
					//atomicAdd((uint32_t*)(target_ii + comVec[n_first_it]), (uint32_t)2);
					++n_first_it;
					++n_seconh_it;
				}
				else if (snapVec[n_first_it] < snapVec[n_seconh_it]){
					++n_first_it;
				}
				else{
					++n_seconh_it;
				}
			}

		}
	};
	/* Equation:
	* ki * kj
	*/
	void get_modularity(comevo::Source sPairs, comevo::Source sSnaps, uint32_t snapId, float &Q){

		vector<uint32_t> scom = sSnaps.get_scom(snapId);
		snapshot_t snap = sSnaps.get_snap(snapId);
		uint32_t com_max = *max_element(scom.begin(), scom.end());
		pairs_t pairs = sPairs.get_edges(snapId);
		T_HV<pair_t> h_pairs(pairs.begin(), pairs.end());
		int eTot = pairs.size();

		// go through all edges
		// store fraction that has both ends in one
		T_HV<uint32_t> eii(scom.size(), 0);
		T_HV<uint32_t> eij(scom.size(), 0);

		T_HV<uint32_t> h_snapVec;
		translate_snapshot_to_vector(snap, h_snapVec);
		T_HV<uint32_t> h_comVec;
		T_HV<uint32_t>h_comSizes(scom.begin(), scom.end());
		translate_scom_to_vector(h_comSizes, h_comVec);
		thrust::sort_by_key(h_snapVec.begin(), h_snapVec.end(), h_comVec.begin());

		thrust::for_each(thrust::host, h_pairs.begin(), h_pairs.end(), increase_community(RAWD(h_snapVec), RAWD(h_comVec), h_snapVec.size(), RAWD(eii), RAWD(eij)));

		Q = thrust::reduce(
			T_MTI(T_MZIMT(eii.begin(), T_MCONSI<uint32_t>(2 * eTot)), zip_div<uint32_t>()),
			T_MTI(T_MZIMT(eii.end(), T_MCONSI<uint32_t>(2 * eTot)), zip_div<uint32_t>()))
			- thrust::reduce(
			T_MTI(T_MTI(T_MZIMT(eij.begin(), T_MCONSI<uint32_t>(2 * eTot)), zip_div<uint32_t>()), set_square<float>()),
			T_MTI(T_MTI(T_MZIMT(eij.end(), T_MCONSI<uint32_t>(2 * eTot)), zip_div<uint32_t>()), set_square<float>()));
	}
}
