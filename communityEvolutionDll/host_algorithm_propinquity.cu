#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/host_algorithm_propinquity.h"
#include "include/host_analytic.h"
#include "include/general_pair_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/host_pair.h"
#include "include/data_info.h"
#include "include/display_elements.h"
#include "include/host_pair_construct.h"
#include "include/general_arithmetic_structs.h"
#include "include/host_storage_serilization.h"
#include "include/general_search.h"
#include "include/general_structs.h"
#include "include/device_storage_serilization.h"

using namespace std;

namespace comevohost{

#define CPULIMIT 5000000
#define CPULIMIT2 50000000

	struct g_bfs : public thrust::unary_function < uint32_t, uint32_t >
	{
		pair_t* pairs;
		uint32_t* firsts, *nodes, size, nodes_size;
		bool *queue, *visited, *pre_visited;

		__host__ __host__
			g_bfs(pair_t* pairs, uint32_t size, uint32_t* firsts, uint32_t* nodes, uint32_t nodes_size, bool* visited, bool* pre_visited, bool* queue) :
			pairs(pairs), size(size), firsts(firsts), nodes(nodes), nodes_size(nodes_size), visited(visited), pre_visited(pre_visited), queue(queue){}

		__host__ __host__
			void operator()(const uint32_t &x)
		{
			bool found;
			uint32_t count = 0;
			if (!queue[x])return;
			visited[x] = 1;
			// add neighbours
			uint32_t it_first = firsts[x];
			while (it_first < firsts[x + 1]){
				uint32_t v = pairs[it_first].second;
				uint32_t nmb = g_binary_search(nodes, nodes + nodes_size, v, found);
				// check node, add it, if visited
				if (!visited[nmb] && !pre_visited[nmb]){
					queue[nmb] = 1;
				}
				++it_first;
			}
			queue[x] = 0;
			return;

		}
	};

	static uint32_t storageCounter = 0;
	struct get_indices : public thrust::unary_function < uint32_t, bool >
	{
		uint32_t* firsts, *degree, n;
		__host__ __host__
			get_indices(uint32_t* firsts, uint32_t* degree, uint32_t n) : firsts(firsts), degree(degree), n(n){}

		__host__ __host__
			bool operator()(const uint32_t &x){
			for (uint32_t i = 0; i < n; ++i){
				if (firsts[i] + degree[i] > x){
					if (x >= firsts[i])return 1;
					return 0;
				}
			}
			return 0;
		}
	};


	void setStorageCounter(uint32_t value){
		storageCounter = value;
	}

	uint32_t getStorageCounter(){
		return storageCounter;
	}

	bool bfs(T_HV<pair_t>& hs_pairs, T_HV<uint8_t>& propinquities, uint32_t minimum, snapshot_t& communities){
		communities.clear();
		if (propinquities.size() == hs_pairs.size())
			hs_pairs.erase(thrust::remove_if(hs_pairs.begin(), hs_pairs.end(), propinquities.begin(), is_smaller<uint8_t>(minimum)), hs_pairs.end());
		if (hs_pairs.empty())return 0;
		T_HV<uint32_t> hs_degree;
		get_degree(hs_pairs, hs_degree);
		T_HV<pair_t> hs_pairs_mirror(hs_pairs.begin(), hs_pairs.end());
		mirror_pairs_inplace(hs_pairs_mirror);
        T_HV<uint32_t> number_of_diff_elements(T_MTI(hs_pairs_mirror.begin(), first_element()), T_MTI(hs_pairs_mirror.end(), first_element()));
        uint32_t n = get_number_of_diff_elements(number_of_diff_elements);
		TH_CLEAR(hs_pairs, pair_t);

		T_HV<uint32_t> hs_firsts, hs_nodes;
		get_firsts(hs_degree, hs_firsts);
		get_nodes(hs_pairs_mirror, hs_firsts, hs_nodes);
		hs_firsts.push_back(hs_firsts.back() + hs_degree.back()); // add last

		// init
		T_HV < bool > hs_queue(n, 0);
		T_HV<bool> hs_visited(n, 0);
		T_HV < bool > h_visited(n);
		T_HV<bool> total_visited(n, 0);
		T_HV<bool> hs_pre_visited(n);
		thrust::transform_if(hs_degree.begin(), hs_degree.end(), hs_pre_visited.begin(), set_value<uint32_t>(1), is_smaller<uint32_t>(1));
		thrust::copy(hs_pre_visited.begin(), hs_pre_visited.end(), total_visited.begin());

		while (total_visited.end() != thrust::find(total_visited.begin(), total_visited.end(), 0)){
			uint32_t s = thrust::find(total_visited.begin(), total_visited.end(), 0) - total_visited.begin();
			thrust::fill(hs_visited.begin(), hs_visited.end(), 0);

			// search from s
			hs_queue[s] = 1;

			// fill queue (check and add neighbours)
			uint32_t q_val = 1;
			do{
				T_TRYCATCH(
					thrust::for_each(thrust::host, T_MCI<U32>(0), T_MCI<U32>(n),
					g_bfs(
					RAWD(hs_pairs_mirror), hs_pairs_mirror.size(),
					RAWD(hs_firsts), RAWD(hs_nodes), hs_nodes.size(),
					RAWD(hs_visited), RAWD(hs_pre_visited), RAWD(hs_queue)
					));
				);
				q_val = thrust::count_if(thrust::host, hs_queue.begin(), hs_queue.end(), is_one<bool>());
			} while (q_val > 0);

			// set total
			thrust::copy(hs_visited.begin(), hs_visited.end(), h_visited.begin());
			thrust::transform(total_visited.begin(), total_visited.end(), h_visited.begin(), total_visited.begin(), thrust::logical_or<bool>());
			thrust::copy(total_visited.begin(), total_visited.end(), hs_pre_visited.begin());

			// create and add com
			T_HV < uint32_t > hs_com(hs_visited.size());
			hs_com.erase(thrust::copy_if(hs_nodes.begin(), hs_nodes.end(), hs_visited.begin(), hs_com.begin(), is_one<bool>()), hs_com.end());
			if (hs_com.size() > 2){
				T_HV < uint32_t > h_com(hs_com.begin(), hs_com.end());
				communities.push_back(vector < uint32_t >(h_com.begin(), h_com.end()));
			}
		}
		return 1;
	}

	/* The idea of this function is to compress stored files to their limit and removing empty files
	*/
	bool compress_files(){
		uint32_t cpuLimit = CPULIMIT;

		T_HV<pair_t> hs_pair(0), hs_fillingPair(0);
		T_HV<uint32_t> hs_val(0), hs_fillingVal(0);
		uint32_t offset, stCount = 0;
		uint32_t count = 0;
		for (uint32_t i = 0; i < storageCounter; ++i){
			cout << i << endl;
			uint32_t count = 0;
			// load data
			hs_pair.clear();
			hs_val.clear();
			if (!to_host_load(hs_pair, "pvp", i, true))return 0;
			if (!to_host_load(hs_val, "pvi", i, true))return 0;
			if (hs_pair.empty())continue;

			cout << i << " " << ++count << endl;
			cout << "size: " << hs_pair.size() << endl;
			if (hs_fillingPair.empty()){
				hs_fillingPair = hs_pair;
				hs_fillingVal = hs_val;
				continue;
			}
			// combine data
			combine_pairs(hs_fillingPair, hs_fillingVal, hs_pair, hs_val);
			// check size
			while (hs_fillingPair.size() > cpuLimit){
				from_host_store(hs_fillingPair, "pvp", 0, cpuLimit, stCount);
				from_host_store(hs_fillingVal, "pvi", 0, cpuLimit, stCount);
				++stCount;
				cout << i << " " << ++count << endl;
				// reduce
				hs_fillingPair.erase(hs_fillingPair.begin(), hs_fillingPair.begin() + cpuLimit);
				hs_fillingVal.erase(hs_fillingVal.begin(), hs_fillingVal.begin() + cpuLimit);
			}
		}
		// store the rest
		if (!from_host_store(hs_fillingPair, "pvp", 0, hs_fillingPair.size(), stCount))return 0;
		if (!from_host_store(hs_fillingVal, "pvi", 0, hs_fillingVal.size(), stCount)) return 0;
		++stCount;
		storageCounter = stCount;
		return 1;
	}
	uint32_t global_snap_id;
	bool cummulate_pairs(T_HV<pair_t>& hs_pairs, uint32_t offset, T_HV<uint32_t>& hs_cn){
		hs_cn.resize(hs_pairs.size(), 0);
		if (hs_pairs.empty())return 1;
		T_HV<bool> hs_found(hs_pairs.size());

		for (int i = offset; i < storageCounter; ++i){

			// load and init
			T_HV<pair_t>hs_pPair;
			T_HV<uint32_t>hs_pVal;
			if (!to_host_load(hs_pPair, "pvp", i, true))return 0;
			if (!to_host_load(hs_pVal, "pvi", i, true))return 0;
			if (hs_pPair.size() == 0 || hs_pPair.size() != hs_pVal.size()){
				if (!from_host_store(hs_pPair, "pvp", 0, 0, i))return 0;
				if (!from_host_store(hs_pVal, "pvi", 0, 0, i))return 0;
				continue;
			}
			combine_values(hs_pairs, hs_cn, hs_pPair, hs_pVal);

			T_HV<bool> hs_found(hs_pPair.size(), 0);
			thrust::binary_search(thrust::host, hs_pairs.begin(), hs_pairs.end(), hs_pPair.begin(), hs_pPair.end(), hs_found.begin());
		
			T_TRYCATCH(
				hs_pPair.erase(thrust::remove_if(hs_pPair.begin(), hs_pPair.end(), hs_found.begin(), is_one<bool>()), hs_pPair.end());
				hs_pVal.erase(thrust::remove_if(hs_pVal.begin(), hs_pVal.end(), hs_found.begin(), is_one<bool>()), hs_pVal.end());
				)

			// store
			if (!from_host_store(hs_pPair, "pvp", 0, hs_pPair.size(), i))return 0;
			if (!from_host_store(hs_pVal, "pvi", 0, hs_pVal.size(), i))return 0;
		}
		return 1;
	}

	bool set_new_pairs(T_HV<pair_t>& hs_pairs, uint32_t beta, T_HV<uint32_t>& hs_propinquity){

		for (int i = 0; i < storageCounter; ++i){
			T_HV<pair_t>hs_pPair;
			T_HV<uint32_t>hs_pVal;
			if (!to_host_load(hs_pPair, "pvp", i, true))return 0;
			if (!to_host_load(hs_pVal, "pvi", i, true))return 0;
			if (hs_pPair.size() == 0)continue;
			cummulate_pairs(hs_pPair, i + 1, hs_pVal);

			// 1. count relevant nodes
			uint32_t n_new_nodes = thrust::count_if(hs_pVal.begin(), hs_pVal.end(), is_greater<uint32_t>(beta - 1));
			uint32_t oldSize = hs_propinquity.size();
			hs_propinquity.resize(hs_propinquity.size() + n_new_nodes);

			uint32_t olhs_size = hs_pairs.size();
			hs_pairs.resize(hs_pairs.size() + n_new_nodes);

			// add relevant nodes
			thrust::copy_if(
				hs_pPair.begin(),
				hs_pPair.end(),
				hs_pVal.begin(),
				hs_pairs.begin() + olhs_size,
				is_greater<uint32_t>(beta - 1));
			thrust::copy_if(hs_pVal.begin(), hs_pVal.end(), hs_propinquity.begin() + oldSize, is_greater<uint32_t>(beta - 1));

			thrust::sort_by_key(hs_pairs.begin(), hs_pairs.end(), hs_propinquity.begin());
		}
		hs_pairs.resize(thrust::unique(hs_pairs.begin(), hs_pairs.end()) - hs_pairs.begin());
		return 1;
	}

	bool handle_increment(T_HV<pair_t>& all_pairs){
		T_HV<uint32_t>::iterator new_enhs_i;
		T_HV<pair_t>::iterator new_enhs_p;
		T_HV<pair_t>::iterator h_new_enhs_p;
		// sort and create map

		thrust::stable_sort(all_pairs.begin(), all_pairs.end());
		T_HV<pair_t> hs_unique_pairs;
		T_HV<uint32_t> hs_unique_values;
		get_count(all_pairs, hs_unique_pairs, hs_unique_values);

		uint32_t offset = 0;
		do{
            uint32_t allocate = min((unsigned long long)CPULIMIT, (unsigned long long)(hs_unique_pairs.size() - offset));
			from_host_store(hs_unique_pairs, "pvp", offset, allocate, storageCounter);
			from_host_store(hs_unique_values, "pvi", offset, allocate, storageCounter);

			offset += allocate;
			++storageCounter;
		} while (offset != hs_unique_pairs.size());

		return 1;
	}

	// needs mirror
	T_HV<pair_t> get_specific_pairs(T_HV<pair_t> &h_pairs, T_HV<uint32_t> &hs_firsts, T_HV<uint32_t> &hs_degree, T_HV<uint32_t> &hs_id){
		thrust::sort(hs_id.begin(), hs_id.end());
		T_HV<uint32_t> hs_degree_red, hs_firsts_red;
		hs_firsts_red.assign(
			T_MPI(hs_firsts.begin(), hs_id.begin()),
			T_MPI(hs_firsts.begin(), hs_id.end()));
		hs_degree_red.assign(
			T_MPI(hs_degree.begin(), hs_id.begin()),
			T_MPI(hs_degree.begin(), hs_id.end()));

		T_HV<uint32_t> hs_indices(thrust::reduce(hs_degree_red.begin(), hs_degree_red.end()), 0);
		thrust::copy_if(T_MCI<uint32_t>(0), T_MCI<uint32_t>(h_pairs.size()), hs_indices.begin(), get_indices(RAWD(hs_firsts_red), RAWD(hs_degree_red), hs_firsts_red.size()));
		TH_CLEAR(hs_firsts_red, uint32_t);
		TH_CLEAR(hs_degree_red, uint32_t);

		T_HV<uint32_t> h_indices(hs_indices.begin(), hs_indices.end());
		TH_CLEAR(hs_indices, uint32_t);
		T_HV<pair_t> hs_pairs(T_MPI(h_pairs.begin(), h_indices.begin()), T_MPI(h_pairs.begin(), h_indices.end()));
		return hs_pairs;
	}
	
	// needs mirrored
	bool couple_increment(T_HV<pair_t> &h_pairs, T_HV<uint32_t> &hs_degree, T_HV<uint32_t> &hs_firsts, uint32_t limit){

		if (h_pairs.size() == 0) return 1;
		// 3. split in small and big 
		// 3.1 calculate degree_limit
		T_HV<uint32_t> hs_smallId(0);
		T_HV<uint32_t> hs_idStorage(T_MCI<uint32_t>(0), T_MCI<uint32_t>(hs_degree.size()));

		T_HV<uint32_t> hs_degree_sorted(hs_degree.begin(), hs_degree.end());
		thrust::stable_sort_by_key(hs_degree_sorted.begin(), hs_degree_sorted.end(), hs_idStorage.begin());
		T_HV<uint32_t> scancombinations = get_max_combinations_scanned(hs_degree_sorted);
		TH_CLEAR(hs_degree_sorted, uint32_t);

		// space calculaion:
		uint32_t possiblePairs = limit; //  availableMemory;
		uint32_t possiblePairsBig = limit * 1000; //  availableMemory

		// calculate small ones until 90% done
		uint32_t limitId, doneId = 0, handled = 0, run = 0;

		// else not possible
		uint32_t limitPairs = possiblePairs;
		bool small = true;
		cout << "in the end: " << scancombinations.back() << " ids: " << hs_idStorage.size() << endl;
		if (scancombinations[0] < possiblePairsBig){
			do{
				limitId = thrust::upper_bound(scancombinations.begin(), scancombinations.end(), handled + limitPairs) - scancombinations.begin();

				if (limitId == doneId){
					return 1;
					if (doneId < scancombinations.size())
						cout << scancombinations[limitId] - handled << endl;
					cout << scancombinations.back() - handled << endl;
					scancombinations[limitId];

					limitPairs = possiblePairsBig;
					small = false;
					cout << "switch to big" << endl;
					continue;
				}
				++run;
				uint32_t cur_size = scancombinations[limitId - 1] - handled;
				cout << "handling this time run(" << run << "): " << cur_size << " doing: " << limitId << "todo: " << scancombinations.back() - handled << endl;
				if (cur_size != 0){
					hs_smallId.assign(hs_idStorage.begin() + doneId, hs_idStorage.begin() + limitId);

					// get pairs and work on it

					T_HV<pair_t> hs_pairs = get_specific_pairs(h_pairs, hs_firsts, hs_degree, hs_smallId);
					TH_CLEAR(hs_smallId, uint32_t);
					T_HV<pair_t>hs_target(0);
					if (small){
						if (!generate_pairs(hs_pairs, hs_target))return 0;
						if (!handle_increment(hs_target))return 0;
					}
					else{
						bool done = false;
						uint32_t off = 0;
						while (!done){
							if (!generate_pairs_limit(hs_pairs, hs_target, off, limit, done))return 0;
							//display_vector<uint32_t, uint32_t>(hs_target, "hs_target");
							if (!handle_increment(hs_target))return 0;
						}
					}
					TH_CLEAR(hs_pairs, pair_t);
				}
				doneId = limitId;
				handled = scancombinations[limitId - 1];
			} while (limitId / hs_idStorage.size() < 1);
		}
		else{
			cerr << "problem with size!" << endl;
		}

		return 1;
	}

	bool calculate_propinquity(T_HV<pair_t>& h_pairs){
		storageCounter = 0;
		T_HV<uint32_t> h_values(h_pairs.size(), 1);
		comevohost::from_host_store(h_pairs, "pvp", 0, h_pairs.size(), storageCounter);
		comevohost::from_host_store(h_values, "pvi", 0, h_values.size(), storageCounter);
		/*comevo::Serilization::store(T_HV<uint32_t>(
		T_MTI(h_pairs.begin(), first_element()),
		T_MTI(h_pairs.end(), first_element()))
		, "pvpf", storageCounter);
		comevo::Serilization::store(T_HV<uint32_t>(
		T_MTI(h_pairs.begin(), seconhs_element()),
		T_MTI(h_pairs.end(), seconhs_element()))
		, "pvps", storageCounter);
		comevo::Serilization::store(h_values, "pvi", storageCounter);*/
		h_values.clear();
		++storageCounter;

		T_HV<uint32_t> hs_degree, hs_firsts;
		T_HV<pair_t>hs_pairs_mirror(h_pairs.begin(), h_pairs.end());
		mirror_pairs_inplace(hs_pairs_mirror);
		T_HV<pair_t> h_pairs_mirror(hs_pairs_mirror.begin(), hs_pairs_mirror.end());

        T_HV<uint32_t> degree_mirror(T_MTI(hs_pairs_mirror.begin(), first_element()), T_MTI(hs_pairs_mirror.end(), first_element()));
        get_degree_mirror(degree_mirror, hs_degree);
		get_firsts(hs_degree, hs_firsts);
		TH_CLEAR(hs_pairs_mirror, pair_t);

		// 2. get limit (device_info)
		//pair_t p = get_device_memory();
		uint32_t availableMemory = (CPULIMIT2)* 0.9;
		uint32_t n = hs_degree.size();
		uint32_t m = h_pairs_mirror.size();

		uint32_t A = (availableMemory / 8 - (4 * n + 4 * m)) / 2;
		uint32_t B = (availableMemory / 8 - (4 * n + 2 * m)) / 5;
		uint32_t C = min(A, B);
		printf("avl: %d A: %d B: %d, C: %d \n", availableMemory, A, B, C);
		couple_increment(h_pairs_mirror, hs_degree, hs_firsts, C);

		T_HV<pair_t>hs_intersection;
        T_HV<pair_t> arg1(h_pairs.begin(), h_pairs.end());
        T_HV<pair_t> arg2(h_pairs_mirror.begin(), h_pairs_mirror.end());
        get_intersection(arg1, arg2, hs_intersection, hs_degree);
		if (hs_intersection.size() > 0){
			h_pairs_mirror.assign(hs_intersection.begin(), hs_intersection.end());
			TH_CLEAR(hs_intersection, pair_t);
			get_firsts(hs_degree, hs_firsts);

			couple_increment(h_pairs_mirror, hs_degree, hs_firsts, C);
		}
		TH_CLEAR(hs_intersection, pair_t);


		return 1;
	}

	bool update_graph(T_HV<pair_t>& h_pairs, Threshold &threshold, T_HV<uint32_t>& h_propinquity){
		// A: consider limits
		T_HV<pair_t> hs_pairs;
		T_HV<uint32_t> hs_degree, hs_firsts;
		hs_pairs.assign(h_pairs.begin(), h_pairs.end());

		// remove if lower than alpha, add if higher than beta
		// 1. find and remove existing ones, 
		T_HV<uint32_t> hs_cn;
		cummulate_pairs(hs_pairs, 0, hs_cn);

		// 2. if val low alpha remove
		hs_pairs.erase(thrust::remove_if(hs_pairs.begin(), hs_pairs.end(), hs_cn.begin(), is_smaller<uint32_t>(threshold.alpha)), hs_pairs.end());
		hs_cn.erase(thrust::remove_if(hs_cn.begin(), hs_cn.end(), is_smaller<uint32_t>(threshold.alpha)), hs_cn.end());
		// sumup all other pairs
		set_new_pairs(hs_pairs, threshold.beta, hs_cn);
		h_propinquity.assign(hs_cn.begin(), hs_cn.end());
		h_pairs.assign(hs_pairs.begin(), hs_pairs.end());

		return 1;
	}

	bool algorithm_propinquity(comevo::Source &source, comevo::Source &target, uint32_t from, uint32_t to, Threshold &threshold, uint32_t bfsMinimum, uint32_t maxIterations, uint32_t maxSnap){
		U32 propinquityLimit = maxIterations;
		U32 nSnaps = source.get_n().size(); // number of Snaps
		if (maxSnap != 0)nSnaps = maxSnap;
		vector<snapshot_t> snaps;
		for (U32 snapId = 0; snapId != nSnaps; ++snapId){
			cout << endl;
			cout << "snap: " << snapId << endl;
			global_snap_id = snapId;
			storageCounter = 0;
			snapshot_t communities(0);
			// get edges
			// A: consider limits
			uint32_t nPairs = source.get_m(snapId);
			if (nPairs == 0){
				snaps.push_back(communities);
				continue;
			}

			// 1. get degree
			T_HV<pair_t> h_pairs;
			T_HV<uint32_t> h_propinquity(0);
			if (to != 0)h_pairs = source.get_edges(snapId, from, to);
			if (to == 0)h_pairs = source.get_edges(snapId);
			uint32_t oldSize;
			// calc prop
			for (U32 run = 0; run < propinquityLimit; ++run){
				cout << "iteration: " << run << endl;
				if (h_pairs.empty())break;
				//display_vector<uint32_t, uint32_t>(h_pairs, "h_pairs");

				if (!calculate_propinquity(h_pairs))return 0;
				//cout << "stc: " << storageCounter << endl;
				// compress
				//if(!compress_files())return 0;
				// update
				oldSize = h_pairs.size();
				if (!update_graph(h_pairs, threshold, h_propinquity))return 0;
				storageCounter = 0;

				if (run > 2 && oldSize == h_pairs.size())break;
				//display_vector<uint32_t>(h_propinquity, "h_propinquity");
			}
			communities.clear();
            if (!h_pairs.empty()) {
                T_HV<pair_t> arg1(h_pairs.begin(), h_pairs.end());
                T_HV<uint8_t> arg2(0);
                bfs(arg1, arg2, bfsMinimum, communities);
            }
			snaps.push_back(communities);
		}
		vector<pairs_t> vecPairs;
		if (!target.set_source(vecPairs, snaps, SNAPS))return 0;


		return 1;
	}

}
