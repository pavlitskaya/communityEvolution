#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/algorithm_event_extraction.h"
#include "include/device_convert.h"
#include "include/general_arithmetic_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/host_storage_human.h"

using namespace std;

#define _OR 1
#define _AND 2

void displayTripleVector(T_DV<tuple_triple>& vec, char* name, bool split){
	std::cout << name << ": " << endl;
	for (T_DV<tuple_triple>::iterator it = vec.begin(); it != vec.end(); ++it){
		tuple_triple trip = *it;
		if (split)
			cout << thrust::get<0>(trip) << " into " << thrust::get<1>(trip) << " + " << thrust::get<2>(trip) << " # ";
		else
			cout << thrust::get<0>(trip) << " + " << thrust::get<1>(trip) << " into " << thrust::get<2>(trip) << " # ";
	}
	std::cout << "\n";
	std::cout << "\n";
}

void displayPairVector(T_DV<pair_ti>& vec, char* name, char* str){
	std::cout << name << ": " << endl;
	for (T_DV<pair_ti>::iterator it = vec.begin(); it != vec.end(); ++it){
		pair_ti p = *it;
		std::cout << p.first << " " << str << " " << p.second << " # ";
	}
	std::cout << "\n";
	std::cout << "\n";
}

template <typename T>
struct linear_index_to_row_index : public thrust::unary_function < T, T >
{
	T C; // number of columns

	__host__ __device__
		linear_index_to_row_index(T C) : C(C) {}

	__host__ __device__
		T operator()(T i)
	{
		return i / C;
	}
};
struct _sum_row_b
{
	bool* source;
	uint32_t n_cols, n_rows;
	uint32_t* target;

	__host__ __device__
		_sum_row_b(bool* source, uint32_t n_cols, uint32_t* target, uint32_t n_rows) :
		source(source), n_cols(n_cols), target(target), n_rows(n_rows) {}

	__device__
		void operator()(uint32_t i)
	{
		// mapped
		uint32_t x = (uint32_t)(i / n_rows);
		if (source[i] == 1)
			++target[x];
		//atomicAdd(target + x, 1);
	}
};

struct d_mergesplit{
	uint32_t *_or_ii_sum, *_and_ij_sum, *_or_iij_sum, *_j_sum;
	float* _i_f_sum;
	float kappa;
	tuple_triple* target;
	uint32_t i_cols, j_cols;
	bool split;
	__host__ __device__
		d_mergesplit(uint32_t i_cols, uint32_t j_cols, float kappa, tuple_triple* target,
		uint32_t *_or_ii_sum, uint32_t *_and_ij_sum, uint32_t *_or_iij_sum, float *_i_f_sum, uint32_t *_j_sum, bool split) :
		i_cols(i_cols), j_cols(j_cols), kappa(kappa), target(target),
		_or_ii_sum(_or_ii_sum), _and_ij_sum(_and_ij_sum), _or_iij_sum(_or_iij_sum), _i_f_sum(_i_f_sum), _j_sum(_j_sum), split(split) {}

	__host__ __device__
		void operator()(uint32_t i1_ind){
		// pouint32_t it:
		uint32_t tgt = i1_ind*i_cols;
		uint32_t tgt_last = i_cols*i_cols;
		uint32_t m = i_cols;
		uint32_t c2_m = j_cols;
		for (int i2_ind = i1_ind + 1; i2_ind < i_cols; ++i2_ind){
			for (int j = 0; j < j_cols; ++j){
				uint32_t tmp_ii = uint32_t((0.5*m*(m - 1) - 0.5*(m - i1_ind)*(m - i1_ind - 1) + (i2_ind - 1) - i1_ind));
				uint32_t tmp_ij = i1_ind*c2_m + j;
				uint32_t tmp_i2j = i2_ind*c2_m + j;
				if (_or_iij_sum[tmp_ii*c2_m + j] >= kappa * max((uint32_t)_or_ii_sum[tmp_ii], (uint32_t)_j_sum[j])){
					if (_and_ij_sum[tmp_ij] >= _i_f_sum[i1_ind]){
						if (_and_ij_sum[tmp_i2j] >= _i_f_sum[i2_ind]){
							while (thrust::get<0>(target[tgt]) >= 0 && tgt < tgt_last){
								++tgt;
							}
							if (tgt < tgt_last){
								if (split)
									target[tgt] = tuple_triple(j, (int)i1_ind, i2_ind);
								else
									target[tgt] = tuple_triple((int)i1_ind, i2_ind, j);
							}
						}
					}
				}
			}
		}
	}
};

struct d_continue{
	bool* _or_ij, *_and_ij;
	uint32_t j_cols, n;
	pair_ti* target;
	__host__ __device__
		d_continue(uint32_t n, uint32_t j_cols, bool* _and_ij, bool* _or_ij, pair_ti* target) : n(n), _or_ij(_or_ij), _and_ij(_and_ij), j_cols(j_cols), target(target) {}

	__host__ __device__
		void operator()(uint32_t i_ind){
		// pouint32_t it:
		uint32_t tgt = i_ind*j_cols;
		bool add;
		for (uint32_t j = 0; j < j_cols; ++j){
			add = true;
			for (uint32_t v = 0; v < n; ++v){
				if (_or_ij[(i_ind*j_cols + j)*n + v] != _and_ij[(i_ind*j_cols + j)*n + v]){
					add = false;
				}
			}
			if (add){
				while (target[tgt].first >= 0){
					++tgt;
				}
				target[tgt] = pair_ti(i_ind, j);
			}
		}
	}
};

struct d_join{
	pair_ti * target;
	float* _i_f_sum;
	uint32_t* _and_ij_sum;
	bool *d_preMatrix, *d_curMatrix;
	uint32_t nRelevant, i_cols, j_cols;
	__host__ __device__
		d_join(uint32_t nRelevant, uint32_t i_cols, uint32_t j_cols, uint32_t* _and_ij_sum, bool* d_preMatrix, bool* d_curMatrix, float* _i_f_sum, pair_ti* target) :
		nRelevant(nRelevant), i_cols(i_cols), j_cols(j_cols), _and_ij_sum(_and_ij_sum), d_preMatrix(d_preMatrix), d_curMatrix(d_curMatrix), _i_f_sum(_i_f_sum), target(target) {}

	__host__ __device__
		void operator()(uint32_t j){
		uint32_t tgt = j;
		for (uint32_t v = 0; v < nRelevant; ++v){
			if (d_curMatrix[j*nRelevant + v] == 1){
				for (uint32_t i = 0; i < i_cols; ++i){
					if (_and_ij_sum[i*j_cols + j] > _i_f_sum[i]){
						if (d_preMatrix[i*nRelevant + v] == 0){
							while (target[tgt].first >= 0){
								++tgt;
							}
							target[tgt] = (pair_ti(v, j));
						}
					}
				}
			}
		}
	}
};

struct d_leve{
	pair_ti * target;
	float* _i_f_sum;
	uint32_t* _and_ij_sum;
	bool *d_preMatrix, *d_curMatrix;
	uint32_t nRelevant, i_cols, j_cols;
	__host__ __device__
		d_leve(uint32_t nRelevant, uint32_t i_cols, uint32_t j_cols, uint32_t* _and_ij_sum, bool* d_preMatrix, bool* d_curMatrix, float* _i_f_sum, pair_ti* target) :
		nRelevant(nRelevant), i_cols(i_cols), j_cols(j_cols), _and_ij_sum(_and_ij_sum), d_preMatrix(d_preMatrix), d_curMatrix(d_curMatrix), _i_f_sum(_i_f_sum), target(target) {}

	__host__ __device__
		void operator()(uint32_t i){
		uint32_t tgt = i;
		for (uint32_t v = 0; v < nRelevant; ++v){
			if (d_preMatrix[i*nRelevant + v] == 1){
				for (uint32_t j = 0; j < j_cols; ++j){
					if (_and_ij_sum[i*j_cols + j] > _i_f_sum[i]){
						if (d_curMatrix[j*nRelevant + v] == 0){
							while (target[tgt].first >= 0){
								++tgt;
							}
							target[tgt] = (pair_ti(v, i));
						}
					}
				}
			}
		}
	}
};

struct d_dissolve{
	uint32_t* source;
	bool* target;
	uint32_t s1_size, s2_size;
	__host__ __device__
		d_dissolve(uint32_t s1_size, uint32_t s2_size, uint32_t* source, bool* target) : source(source), s1_size(s1_size), target(target), s2_size(s2_size) {}

	__host__ __device__
		void operator()(uint32_t s1_ind){
		// pouint32_t it:

		target[s1_ind] = 0;
	}
};

struct d_transpose{
	bool* source, *target;
	uint32_t row_size, col_size;
	__host__ __device__
		d_transpose(uint32_t row_size, uint32_t col_size, bool* source, bool* target) : source(source), row_size(row_size), target(target), col_size(col_size) {}

	__host__ __device__
		void operator()(uint32_t val){
		// y, x
		pair_t res = pair_t((uint32_t)val / row_size, val % row_size);

		// y*m + x
		*(target + res.second*col_size + res.first) = *(source + val);
	}
};


T_DV<uint32_t> sum_row(T_DV<uint32_t>& source, uint32_t m, uint32_t n){
	T_DV<uint32_t> result(m);
	T_DV<uint32_t> row_indices(m);

	thrust::reduce_by_key
		(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(m)),
		thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(m)) + (n*m),
		source.begin(),
		row_indices.begin(),
		result.begin());
	return result;
}

bool transpose(uint32_t row_size, uint32_t col_size, vector<bool>& source, vector<bool>& target){
	bool source_val = source.front();
	bool target_val = target.front();
	std::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(0) + row_size * col_size, d_transpose(
		row_size, col_size, &source_val, &target_val));

	return 1;
}

bool transpose(uint32_t row_size, uint32_t col_size, thrust::host_vector<bool>& source, thrust::host_vector<bool>& target){
	std::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(0) + row_size * col_size, d_transpose(
		row_size, col_size, RAWD(source), RAWD(target)));

	return 1;
}

struct set_diag_max : public thrust::unary_function < uint32_t, uint32_t >
{
	uint32_t C;
	uint32_t val;

	__host__ __device__
		set_diag_max(uint32_t C, uint32_t val) : C(C), val(val) {}

	__host__ __device__
		uint32_t operator()(uint32_t x) const
	{
		//printf("C: %d, val: %d, ind: %d, res: %d", C, val, x, x % (C + 1));
		return (x % (C + 1) == 0) ? val : 0;
	}
};

bool transpose(uint32_t row_size, uint32_t col_size, T_DV<bool>& source, T_DV<bool>& target){
	thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(0) + row_size * col_size, d_transpose(
		row_size, col_size, RAWD(source), RAWD(target)));

	return 1;
}

bool max_diagonal(T_DV<uint32_t>& source, uint32_t row_size){
	thrust::transform(T_MCI<uint32_t>(0), T_MCI<uint32_t>(row_size*row_size), source.begin(), set_diag_max(row_size, 1000));
	return 1;
}

bool sum_row(vector<bool>& source, uint32_t n, vector<uint32_t>& result){

	uint32_t m = source.size() / n;
	result.resize(m);
	vector<uint32_t> row_indices(m);
	try
	{
		thrust::reduce_by_key
			(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)),
			thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)) + (n*m),
			source.begin(),
			row_indices.begin(),
			result.begin());
	}
	catch (thrust::system_error &e)
	{
		// output an error message and exit
		printf("parameters are: n=%d", n);
		std::cerr << "Error accessing vector element: " << e.what() << std::endl;
		system("pause");
		exit(-1);
	}
	return 1;
}

/*
bool sum_row(T_DV<bool>& source, uint32_t n, T_DV<uint32_t>& result){
uint32_t m = source.size() / n;
result.resize(m);
T_DV<uint32_t> row_indices(m);
uint32_t n_done = 0, n_todo = 0, n_rest = 0, cur_start = 0, cur_end = 0;
set_start_parameter(n_done, n_todo, n_rest, cur_start, cur_end, n, 1);
while (n_todo != 0){
try
{
thrust::reduce_by_key
(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(cur_start), linear_index_to_row_index<uint32_t>(cur_end)),
thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(cur_start), linear_index_to_row_index<uint32_t>(cur_end)) + (n_todo*m),
source.begin() + cur_start,
row_indices.begin() + cur_start,
result.begin() + cur_start);
}
catch (thrust::system_error &e)
{
// output an error message and exit
printf("parameters are: n=%d \n", n);
std::cerr << "Error accessing vector element: " << e.what() << std::endl;
system("pause");
exit(-1);
}
change_parameter(n_done, n_todo, n_rest, cur_start, cur_end, n, 1);
}

return 1;
}
*/

bool sum_row(T_DV<bool>& source, uint32_t n, T_DV<uint32_t>& result){

	uint32_t m = source.size() / n;
	result.resize(m);
	T_DV<uint32_t> row_indices(m);

	thrust::for_each_n(T_MCI<uint32_t>(0), m*n, _sum_row_b(
		RAWD(source), m, RAWD(result), n));

	return 1;
}


bool sum_row(thrust::host_vector<bool>& source, uint32_t n, thrust::host_vector<uint32_t>& result){

	uint32_t m = source.size() / n;
	result.resize(m);
	thrust::host_vector<uint32_t> row_indices(m);

	thrust::reduce_by_key
		(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)),
		thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)) + (n*m),
		source.begin(),
		row_indices.begin(),
		result.begin());
	return 1;
}

struct d_ii {
	uint32_t n, m, s1_ind, op;
	bool *source, *target;

	__host__ __device__
		d_ii(uint32_t n, bool *source, uint32_t m, uint32_t s1_ind, bool *target, uint32_t op) : n(n), source(source), m(m), s1_ind(s1_ind), target(target), op(op) {}

	__host__ __device__
		void operator()(uint32_t s2_ind){

		bool* s1_ptr = source + s1_ind*n;
		bool* s2_ptr = source + s2_ind*n;
		uint32_t tmp = uint32_t((0.5*m*(m - 1) - 0.5*(m - s1_ind)*(m - s1_ind - 1) + (s2_ind - 1) - s1_ind));
		bool* t_ptr = target + tmp * n;

		for (uint32_t i = 0; i < n; ++i){
			if (op == 1)*t_ptr = *s1_ptr || *s2_ptr;
			if (op == 2)*t_ptr = *s1_ptr && *s2_ptr;
			++t_ptr;
			++s1_ptr;
			++s2_ptr;
		}
	}
};

struct d_ij {
	uint32_t n, s1_m, s2_m, s1_ind, op;
	bool *s1_source, *s2_source, *target;

	__host__ __device__
		d_ij(uint32_t n, bool *s1_source, uint32_t s1_m, uint32_t s1_ind, bool *s2_source, uint32_t s2_m, bool *target, uint32_t op) :
		n(n), s1_source(s1_source), s1_m(s1_m), s1_ind(s1_ind), s2_source(s2_source), s2_m(s2_m), target(target), op(op) {}

	__host__ __device__
		void operator()(uint32_t s2_ind){

		bool* s1_ptr = s1_source + s1_ind*n;
		bool* s2_ptr = s2_source + s2_ind*n;
		uint32_t tmp = s1_ind*s2_m + s2_ind;
		bool* t_ptr = target + tmp * n;

		for (uint32_t i = 0; i < n; ++i){
			if (op == 1)*t_ptr = *s1_ptr || *s2_ptr;
			if (op == 2)*t_ptr = *s1_ptr && *s2_ptr;
			++t_ptr;
			++s1_ptr;
			++s2_ptr;
		}

	}
};

bool ii(uint32_t n, T_DV<bool>& d_preMatrix, uint32_t k_preCom, T_DV<bool>& _ii, uint32_t op){
	uint32_t _ii_size = 0.5*(k_preCom - 1)*k_preCom * n;
	_ii.resize(_ii_size, 0);

	// for each row
	for (uint32_t ind = 0; ind < k_preCom - 1; ++ind){

		thrust::for_each(T_MCI<uint32_t>(0) + ind + 1, T_MCI<uint32_t>(0) + k_preCom,
			d_ii(n, RAWD(d_preMatrix), k_preCom, ind, RAWD(_ii), op));
	}

	return 1;
}

bool ij(uint32_t n, T_DV<bool>& d_preMatrix, uint32_t k_preCom, T_DV<bool>& d_curMatrix, uint32_t k_curCom, T_DV<bool>& _ij, uint32_t op){
	uint32_t _ij_size = k_preCom*k_curCom * n;
	_ij.resize(_ij_size, 0);

	// for each row
	for (uint32_t ind = 0; ind < k_preCom; ++ind){

		thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(0) + k_curCom,
			d_ij(n, RAWD(d_preMatrix), k_preCom, ind,
			RAWD(d_curMatrix), k_curCom,
			RAWD(_ij), op));
	}

	return 1;
}
/*
*/

/* Asurs Event Extraction
 */
bool algorithm_event_extraction(comevo::Source &source, float k, bool display, bool create_file){
	// number of snaps and communities
	vector<community_t> communitySizes = source.get_scom();
	
	snapshot_t preSnap;
	snapshot_t curSnap;
	uint32_t preId = 0, curId, nRelevant;
	T_HV<uint32_t> preSizes, curSizes;
	T_DV<uint32_t> d_preSizes, d_curSizes;
	T_DV<uint32_t> d_preSnapVec, d_curSnapVec;
	T_DV<uint32_t> d_preSnapVecSort, d_curSnapVecSort;
	T_DV<uint32_t> d_preSnapVecUnique, d_curSnapVecUnique;
	T_DV<bool> d_preMatrix, d_curMatrix;
	uint32_t k_preCom, k_curCom;
	if (communitySizes.size() < 2)
		return 1;
	
	preSnap = source.get_snap(preId);
	preSizes = communitySizes[preId];
	d_preSizes.assign(preSizes.begin(), preSizes.end());
	translate_snapshot_to_vector(preSnap, d_preSnapVec);
	d_preSnapVecUnique.assign(d_curSnapVec.begin(), d_curSnapVec.end());
	thrust::sort(d_preSnapVecUnique.begin(), d_preSnapVecUnique.end());
	d_preSnapVecUnique.erase(thrust::unique(d_preSnapVecUnique.begin(), d_preSnapVecUnique.end()), d_preSnapVecUnique.end());
	k_preCom = d_preSizes.size();

	// for each two Snapshots
	for (uint32_t snapId = 1; snapId < communitySizes.size(); ++snapId){
		cout << "current snaps: " << preId << " and " << snapId << endl;

		// extract snaps, prepare values
		preId = snapId - 1;
		curId = snapId;
		curSnap = source.get_snap(curId);
		curSizes = communitySizes[curId];
		d_curSizes.assign(curSizes.begin(), curSizes.end());
		
		k_curCom = d_curSizes.size();
		// translate Snapshot into Vector
		translate_snapshot_to_vector(curSnap, d_curSnapVec);
		d_curSnapVecUnique.assign(d_curSnapVec.begin(), d_curSnapVec.end());
		thrust::sort(d_curSnapVecUnique.begin(), d_curSnapVecUnique.end());
		d_curSnapVecUnique.erase(thrust::unique(d_curSnapVecUnique.begin(), d_curSnapVecUnique.end()), d_curSnapVecUnique.end());
		
		// get relevant nodes
		T_DV<uint32_t> d_relevant(d_preSnapVecUnique.size() + d_curSnapVecUnique.size());
		d_relevant.erase(thrust::set_union(d_preSnapVecUnique.begin(), d_preSnapVecUnique.end(), d_curSnapVecUnique.begin(), d_curSnapVecUnique.end(), d_relevant.begin()), d_relevant.end());
		nRelevant = d_relevant.size();

		if (nRelevant > 0 && d_preSnapVec.size() > 0 && d_curSnapVec.size() > 0){

			// create Matrices of two Snapshots
			translate_snapshot_to_matrix(d_preSnapVec, d_preSizes, d_relevant, d_preMatrix);
			translate_snapshot_to_matrix(d_curSnapVec, d_curSizes, d_relevant, d_curMatrix);
			//display_vector<bool>(d_preMatrix, 0, nRelevant, "d_preMatrix");
			//display_vector<bool>(d_curMatrix, 0, nRelevant, "d_curMatrix");

			T_DV<uint32_t> _i_occurences;
			T_DV<uint32_t> _j_occurences;
			/*
			T_DV<bool> d_preMatrix_t(d_preMatrix.size(), 0);
			transpose(nRelevant, k_preCom, d_preMatrix, d_preMatrix_t);
			sum_row(d_preMatrix_t, k_preCom, _i_occurences);

			T_DV<bool> d_curMatrix_t(d_curMatrix.size(), 0);
			transpose(nRelevant, k_curCom, d_curMatrix, d_curMatrix_t);
			sum_row(d_curMatrix_t, k_curCom, _j_occurences);
			*/
			// start calculation: 
			// create iterator
			// A, A*: or_ii
			uint32_t _or_ii_n = 0.5*(k_preCom - 1)*k_preCom;
			T_DV<bool> _or_ii(_or_ii_n * nRelevant);
			ii(nRelevant, d_preMatrix, k_preCom, _or_ii, _OR);
			T_DV<uint32_t> _or_ii_sum;
			sum_row(_or_ii, nRelevant, _or_ii_sum);



			// or_jj
			uint32_t _or_jj_n = 0.5*(k_curCom - 1)*k_curCom;
			T_DV<bool> _or_jj(_or_jj_n * nRelevant);
			ii(nRelevant, d_curMatrix, k_curCom, _or_jj, _OR);
			T_DV<uint32_t> _or_jj_sum;
			sum_row(_or_jj, nRelevant, _or_jj_sum);

			// E
			T_DV<uint32_t> _i_sum;
			sum_row(d_preMatrix, nRelevant, _i_sum);
			T_DV<float> _i_f_sum(_i_sum.size());
			thrust::transform(_i_sum.begin(), _i_sum.end(), _i_f_sum.begin(), set_multiply<float>(0.5));

			// F
			T_DV<uint32_t> _j_sum;
			sum_row(d_curMatrix, nRelevant, _j_sum);
			T_DV<float> _j_f_sum(_j_sum.size());
			thrust::transform(_j_sum.begin(), _j_sum.end(), _j_f_sum.begin(), set_multiply<float>(0.5));

			// AF
			T_DV<bool> _and_iij;
			ij(nRelevant, _or_ii, _or_ii_n, d_curMatrix, k_curCom, _and_iij, _AND);
			T_DV<uint32_t> _or_iij_sum;
			sum_row(_and_iij, nRelevant, _or_iij_sum);

			T_DV<bool> _and_jji;
			ij(nRelevant, _or_jj, _or_jj_n, d_preMatrix, k_preCom, _and_jji, _AND);
			T_DV<uint32_t> _or_jji_sum;
			sum_row(_and_jji, nRelevant, _or_jji_sum);

			// B: or_ij
			T_DV<bool> _or_ij;
			ij(nRelevant, d_preMatrix, k_preCom, d_curMatrix, k_curCom, _or_ij, _OR);

			T_DV<bool> _or_ji;
			ij(nRelevant, d_curMatrix, k_curCom, d_preMatrix, k_preCom, _or_ji, _OR);

			// D, D*: and_ij
			T_DV<bool> _and_ij;
			ij(nRelevant, d_preMatrix, k_preCom, d_curMatrix, k_curCom, _and_ij, _AND);
			T_DV<uint32_t> _and_ij_sum;
			sum_row(_and_ij, nRelevant, _and_ij_sum);

			T_DV<bool> _and_ji;
			ij(nRelevant, d_curMatrix, k_curCom, d_preMatrix, k_preCom, _and_ji, _AND);
			T_DV<uint32_t> _and_ji_sum;
			sum_row(_and_ji, nRelevant, _and_ji_sum);

			// G
			if (k_preCom > 0){
				T_DV<bool> d_preMatrix_t(d_preMatrix.size(), 0);
				transpose(nRelevant, k_preCom, d_preMatrix, d_preMatrix_t);
				//T_DV<uint32_t> _i_occurences;
				sum_row(d_preMatrix_t, k_preCom, _i_occurences);
			}

			// H
			if (k_curCom > 0){
				T_DV<bool> d_curMatrix_t(d_curMatrix.size(), 0);
				transpose(nRelevant, k_curCom, d_curMatrix, d_curMatrix_t);
				//T_DV<uint32_t> _j_occurences;
				sum_row(d_curMatrix_t, k_curCom, _j_occurences);
			}
	//*/

			// do the real work
			//displayVector(d_preMatrix, "d_preMatrix", nRelevant);
			//displayVector(d_curMatrix, "d_curMatrix", nRelevant);
			T_DV<bool> _dissolve;
			T_DV<bool> _form;
			T_DV<tuple_triple> _merge;
			T_DV<tuple_triple> _split;
			T_DV<pair_ti> _continue;
			T_DV<bool> _appear;
			T_DV<bool> _disappear;
			T_DV<pair_ti> _join;
			T_DV<pair_ti> _leve;
			// dissolve
			if (k_preCom > 0 && k_curCom > 0){
				_dissolve.assign(k_preCom, 0);

				for (uint32_t i = 0; i < k_preCom; ++i){
					T_DV<uint32_t>::iterator it = thrust::max_element(_and_ij_sum.begin() + i * k_curCom, _and_ij_sum.begin() + i * k_curCom + k_curCom);
					if (*it < 1)
						_dissolve[i] = 1;
				}

				if (display)display_vector<bool>(_dissolve, "dissolve");
			}

			// form
			if (k_preCom > 0 && k_curCom > 0){
				_form.assign(k_curCom, 0);

				for (uint32_t i = 0; i < k_curCom; ++i){
					T_DV<uint32_t>::iterator it = thrust::max_element(_and_ji_sum.begin() + i * k_preCom, _and_ji_sum.begin() + i * k_preCom + k_preCom);
					if (*it < 1)
						_form[i] = 1;
				}

				if (display)display_vector<bool>(_form, "_form");
			}

			// merge
			if (k_preCom > 1 && k_curCom > 0){
				float kappa = 0.5;
				_merge.assign(k_preCom*k_preCom, -1);
				thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_preCom - 1), d_mergesplit(
					k_preCom, k_curCom, kappa, RAWD(_merge), RAWD(_or_ii_sum), RAWD(_and_ij_sum),
					RAWD(_or_iij_sum), RAWD(_i_f_sum), RAWD(_j_sum), false
					));

				T_DV<tuple_triple>::iterator tend = thrust::remove_if(_merge.begin(), _merge.end(), is_negative_triple());
				_merge.resize(tend - _merge.begin());
				//displayTripleVector(_merge, "_merge", false);
			}


			// split
			if (k_preCom > 0 && k_curCom > 1){
				float kappa = 0.5;
				_split.assign(k_curCom*k_curCom, -1);

				thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_curCom - 1), d_mergesplit(
					k_curCom, k_preCom, kappa, RAWD(_split), RAWD(_or_jj_sum), RAWD(_and_ji_sum),
					RAWD(_or_jji_sum), RAWD(_j_f_sum), RAWD(_i_sum), true
					));

				T_DV<tuple_triple>::iterator tend = thrust::remove_if(_split.begin(), _split.end(), is_negative_triple());
				_split.resize(tend - _split.begin());
				if (display)displayTripleVector(_split, "_split", true);
			}

			// continue
			if (k_preCom > 0 && k_curCom > 0){
				_continue.assign(k_preCom*k_curCom, pair_ti(-1, -1));

				thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_preCom), d_continue(
					nRelevant, k_curCom, RAWD(_and_ij), RAWD(_or_ij), RAWD(_continue)
					));

				T_DV<pair_ti>::iterator tend = thrust::remove_if(_continue.begin(), _continue.end(), is_negative_pair());
				_continue.resize(tend - _continue.begin());

				if (display)display_vector<int, int>(_continue, "_continue");
			}

			// appear
			if (k_preCom > 0){
				_appear.assign(nRelevant, 0);
				for (uint32_t i = 0; i < nRelevant; ++i){
					if (_i_occurences[i] == 0)
						if (_j_occurences[i] == 1)
							_appear[i] = 1;
				}
				if (display)display_vector<bool>(_appear, "_appear");
			}

			// disappear
			if (k_curCom > 0){
				_disappear.assign(nRelevant, 0);
				for (uint32_t i = 0; i < nRelevant; ++i){
					if (_i_occurences[i] == 1)
						if (_j_occurences[i] == 0)
							_disappear[i] = 1;
				}
				if (display)display_vector<bool>(_disappear, "_disappear");
			}

			// join
			if (k_curCom > 0){
				_join.assign(k_curCom*nRelevant, pair_ti(-1, -1));

				thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_curCom), d_join(
					nRelevant, k_preCom, k_curCom, RAWD(_and_ij_sum), RAWD(d_preMatrix),
					RAWD(d_curMatrix), RAWD(_i_f_sum), RAWD(_join)
					));

				T_DV<pair_ti>::iterator tend = thrust::remove_if(_join.begin(), _join.end(), is_negative_pair());
				_join.resize(tend - _join.begin());

				if (display)displayPairVector(_join, "_join", "to");
			}

			// leave
			if (k_preCom > 0){
				_leve.assign(k_preCom*nRelevant, pair_ti(-1, -1));

				thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_preCom), d_leve(
					nRelevant, k_preCom, k_curCom, RAWD(_and_ij_sum), RAWD(d_preMatrix),
					RAWD(d_curMatrix), RAWD(_i_f_sum), RAWD(_leve)
					));

				T_DV<pair_ti>::iterator tend = thrust::remove_if(_leve.begin(), _leve.end(), is_negative_pair());
				_leve.resize(tend - _leve.begin());

				if(display)displayPairVector(_leve, "_leve", "from");
			}

			if (create_file){
				time_t t = time(0);
				stringstream ss;
				ss << "events_";
				ss << t;
				string filename = ss.str();

                std::ifstream fileExistanceTest(filename);
                while(fileExistanceTest.good()) {
                    filename.append(std::string("-0"));
                    fileExistanceTest.open(filename);
                }
                fileExistanceTest.close();


				T_HV<bool> h_dissolve = _dissolve;
				T_HV<bool> h_form = _form;
				T_HV<tuple_triple> h_merge = _merge;
				T_HV<tuple_triple> h_split = _split;
				T_HV<pair_ti> h_continue = _continue;
				T_HV<pair_ti> h_join = _join;
				T_HV<pair_ti> h_leve = _leve;

                std::vector<bool> h_dissolve_c(h_dissolve.begin(), h_dissolve.end());
                std::vector<bool> h_form_c(h_form.begin(), h_form.end());
                std::vector<tuple_triple> h_merge_c(h_merge.begin(), h_merge.end());
                std::vector<tuple_triple> h_split_c(h_split.begin(), h_split.end());
                std::vector<pair_ti> h_continue_c(h_continue.begin(), h_continue.end());
                std::vector<pair_ti> h_join_c(h_join.begin(), h_join.end());
                std::vector<pair_ti> h_leve_c(h_leve.begin(), h_leve.end());
				comevo::save_to_file(filename,
                                     preId,
                                     curId,
                     h_dissolve_c,
                     h_form_c,
                     h_merge_c,
                     h_split_c,
                     h_continue_c,
                     thrust::count(_appear.begin(), _appear.end(), 1),
                     thrust::count(_disappear.begin(), _disappear.end(), 1),
                     h_leve_c,
                     h_join_c
                    );
			}

			printf("sizes: \n dissolve: %lu, form: %lu, merge: %lu, split: %lu, continue: %lu \n appear: %lu, disappear: %lu, join: %lu, leave: %lu \n\n", 
				thrust::count(_dissolve.begin(), _dissolve.end(), 1),
				thrust::count(_form.begin(), _form.end(), 1),
                _merge.size(),
				_split.size(),
				_continue.size(),
				thrust::count(_appear.begin(), _appear.end(), 1),
				thrust::count(_disappear.begin(), _disappear.end(), 1),
				_join.size(),
				_leve.size()
			);


		}

		preSnap = curSnap;
		preSizes = curSizes;
		d_preSizes = d_curSizes;
		d_preSnapVec = d_curSnapVec;
		d_preSnapVecSort = d_curSnapVecSort;
		d_preSnapVecUnique = d_curSnapVecUnique;
		d_preMatrix = d_curMatrix;
		k_preCom = k_curCom;
		preId = curId;

	}

	
	return 1;
}
//*/
