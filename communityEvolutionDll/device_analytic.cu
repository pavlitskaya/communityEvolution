#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/device_analytic.h"
#include "include/data_source.h"
#include "include/general_comparsion_structs.h"
#include "include/general_arithmetic_structs.h"
#include "include/display_elements.h"
#include "include/general_pair_structs.h"
#include "include/device_pair.h"
#include "include/device_analytic_structs.h"
#include "include/general_search.h"
#include "include/device_convert.h"


using namespace std;

uint32_t get_number_of_diff_elements(T_DV<uint32_t>& d_source){
	return get_number_of_diff_elements(d_source.begin(), d_source.end());
}

uint32_t get_number_of_diff_elements(T_DV<uint32_t>::iterator first, T_DV<uint32_t>::iterator last){
	if (last - first < 2)return (last - first);
	return thrust::count_if(
		T_MTI(T_MZIMT(first, first + 1), is_equal_to()),
		T_MTI(T_MZIMT(last - 1, last), is_equal_to()),
		thrust::logical_not<bool>()) + 1;
}

uint32_t get_number_of_diff_elements(T_DV<pair_t>& d_source){
	return get_number_of_diff_elements(d_source.begin(), d_source.end());
}

uint32_t get_number_of_diff_elements(T_DV<pair_t>::iterator d_source_first, T_DV<pair_t>::iterator d_source_last){
	if (d_source_last - d_source_first < 2)return (d_source_last - d_source_first);
	return thrust::count_if(
		T_MTI(T_MZIMT(d_source_first, d_source_first + 1), is_equal_to_pair()),
		T_MTI(T_MZIMT(d_source_last - 1, d_source_last), is_equal_to_pair()),
		thrust::logical_not<bool>()) + 1;
}

bool get_count(T_DV<pair_t>::iterator d_source_first, T_DV<pair_t>::iterator d_source_last, T_DV<pair_t>& d_unique_pairs, T_DV<uint32_t>& d_unique_count){
	uint32_t n = get_number_of_diff_elements(d_source_first, d_source_last);
	d_unique_pairs.assign(n, pair_t(0,0));
	d_unique_pairs.erase(thrust::unique_copy(d_source_first, d_source_last, d_unique_pairs.begin()), d_unique_pairs.end());
	d_unique_count.assign(n, 0);
	thrust::reduce_by_key(d_source_first, d_source_last, thrust::make_constant_iterator<uint32_t>(1), thrust::make_discard_iterator(), d_unique_count.begin());
	return 1;
}

bool get_count(T_DV<pair_t>& d_source, T_DV<pair_t>& d_unique_pairs, T_DV<uint32_t>& d_unique_count){
	return get_count(d_source.begin(), d_source.end(), d_unique_pairs, d_unique_count);
}

// needs mirrored
bool get_degree_mirror(T_DV<uint32_t>& d_source, T_DV<uint32_t>& d_degree){
	return get_degree_mirror(d_source.begin(), d_source.end(), d_degree);
}

bool get_degree_mirror(T_DV<pair_t>& d_source, T_DV<uint32_t>& d_degree){
    T_DV<uint32_t> source(T_MTI(d_source.begin(), first_element()), T_MTI(d_source.end(), first_element()));
    return get_degree_mirror(source, d_degree);
}

bool get_degree_mirror(T_DV<uint32_t>::iterator d_source_first, T_DV<uint32_t>::iterator d_source_last, T_DV<uint32_t>& d_degree){
	uint32_t n = get_number_of_diff_elements(d_source_first, d_source_last);
	d_degree.assign(n, 0);
	thrust::reduce_by_key(d_source_first, d_source_last, thrust::make_constant_iterator<uint32_t>(1), thrust::make_discard_iterator(), d_degree.begin());
	return 1;
}

// needs one way source
bool get_degree(T_DV<pair_t>& d_source, T_DV<uint32_t>& d_degree){
	T_DV<uint32_t> d_nodes;
	pairsToNodes(d_source, d_nodes);
	thrust::sort(d_nodes.begin(), d_nodes.end());
	get_degree_mirror(d_nodes, d_degree);
	return 1;
}

bool get_nodes(T_DV<pair_t>& d_pairs, T_DV<uint32_t>& d_firsts, T_DV<uint32_t>& d_nodes){
	d_nodes.resize(d_firsts.size());
	thrust::copy(T_MTI(T_MPI(d_pairs.begin(), d_firsts.begin()), first_element()), T_MTI(T_MPI(d_pairs.begin(), d_firsts.end()), first_element()), d_nodes.begin());
	return 1;
}

bool get_firsts(T_DV<uint32_t>& d_degree, T_DV<uint32_t>& d_firsts){
	d_firsts.resize(d_degree.size());
	thrust::exclusive_scan(d_degree.begin(), d_degree.end(), d_firsts.begin());
	return 1;
}

bool get_lasts(T_DV<uint32_t>& d_degree, T_DV<uint32_t>& d_lasts){
	d_lasts.resize(d_degree.size());
	thrust::inclusive_scan(d_degree.begin(), d_degree.end(), d_lasts.begin());
	return 1;
}

T_DV<uint32_t> get_max_combinations(T_DV<uint32_t>& d_degree){
	return T_DV<uint32_t>(
		T_MTI(T_MTI(T_MZIMT(d_degree.begin(), T_MTI(d_degree.begin(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)),
		T_MTI(T_MTI(T_MZIMT(d_degree.end(), T_MTI(d_degree.end(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)));
}

uint32_t get_max_combination(T_DV<uint32_t>& d_degree){
	return get_max_combination(d_degree.begin(), d_degree.end());
}

uint32_t get_max_combination(T_DV<uint32_t>::iterator d_degree_first, T_DV<uint32_t>::iterator d_degree_last){
	return thrust::reduce(
		T_MTI(T_MZIMT(d_degree_first, T_MTI(d_degree_first, set_decrease<uint32_t>())), zip_mul<uint32_t>()),
		T_MTI(T_MZIMT(d_degree_last, T_MTI(d_degree_last, set_decrease<uint32_t>())), zip_mul<uint32_t>()))*0.5;
}

T_DV<uint32_t> get_max_combinations_scanned(T_DV<uint32_t>& d_degree){
	T_DV<uint32_t> result(d_degree.size());
	thrust::inclusive_scan(T_MTI(T_MTI(T_MZIMT(d_degree.begin(), T_MTI(d_degree.begin(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)),
		T_MTI(T_MTI(T_MZIMT(d_degree.end(), T_MTI(d_degree.end(), set_decrease<uint32_t>())), zip_mul<uint32_t>()), set_multiply<uint32_t>((float)0.5)), result.begin());
	return result;
}

bool get_intersection(T_DV<pair_t>& d_pairs, T_DV<pair_t>& d_pairs_mirror, T_DV<pair_t>& d_target_mirror, T_DV<uint32_t>& d_target_degree){
	T_DV<uint32_t> d_target_firsts;
	
	T_DV<uint32_t> d_degree, d_firsts, d_nodes;
	get_degree_mirror(d_pairs_mirror, d_degree);
	get_firsts(d_degree, d_firsts);
	get_nodes(d_pairs_mirror, d_firsts, d_nodes);
	d_firsts.push_back(d_firsts.back() + d_degree.back()); // add last

	d_target_degree.assign(d_pairs.size(), 0);
	thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(d_pairs.size()),
		countCommonNeighbours(RAWD(d_pairs), RAWD(d_pairs_mirror), RAWD(d_nodes), d_nodes.size(), RAWD(d_firsts), d_pairs_mirror.size(), RAWD(d_target_mirror), RAWD(d_target_firsts), RAWD(d_target_degree)));
	d_target_firsts.resize(d_target_degree.size());
	thrust::exclusive_scan(d_target_degree.begin(), d_target_degree.end(), d_target_firsts.begin());

	d_target_mirror.assign(thrust::reduce(d_target_degree.begin(), d_target_degree.end()), pair_t(0, 0));
	if (d_target_mirror.size() > 0){
		thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(d_pairs.size()),
			setCommonNeighbours(RAWD(d_pairs), RAWD(d_pairs_mirror), RAWD(d_nodes), d_nodes.size(), RAWD(d_firsts), d_pairs_mirror.size(), RAWD(d_target_mirror), RAWD(d_target_firsts), RAWD(d_target_degree)));
	}
	d_target_degree.erase(thrust::remove(d_target_degree.begin(), d_target_degree.end(), 0), d_target_degree.end());

	return 1;
}

struct increase_community{
	uint32_t* snapVec, *comVec, size, *target_ii, *target_ij;

	__host__ __device__
		increase_community(uint32_t* snapVec, uint32_t* comVec, uint32_t size, uint32_t* target_ii, uint32_t* target_ij) :
		snapVec(snapVec), comVec(comVec), size(size), target_ii(target_ii), target_ij(target_ij) {}

	__device__
		void operator()(pair_t p){

		// task 1 find common communities
		bool found;
		uint32_t n_first_it = g_binary_search(snapVec, snapVec + size, p.first, found);
		while (n_first_it > 0 && snapVec[n_first_it] == p.first)--n_first_it;
		if (snapVec[n_first_it] != p.first)++n_first_it;
		
		uint32_t n_second_it = g_binary_search(snapVec, snapVec + size, p.second, found);
		while (n_second_it > 0 && snapVec[n_second_it] == p.second)--n_second_it;
		if (snapVec[n_second_it] != p.second)++n_second_it;

		uint32_t n_first_nd = n_first_it;
		while (n_first_nd < size){
			if (snapVec[n_first_nd] != p.first)
				break;
			atomicAdd((uint32_t*)(target_ij + comVec[n_first_nd]), (uint32_t)1);
			++n_first_nd;
		}

		uint32_t n_second_nd = n_second_it;
		while (n_second_nd < size){
			if (snapVec[n_second_nd] != p.second)
				break;
			atomicAdd((uint32_t*)(target_ij + comVec[n_second_nd]), (uint32_t)1);
			++n_second_nd;
		}

		// find common neighbour
		while (n_first_it < size && n_second_it < size){
			if (snapVec[n_first_it] != p.first || snapVec[n_second_it] != p.second)break;
			if (comVec[n_first_it] == comVec[n_second_it]){
				atomicAdd((uint32_t*)(target_ii + comVec[n_first_it]), (uint32_t)2);
				++n_first_it;
				++n_second_it;
			}
			else if (snapVec[n_first_it] < snapVec[n_second_it]){
				++n_first_it;
			}
			else{
				++n_second_it;
			}
		}

	}
};
/* Equation: 
 * ki * kj
 */
bool get_modularity(comevo::Source& sPairs, comevo::Source& sSnaps, uint32_t snapId, float &Q){

	vector<uint32_t> scom = sSnaps.get_scom(snapId);
	if (scom.empty())return 1;
	snapshot_t snap = sSnaps.get_snap(snapId);
	uint32_t com_max = *max_element(scom.begin(), scom.end());
	pairs_t pairs = sPairs.get_edges(snapId);
	T_DV<pair_t> d_pairs(pairs.begin(), pairs.end());
	int eTot = pairs.size();

	// go through all edges
	// store fraction that has both ends in one
	T_DV<uint32_t> eii(scom.size(), 0);
	T_DV<uint32_t> eij(scom.size(), 0);

	T_DV<uint32_t> d_snapVec;
	translate_snapshot_to_vector(snap, d_snapVec);
	T_DV<uint32_t> d_comVec;
	T_DV<uint32_t>d_comSizes(scom.begin(), scom.end());
	translate_scom_to_vector(d_comSizes, d_comVec);
	thrust::sort_by_key(d_snapVec.begin(), d_snapVec.end(), d_comVec.begin());

	thrust::for_each(d_pairs.begin(), d_pairs.end(), increase_community(RAWD(d_snapVec), RAWD(d_comVec), d_snapVec.size(), RAWD(eii), RAWD(eij)));

	Q = thrust::reduce(
		T_MTI(T_MZIMT(eii.begin(), T_MCONSI<uint32_t>(2*eTot)), zip_div<uint32_t>()),
		T_MTI(T_MZIMT(eii.end(), T_MCONSI<uint32_t>(2*eTot)), zip_div<uint32_t>()))
		- thrust::reduce(
		T_MTI(T_MTI(T_MZIMT(eij.begin(), T_MCONSI<uint32_t>(2 * eTot)), zip_div<uint32_t>()), set_square<float>()),
		T_MTI(T_MTI(T_MZIMT(eij.end(), T_MCONSI<uint32_t>(2 * eTot)), zip_div<uint32_t>()), set_square<float>()));
	return 1;
}
