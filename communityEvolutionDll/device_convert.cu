#include "stdafx.h"
#include "include/device_convert.h"
#include "include/display_elements.h"
#include "include/general_pair.h"
#include "include/device_pair.h"
#include "include/general_structs.h"

using namespace std;

bool translate_snapshot_to_vector(snapshot_t& snap, T_DV<uint32_t>& d_snapVec){
	d_snapVec.clear();
	for (uint32_t i = 0; i < snap.size(); ++i){
		d_snapVec.insert(d_snapVec.end(), snap[i].begin(), snap[i].end());
	}
	return 1;
}

bool translate_snapshot_to_matrix(T_DV<uint32_t>& d_snapVec, T_DV<uint32_t>& d_sizes, T_DV<uint32_t>& d_relevant, T_DV<bool>& d_matrix){
	uint32_t offset = 0;
	T_DV<bool> d_found(0);
	T_DV<uint32_t> d_indices(0);
	uint32_t n = d_relevant.size();
	d_found.resize(n);
	// target: snap in matrix
	d_matrix.assign(d_sizes.size() * n, 0);
	
	for (uint32_t i = 0; i < d_sizes.size(); ++i){
		thrust::binary_search(
			d_snapVec.begin() + offset,
			d_snapVec.begin() + offset + d_sizes[i],
			d_relevant.begin(),
			d_relevant.end(),
			d_found.begin());
		d_indices.resize(thrust::count(d_found.begin(), d_found.end(), 1));
		thrust::copy_if(T_MCI<uint32_t>(0), T_MCI<uint32_t>(n), d_found.begin(), d_indices.begin(), thrust::identity<bool>());

		thrust::transform(T_MPI(d_matrix.begin() + i*n, d_indices.begin()),
			T_MPI(d_matrix.begin() + i*n, d_indices.end()),
			T_MPI(d_matrix.begin() + i*n, d_indices.begin()),
			set_one<bool>());
		offset += d_sizes[i];

	}
	return 1;
}


struct fill_com_vec{
	uint32_t *comSizes, *comVec, *comFirsts;

	__host__ __device__
		fill_com_vec(uint32_t* comSizes, uint32_t* comVec, uint32_t* comFirsts) :
		comSizes(comSizes), comVec(comVec), comFirsts(comFirsts) {}

	__device__
		void operator()(uint32_t i){
		uint32_t size = comSizes[i];
		for (uint32_t j = 0; j < comSizes[i]; ++j){
			comVec[comFirsts[i]+j] = i;
		}
	}
};



bool translate_scom_to_vector(T_DV<uint32_t>& d_comSizes, T_DV<uint32_t>& d_comVec){
	d_comVec.assign(thrust::reduce(d_comSizes.begin(), d_comSizes.end()), 0);
	T_DV<uint32_t>d_comFirsts(d_comSizes.size());
	thrust::exclusive_scan(d_comSizes.begin(), d_comSizes.end(), d_comFirsts.begin());
	thrust::for_each(T_MCI<uint32_t>(0), T_MCI<uint32_t>(d_comSizes.size()),
		fill_com_vec(RAWD(d_comSizes), RAWD(d_comVec), RAWD(d_comFirsts)));
	return 1;
}