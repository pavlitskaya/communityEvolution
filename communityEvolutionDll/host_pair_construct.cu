#include "stdafx.h"
#include "include/host_pair_construct.h"
#include "include/host_analytic.h"
#include "include/general_arithmetic_structs.h"
#include "include/device_pair_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/display_elements.h"
#include "include/general_pair_structs.h"
#include "include/host_pair.h"

using namespace std;

namespace comevohost{

	// takes mirrored
	bool generate_pairs(std::vector<uint32_t>& keys, std::vector<uint32_t>& values, std::vector<pair_t>& pairs){
		T_HV<pair_t> h_target;
        T_HV<uint32_t> h_keys(keys.begin(), keys.end());
        T_HV<uint32_t> h_values(values.begin(), values.end());
        generate_pairs(h_keys, h_values, h_target);
		pairs.assign(h_target.begin(), h_target.end());
		return 1;
	}

	bool generate_unique_pairs(std::vector<uint32_t>& keys, std::vector<uint32_t>& values, std::vector<pair_t>& pairs){
		T_HV<pair_t> h_target;
        T_HV<uint32_t> h_keys(keys.begin(), keys.end());
        T_HV<uint32_t> h_values(values.begin(), values.end());
        generate_pairs(h_keys, h_values, h_target);
		thrust::sort(h_target.begin(), h_target.end());
		h_target.resize(thrust::unique(h_target.begin(), h_target.end()) - h_target.begin());
		pairs.assign(h_target.begin(), h_target.end());
		return 1;
	}

	bool generate_pairs_deep(std::vector<uint32_t>& keys, std::vector<uint32_t>& values, std::vector<pair_t>& pairs){
		T_HV<pair_t> h_target;
        T_HV<pair_t> h_pair(T_MTI(T_MZIMT(keys.begin(), values.begin()), pair_create()), T_MTI(T_MZIMT(keys.end(), values.end()), pair_create()));
        generate_pairs_deep(h_pair, h_target);
		pairs.assign(h_target.begin(), h_target.end());
		return 1;
	}


	/* generates degree deep
	* parameter: mirrored, sorted
	*/
	bool generate_pairs_deep(T_HV<pair_t>& h_keys_value, T_HV<pair_t>& h_target){
		T_HV<uint32_t> h_firsts, h_degree;
        T_HV<uint32_t> h_keys(T_MTI(h_keys_value.begin(), first_element()), T_MTI(h_keys_value.end(), first_element()));
        if (!get_degree_mirror(h_keys, h_degree))return 0;
		if (!get_firsts(h_degree, h_firsts))return 0;
		return generate_pairs_deep(h_keys_value.begin(), h_keys_value.end(), h_firsts, h_degree, h_target);
	}

	/* generates degree deep
	* parameter: mirrored, sorted
	*/
	bool generate_pairs_deep(T_HV<pair_t>::iterator h_keys_value_first, T_HV<pair_t>::iterator h_keys_value_last, T_HV<uint32_t> &h_firsts, T_HV<uint32_t> &h_degree, T_HV<pair_t>& h_target){
		// generate pairs init
		uint32_t max_size = get_max_combination(h_degree.begin(), h_degree.end());
		if (max_size < 1)return 1;

		// generate pairs main
		h_target.assign(max_size, pair_t(0, 0));

		uint32_t offset = 0;
		h_firsts.push_back(h_firsts.back() + h_degree.back()); // add last
		for (uint32_t i = 0; i < h_firsts.size() - 1; ++i){
			T_HV<uint32_t> values(
				T_MPI(T_MTI(h_keys_value_first, second_element()), T_MCI<uint32_t>(h_firsts[i])),
				T_MPI(T_MTI(h_keys_value_first, second_element()), T_MCI<uint32_t>(h_firsts[i + 1]))
				);
			uint32_t n_elements = h_degree[i] * (h_degree[i] - 1)*0.5;
			get_pairs(0, n_elements, h_degree[i], values, h_target, offset);
			offset += n_elements;

		}
		h_firsts.erase(h_firsts.end() - 1);

		return 1;
	}

	/* changes
	* parameter: mirrored, sorted
	*/
	bool generate_pairs(T_HV<pair_t>& h_keys_value, T_HV<pair_t>& h_pairs){
		return generate_pairs(h_keys_value.begin(), h_keys_value.end(), h_pairs);
	}

	/* changes
	* parameter: mirrored, sorted
	*/
	bool generate_pairs_limit(T_HV<pair_t>& h_keys_value, T_HV<pair_t>& h_pairs, uint32_t& offStart, uint32_t limit, bool& done){
		return generate_pairs_limit(h_keys_value.begin(), h_keys_value.end(), h_pairs, offStart, limit, done);
	}

	/* changes
	* parameter: mirrored, sorted, offset
	*/
	bool generate_pairs_limit(T_HV<pair_t>::iterator h_keys_value_first, T_HV<pair_t>::iterator h_keys_value_last, T_HV<pair_t>& h_pairs, uint32_t& offStart, uint32_t limit, bool& done){
		h_pairs.clear();
		uint32_t degree = h_keys_value_last - h_keys_value_first;
		uint32_t max_size = degree * (degree - 1) * 0.5;

		// generate pairs init
		if (max_size < 1){
			done = 1;
			return 1;
		}
		// size calculation
		T_HV<uint32_t>sizes(degree);
		thrust::sequence(sizes.begin(), sizes.end(), (int)degree, -1);
		thrust::exclusive_scan(sizes.begin(), sizes.end(), sizes.begin());
		// generate pairs main
		uint32_t off = offStart + 1;
		uint32_t intervals = degree;

		h_pairs.assign(limit, pair_t(0, 0));
		T_HV<pair_t>::iterator new_end = h_pairs.begin();
		//display_vector<uint32_t>(sizes, "sizes");
		while (off < intervals){
			if (sizes[off] - sizes[offStart] > limit){
				break;
			}
			new_end = thrust::copy_if(
				T_MTI(
				T_MZIMT(
				T_MTI(h_keys_value_first, second_element()),
				T_MTI(h_keys_value_first + off, second_element())),
				pair_create_sort()),
				T_MTI(
				T_MZIMT(
				T_MTI(h_keys_value_first + degree - off, second_element()),
				T_MTI(h_keys_value_first + degree, second_element())),
				pair_create_sort()),
				T_MCI<uint32_t>(0),
				new_end,
				equal_to_next(T_DEREF(h_keys_value_first, pair_t), degree, off));
			off += 1;
		}
		offStart = off - 1;
		if (off == intervals)
			done = true;
		h_pairs.erase(thrust::remove(h_pairs.begin(), h_pairs.end(), pair_t(0, 0)), h_pairs.end());
		h_pairs.erase(thrust::remove_if(h_pairs.begin(), h_pairs.end(), is_loop_pair()), h_pairs.end());
		return 1;
	}

	/* changes
	* parameter: mirrored, sorted
	*/
	bool generate_pairs(T_HV<pair_t>::iterator h_keys_value_first, T_HV<pair_t>::iterator h_keys_value_last, T_HV<pair_t>& h_pairs){
		h_pairs.clear();
		T_HV<uint32_t> h_degree;
        T_HV<uint32_t> h_keys(T_MTI(h_keys_value_first, first_element()), T_MTI(h_keys_value_last, first_element()));
        if (!get_degree_mirror(h_keys, h_degree)) return 0;

		// generate pairs init
		uint32_t max_size = get_max_combination(h_degree.begin(), h_degree.end());
		if (max_size < 1)return 1;
		// generate pairs main

		uint32_t size = h_keys_value_last - h_keys_value_first;
		uint32_t off = 1;
		uint32_t intervals = *thrust::max_element(h_degree.begin(), h_degree.end());
		TH_CLEAR(h_degree, uint32_t);
		h_pairs.assign(max_size, pair_t(0, 0));
		T_HV<pair_t>::iterator new_end = h_pairs.begin();
		while (off < intervals){
			new_end = thrust::copy_if(
				T_MTI(
				T_MZIMT(
				T_MTI(h_keys_value_first, second_element()),
				T_MTI(h_keys_value_first + off, second_element())),
				pair_create_sort()),
				T_MTI(
				T_MZIMT(
				T_MTI(h_keys_value_first + size - off, second_element()),
				T_MTI(h_keys_value_first + size, second_element())),
				pair_create_sort()),
				T_MCI<uint32_t>(0),
				new_end,
				equal_to_next(T_DEREF(h_keys_value_first, pair_t), size, off));
			off += 1;
		}

		h_pairs.erase(thrust::remove_if(h_pairs.begin(), h_pairs.end(), is_loop_pair()), h_pairs.end());

		return 1;
	}

	bool generate_pairs(T_HV<uint32_t>& h_keys, T_HV<uint32_t>& h_values, T_HV<pair_t>& h_pairs){
		return generate_pairs(h_keys.begin(), h_keys.end(), h_values.begin(), h_values.end(), h_pairs);
	}

	/* takes mirrored
	*/
	bool generate_pairs(T_HV<uint32_t>::iterator h_keys_first, T_HV<uint32_t>::iterator h_keys_last, T_HV<uint32_t>::iterator h_values_first, T_HV<uint32_t>::iterator h_values_last, T_HV<pair_t>& h_pairs){
        T_HV<pair_t> h_keys(T_MTI(T_MZIMT(h_keys_first, h_values_first), pair_create()), T_MTI(T_MZIMT(h_keys_last, h_values_last), pair_create()));

        return generate_pairs(h_keys, h_pairs);
	}

	bool pair_create_constant(uint32_t constant, uint32_t start, uint32_t end, T_HV<uint32_t>& values, T_HV<pair_t>::iterator target){
		thrust::transform(
			T_MPI(values.begin(), T_MCI<uint32_t>(start)),
			T_MPI(values.begin(), T_MCI<uint32_t>(end)),
			target,
			pair_create_const(values[constant])
			);
		return 1;
	}

	__host__ __device__ pair_t get_pair(const uint32_t& x, uint32_t& n){
		uint32_t count = 0;
		uint32_t k = n - 1;
		uint32_t old = 0;
		for (uint32_t j = n - 1; j > 0; j += --k){
			if (x < j){
				break;
			}
			++count;
			old = j;
		}
		return pair_t((uint32_t)count, (uint32_t)x - old + 1 + count);
	}

	/*
	Used to get pairs from from to to in tree n.
	n - tree-size
	values - relevant value
	target - contains relevant pairs afterwards
	offset - stores result in target beginning with offset

	example: (0, 3, 4,_, 1) stores [_, (0,1),(0,2),(0,3),(1,2)]

	*/
	bool get_pairs(const uint32_t& from, const uint32_t& to, uint32_t n, T_HV<uint32_t>& values, T_HV<pair_t>& target, uint32_t offset){

		pair_t f = get_pair(from, n);
		pair_t t = get_pair(to, n);
		pair_t tpre = get_pair(to - 1, n);

		// current part
		pair_create_constant(f.first, f.second, n, values, target.begin() + offset);
		uint32_t c = n - f.second;

		// till t.first
		for (uint32_t i = f.first + 1; i < tpre.first; ++i){
			pair_create_constant(i, i + 1, n, values, target.begin() + c + offset);
			c += n - (i + 1);
		}

		if (to > 1)
			pair_create_constant(tpre.first, tpre.first + 1, tpre.second + 1, values, target.begin() + c + offset);

		return 1;
	}
}
