#include "stdafx.h"
#include "include/display_elements.h"

using namespace std;

/*template <typename T> void display_direct(T* first, T* last, char* name){
	std::cout << name << ": " << endl;
	for (T* it = first; it != last; ++it){
		cout << *it << " ";
	}
	std::cout << "\n";
	std::cout << "\n";
}*/

template <typename T1, typename T2> void display_direct(pair_t* first, pair_t* last, char* name){
	std::cout << name << ": " << endl;
	for (pair_t* it = first; it != last; ++it){
		cout << (*it).first << ":" << (*it).second << " # ";
	}
	std::cout << "\n";
	std::cout << "\n";
}

template <typename T> void display_single(T const &single){ display_single<T>(single, ""); }
template <typename T> void display_single(T const &single, char* name){
	std::cout << name << ": " << endl;
	cout << single << endl;
	cout << "\n";
}

template <typename T> void display_vector(vector<T> const &vec){ display_vector<T>(vec, ""); }
template <typename T> void display_vector(vector<T> const &vec, char* name){ display_vector<T>(vec, vec.size(), name); }
template <typename T> void display_vector(vector<T> const &vec, uint32_t limit, char* name){ display_vector<T>(vec, limit, 0, name); }
template <typename T> void display_vector(vector<T> const &vec, uint32_t limit, uint16_t entriesBeforeLineBreak, char* name){
	std::cout << name << ": " << endl;
	if (!vec.empty()){
		uint32_t count = 0;
		for (typename vector<T>::const_iterator it = vec.begin(); it != vec.end(); ++it){
			std::cout << *it << " ";
			++count;
			if (limit > 0 && count == limit) break;
			if (entriesBeforeLineBreak > 0 && count % entriesBeforeLineBreak == 0) std::cout << "\n";
		}
	}
	std::cout << "\n\n";
}

template <typename T> void display_vector(T_DV<T> const &vec){ display_vector<T>(vec, ""); }
template <typename T> void display_vector(T_DV<T> const &vec, char* name){ display_vector<T>(vec, vec.size(), name); }
template <typename T> void display_vector(T_DV<T> const &vec, uint32_t limit, char* name){ display_vector<T>(vec, limit, 0, name); }
template <typename T> void display_vector(T_DV<T> const &vec, uint32_t limit, uint16_t entriesBeforeLineBreak, char* name){
	T_HV<T> h_vec(vec.begin(), vec.end());
	display_vector<T>(h_vec, limit, entriesBeforeLineBreak, name);
}

template <typename T> void display_vector(T_HV<T> const &vec){ display_vector<T>(vec, ""); }
template <typename T> void display_vector(T_HV<T> const &vec, char* name){ display_vector<T>(vec, vec.size(), name); }
template <typename T> void display_vector(T_HV<T> const &vec, uint32_t limit, char* name){ display_vector<T>(vec, limit, 0, name); }
template <typename T> void display_vector(T_HV<T> const &vec, uint32_t limit, uint16_t entriesBeforeLineBreak, char* name){
	std::cout << name << ": " << endl;
	if (!vec.empty()){
		uint32_t count = 0;
		for (typename T_HV<T>::const_iterator it = vec.begin(); it != vec.end(); ++it){
			std::cout << *it << " ";
			++count;
			if (limit > 0 && count == limit) break;
			if (entriesBeforeLineBreak > 0 && count % entriesBeforeLineBreak == 0) std::cout << "\n";
		}
	}
	std::cout << "\n\n";
}

// thrust::pairs
template <typename T1, typename T2> void display_vector(vector<thrust::pair<T1, T2> > const &vec){ display_vector<T1,T2>(vec, ""); }
template <typename T1, typename T2> void display_vector(vector<thrust::pair<T1, T2> > const &vec, char* name){ display_vector<T1, T2>(vec, vec.size(), name); }
template <typename T1, typename T2> void display_vector(vector<thrust::pair<T1, T2> > const &vec, uint32_t limit, char* name){ display_vector<T1, T2>(vec, limit, 0, name); }
template <typename T1, typename T2> void display_vector(vector<thrust::pair<T1,T2> > const &vec, uint32_t limit, uint16_t entriesBeforeLineBreak, char* name){
	std::cout << name << ": " << endl;
	if (!vec.empty()){
		uint32_t count = 0;
		for (typename vector<thrust::pair<T1, T2> >::const_iterator it = vec.begin(); it != vec.end(); ++it){
			thrust::pair<T1, T2> p = *it;
			cout << p.first << ":" << p.second << " # ";
			++count;
			if (limit > 0 && count == limit) break;
			if (entriesBeforeLineBreak > 0 && count % entriesBeforeLineBreak == 0) std::cout << "\n";
		}
	}
	std::cout << "\n\n";
}

template <typename T1, typename T2> void display_vector(T_DV<thrust::pair<T1, T2> > const &vec){ display_vector<T1, T2>(vec, ""); }
template <typename T1, typename T2> void display_vector(T_DV<thrust::pair<T1, T2> > const &vec, char* name){ display_vector<T1, T2>(vec, vec.size(), name); }
template <typename T1, typename T2> void display_vector(T_DV<thrust::pair<T1, T2> > const &vec, uint32_t limit, char* name){ display_vector<T1, T2>(vec, limit, 0, name); }
template <typename T1, typename T2> void display_vector(T_DV<thrust::pair<T1, T2> > const &vec, uint32_t limit, uint16_t entriesBeforeLineBreak, char* name){
	T_HV<thrust::pair<T1, T2 > > h_vec(vec.begin(), vec.end());
	display_vector<T1, T2>(h_vec, limit, entriesBeforeLineBreak, name);
}

template <typename T1, typename T2> void display_vector(T_HV<thrust::pair<T1, T2> > const &vec){ display_vector<T1, T2>(vec, ""); }
template <typename T1, typename T2> void display_vector(T_HV<thrust::pair<T1, T2> > const &vec, char* name){ display_vector<T1, T2>(vec, vec.size(), name); }
template <typename T1, typename T2> void display_vector(T_HV<thrust::pair<T1, T2> > const &vec, uint32_t limit, char* name){ display_vector<T1, T2>(vec, limit, 0, name); }

template <typename T1, typename T2> void display_vector(T_HV<thrust::pair<T1, T2> > const &vec, uint32_t limit, uint16_t entriesBeforeLineBreak, char* name){
	std::cout << name << ": " << endl;
	if (!vec.empty()){
		uint32_t count = 0;
		for (typename T_HV<thrust::pair<T1, T2> >::const_iterator it = vec.begin(); it != vec.end(); ++it){
			thrust::pair<T1, T2> p = *it;
			cout << p.first << ":" << p.second << " # ";
			++count;
			if (limit > 0 && count == limit) break;
			if (entriesBeforeLineBreak > 0 && count % entriesBeforeLineBreak == 0) std::cout << "\n";
		}
	}
	std::cout << "\n\n";
}
