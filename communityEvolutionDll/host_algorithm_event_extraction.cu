#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/host_algorithm_event_extraction.h"
#include "include/host_convert.h"
#include "include/general_arithmetic_structs.h"
#include "include/general_comparsion_structs.h"
#include "include/general_pair_structs.h"
#include "include/general_structs.h"

using namespace std;

namespace comevohost{
#define _OR 1
#define _AND 2

	void displayTripleVector(T_HV<tuple_triple>& vec, char* name, bool split){
		std::cout << name << ": " << endl;
		for (T_HV<tuple_triple>::iterator it = vec.begin(); it != vec.end(); ++it){
			tuple_triple trip = *it;
			if (split)
				cout << thrust::get<0>(trip) << " uint32_to " << thrust::get<1>(trip) << " + " << thrust::get<2>(trip) << " # ";
			else
				cout << thrust::get<0>(trip) << " + " << thrust::get<1>(trip) << " uint32_to " << thrust::get<2>(trip) << " # ";
		}
		std::cout << "\n";
		std::cout << "\n";
	}

	void displayPairVector(T_HV<pair_ti>& vec, char* name, char* str){
		std::cout << name << ": " << endl;
		for (T_HV<pair_ti>::iterator it = vec.begin(); it != vec.end(); ++it){
			pair_ti p = *it;
			std::cout << p.first << " " << str << " " << p.second << " # ";
		}
		std::cout << "\n";
		std::cout << "\n";
	}

	template <typename T>
	struct linear_index_to_row_index : public thrust::unary_function < T, T >
	{
		T C; // number of columns

		__host__ __device__
			linear_index_to_row_index(T C) : C(C) {}

		__host__ __device__
			T operator()(T i)
		{
			return i / C;
		}
	};
	struct _sum_row_b
	{
		bool* source;
		uint32_t n_cols, n_rows;
		uint32_t* target;

		__host__ __device__
			_sum_row_b(bool* source, uint32_t n_cols, uint32_t* target, uint32_t n_rows) :
			source(source), n_cols(n_cols), target(target), n_rows(n_rows) {}

		__host__ __device__
			void operator()(uint32_t i)
		{
			// mapped
			uint32_t x = (uint32_t)(i / n_rows);
			if (source[i] == 1)
				++target[x];
			//atomicAdd(target + x, 1);
		}
	};

	struct h_mergesplit{
		uint32_t *_or_ii_sum, *_anh_ij_sum, *_or_iij_sum, *_j_sum;
		float* _i_f_sum;
		float kappa;
		tuple_triple* target;
		uint32_t i_cols, j_cols;
		bool split;
		__host__ __device__
			h_mergesplit(uint32_t i_cols, uint32_t j_cols, float kappa, tuple_triple* target,
			uint32_t *_or_ii_sum, uint32_t *_anh_ij_sum, uint32_t *_or_iij_sum, float *_i_f_sum, uint32_t *_j_sum, bool split) :
			i_cols(i_cols), j_cols(j_cols), kappa(kappa), target(target),
			_or_ii_sum(_or_ii_sum), _anh_ij_sum(_anh_ij_sum), _or_iij_sum(_or_iij_sum), _i_f_sum(_i_f_sum), _j_sum(_j_sum), split(split) {}

		__host__ __device__
			void operator()(uint32_t i1_ind){
			// pouint32_t it:
			uint32_t tgt = i1_ind*i_cols;
			uint32_t tgt_last = i_cols*i_cols;
			uint32_t m = i_cols;
			uint32_t c2_m = j_cols;
			for (int i2_ind = i1_ind + 1; i2_ind < i_cols; ++i2_ind){
				for (int j = 0; j < j_cols; ++j){
					uint32_t tmp_ii = uint32_t((0.5*m*(m - 1) - 0.5*(m - i1_ind)*(m - i1_ind - 1) + (i2_ind - 1) - i1_ind));
					uint32_t tmp_ij = i1_ind*c2_m + j;
					uint32_t tmp_i2j = i2_ind*c2_m + j;
					if (_or_iij_sum[tmp_ii*c2_m + j] >= kappa * max((uint32_t)_or_ii_sum[tmp_ii], (uint32_t)_j_sum[j])){
						if (_anh_ij_sum[tmp_ij] >= _i_f_sum[i1_ind]){
							if (_anh_ij_sum[tmp_i2j] >= _i_f_sum[i2_ind]){
								while (thrust::get<0>(target[tgt]) >= 0 && tgt < tgt_last){
									++tgt;
								}
								if (tgt < tgt_last){
									if (split)
										target[tgt] = tuple_triple(j, (int)i1_ind, i2_ind);
									else
										target[tgt] = tuple_triple((int)i1_ind, i2_ind, j);
								}
							}
						}
					}
				}
			}
		}
	};

	struct h_continue{
		bool* _or_ij, *_anh_ij;
		uint32_t j_cols, n;
		pair_ti* target;
		__host__ __device__
			h_continue(uint32_t n, uint32_t j_cols, bool* _anh_ij, bool* _or_ij, pair_ti* target) : n(n), _or_ij(_or_ij), _anh_ij(_anh_ij), j_cols(j_cols), target(target) {}

		__host__ __device__
			void operator()(uint32_t i_ind){
			// pouint32_t it:
			uint32_t tgt = i_ind*j_cols;
			bool add;
			for (uint32_t j = 0; j < j_cols; ++j){
				add = true;
				for (uint32_t v = 0; v < n; ++v){
					if (_or_ij[(i_ind*j_cols + j)*n + v] != _anh_ij[(i_ind*j_cols + j)*n + v]){
						add = false;
					}
				}
				if (add){
					while (target[tgt].first >= 0){
						++tgt;
					}
					target[tgt] = pair_ti(i_ind, j);
				}
			}
		}
	};

	struct h_join{
		pair_ti * target;
		float* _i_f_sum;
		uint32_t* _anh_ij_sum;
		bool *h_preMatrix, *h_curMatrix;
		uint32_t nRelevant, i_cols, j_cols;
		__host__ __device__
			h_join(uint32_t nRelevant, uint32_t i_cols, uint32_t j_cols, uint32_t* _anh_ij_sum, bool* h_preMatrix, bool* h_curMatrix, float* _i_f_sum, pair_ti* target) :
			nRelevant(nRelevant), i_cols(i_cols), j_cols(j_cols), _anh_ij_sum(_anh_ij_sum), h_preMatrix(h_preMatrix), h_curMatrix(h_curMatrix), _i_f_sum(_i_f_sum), target(target) {}

		__host__ __device__
			void operator()(uint32_t j){
			uint32_t tgt = j;
			for (uint32_t v = 0; v < nRelevant; ++v){
				if (h_curMatrix[j*nRelevant + v] == 1){
					for (uint32_t i = 0; i < i_cols; ++i){
						if (_anh_ij_sum[i*j_cols + j] > _i_f_sum[i]){
							if (h_preMatrix[i*nRelevant + v] == 0){
								while (target[tgt].first >= 0){
									++tgt;
								}
								target[tgt] = (pair_ti(v, j));
							}
						}
					}
				}
			}
		}
	};

	struct h_leve{
		pair_ti * target;
		float* _i_f_sum;
		uint32_t* _anh_ij_sum;
		bool *h_preMatrix, *h_curMatrix;
		uint32_t nRelevant, i_cols, j_cols;
		__host__ __device__
			h_leve(uint32_t nRelevant, uint32_t i_cols, uint32_t j_cols, uint32_t* _anh_ij_sum, bool* h_preMatrix, bool* h_curMatrix, float* _i_f_sum, pair_ti* target) :
			nRelevant(nRelevant), i_cols(i_cols), j_cols(j_cols), _anh_ij_sum(_anh_ij_sum), h_preMatrix(h_preMatrix), h_curMatrix(h_curMatrix), _i_f_sum(_i_f_sum), target(target) {}

		__host__ __device__
			void operator()(uint32_t i){
			uint32_t tgt = i;
			for (uint32_t v = 0; v < nRelevant; ++v){
				if (h_preMatrix[i*nRelevant + v] == 1){
					for (uint32_t j = 0; j < j_cols; ++j){
						if (_anh_ij_sum[i*j_cols + j] > _i_f_sum[i]){
							if (h_curMatrix[j*nRelevant + v] == 0){
								while (target[tgt].first >= 0){
									++tgt;
								}
								target[tgt] = (pair_ti(v, i));
							}
						}
					}
				}
			}
		}
	};

	struct h_dissolve{
		uint32_t* source;
		bool* target;
		uint32_t s1_size, s2_size;
		__host__ __device__
			h_dissolve(uint32_t s1_size, uint32_t s2_size, uint32_t* source, bool* target) : source(source), s1_size(s1_size), target(target), s2_size(s2_size) {}

		__host__ __device__
			void operator()(uint32_t s1_ind){
			// pouint32_t it:

			target[s1_ind] = 0;
		}
	};

	struct h_transpose{
		bool* source, *target;
		uint32_t row_size, col_size;
		__host__ __device__
			h_transpose(uint32_t row_size, uint32_t col_size, bool* source, bool* target) : source(source), row_size(row_size), target(target), col_size(col_size) {}

		__host__ __device__
			void operator()(uint32_t val){
			// y, x
			pair_t res = pair_t((uint32_t)val / row_size, val % row_size);

			// y*m + x
			*(target + res.second*col_size + res.first) = *(source + val);
		}
	};


	T_HV<uint32_t> sum_row(T_HV<uint32_t>& source, uint32_t m, uint32_t n){
		T_HV<uint32_t> result(m);
		T_HV<uint32_t> row_indices(m);

		thrust::reduce_by_key
			(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(m)),
			thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(m)) + (n*m),
			source.begin(),
			row_indices.begin(),
			result.begin());
		return result;
	}

	bool transpose(uint32_t row_size, uint32_t col_size, vector<bool>& source, vector<bool>& target){
		bool source_val = source.front();
		bool target_val = target.front();
		thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(0) + row_size * col_size, h_transpose(
			row_size, col_size, &source_val, &target_val));

		return 1;
	}

	struct set_diag_max : public thrust::unary_function < uint32_t, uint32_t >
	{
		uint32_t C;
		uint32_t val;

		__host__ __device__
			set_diag_max(uint32_t C, uint32_t val) : C(C), val(val) {}

		__host__ __device__
			uint32_t operator()(uint32_t x) const
		{
			//printf("C: %d, val: %d, ind: %d, res: %d", C, val, x, x % (C + 1));
			return (x % (C + 1) == 0) ? val : 0;
		}
	};

	bool transpose(uint32_t row_size, uint32_t col_size, T_HV<bool>& source, T_HV<bool>& target){
		thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(0) + row_size * col_size, h_transpose(
			row_size, col_size, RAWD(source), RAWD(target)));

		return 1;
	}

	bool max_diagonal(T_HV<uint32_t>& source, uint32_t row_size){
		thrust::transform(T_MCI<uint32_t>(0), T_MCI<uint32_t>(row_size*row_size), source.begin(), set_diag_max(row_size, 1000));
		return 1;
	}

	bool sum_row(vector<bool>& source, uint32_t n, vector<uint32_t>& result){

		uint32_t m = source.size() / n;
		result.resize(m);
		vector<uint32_t> row_indices(m);
		try
		{
			thrust::reduce_by_key
				(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)),
				thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)) + (n*m),
				source.begin(),
				row_indices.begin(),
				result.begin());
		}
		catch (thrust::system_error &e)
		{
			// output an error message and exit
			printf("parameters are: n=%d", n);
			std::cerr << "Error accessing vector element: " << e.what() << std::endl;
			system("pause");
			exit(-1);
		}
		return 1;
	}

	/*
	bool sum_row(T_HV<bool>& source, uint32_t n, T_HV<uint32_t>& result){
	uint32_t m = source.size() / n;
	result.resize(m);
	T_HV<uint32_t> row_indices(m);
	uint32_t n_done = 0, n_todo = 0, n_rest = 0, cur_start = 0, cur_end = 0;
	set_start_parameter(n_done, n_todo, n_rest, cur_start, cur_end, n, 1);
	while (n_todo != 0){
	try
	{
	thrust::reduce_by_key
	(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(cur_start), linear_index_to_row_index<uint32_t>(cur_end)),
	thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(cur_start), linear_index_to_row_index<uint32_t>(cur_end)) + (n_todo*m),
	source.begin() + cur_start,
	row_indices.begin() + cur_start,
	result.begin() + cur_start);
	}
	catch (thrust::system_error &e)
	{
	// output an error message and exit
	printf("parameters are: n=%d \n", n);
	std::cerr << "Error accessing vector element: " << e.what() << std::endl;
	system("pause");
	exit(-1);
	}
	change_parameter(n_done, n_todo, n_rest, cur_start, cur_end, n, 1);
	}

	return 1;
	}
	*/

	bool sum_row(T_HV<bool>& source, uint32_t n, T_HV<uint32_t>& result){

		uint32_t m = source.size() / n;
		result.resize(m);
		T_HV<uint32_t> row_indices(m);

		thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(m*n), _sum_row_b(
			RAWD(source), m, RAWD(result), n));

		return 1;
	}

	/*
	bool sum_row(thrust::host_vector<bool>& source, uint32_t n, thrust::host_vector<uint32_t>& result){

		uint32_t m = source.size() / n;
		result.resize(m);
		thrust::host_vector<uint32_t> row_indices(m);

		thrust::reduce_by_key
			(thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)),
			thrust::make_transform_iterator(thrust::counting_iterator<uint32_t>(0), linear_index_to_row_index<uint32_t>(n)) + (n*m),
			source.begin(),
			row_indices.begin(),
			result.begin());
		return 1;
	}
	*/

	struct h_ii {
		uint32_t n, m, s1_ind, op;
		bool *source, *target;

		__host__ __device__
			h_ii(uint32_t n, bool *source, uint32_t m, uint32_t s1_ind, bool *target, uint32_t op) : n(n), source(source), m(m), s1_ind(s1_ind), target(target), op(op) {}

		__host__ __device__
			void operator()(uint32_t s2_ind){

			bool* s1_ptr = source + s1_ind*n;
			bool* s2_ptr = source + s2_ind*n;
			uint32_t tmp = uint32_t((0.5*m*(m - 1) - 0.5*(m - s1_ind)*(m - s1_ind - 1) + (s2_ind - 1) - s1_ind));
			bool* t_ptr = target + tmp * n;

			for (uint32_t i = 0; i < n; ++i){
				if (op == 1)*t_ptr = *s1_ptr || *s2_ptr;
				if (op == 2)*t_ptr = *s1_ptr && *s2_ptr;
				++t_ptr;
				++s1_ptr;
				++s2_ptr;
			}
		}
	};

	struct h_ij {
		uint32_t n, s1_m, s2_m, s1_ind, op;
		bool *s1_source, *s2_source, *target;

		__host__ __device__
			h_ij(uint32_t n, bool *s1_source, uint32_t s1_m, uint32_t s1_ind, bool *s2_source, uint32_t s2_m, bool *target, uint32_t op) :
			n(n), s1_source(s1_source), s1_m(s1_m), s1_ind(s1_ind), s2_source(s2_source), s2_m(s2_m), target(target), op(op) {}

		__host__ __device__
			void operator()(uint32_t s2_ind){

			bool* s1_ptr = s1_source + s1_ind*n;
			bool* s2_ptr = s2_source + s2_ind*n;
			uint32_t tmp = s1_ind*s2_m + s2_ind;
			bool* t_ptr = target + tmp * n;

			for (uint32_t i = 0; i < n; ++i){
				if (op == 1)*t_ptr = *s1_ptr || *s2_ptr;
				if (op == 2)*t_ptr = *s1_ptr && *s2_ptr;
				++t_ptr;
				++s1_ptr;
				++s2_ptr;
			}

		}
	};

	bool ii(uint32_t n, T_HV<bool>& h_preMatrix, uint32_t k_preCom, T_HV<bool>& _ii, uint32_t op){
		uint32_t _ii_size = 0.5*(k_preCom - 1)*k_preCom * n;
		_ii.resize(_ii_size, 0);

		// for each row
		for (uint32_t ind = 0; ind < k_preCom - 1; ++ind){

			thrust::for_each(thrust::host, T_MCI<uint32_t>(0) + ind + 1, T_MCI<uint32_t>(0) + k_preCom,
				h_ii(n, RAWD(h_preMatrix), k_preCom, ind, RAWD(_ii), op));
		}

		return 1;
	}

	bool ij(uint32_t n, T_HV<bool>& h_preMatrix, uint32_t k_preCom, T_HV<bool>& h_curMatrix, uint32_t k_curCom, T_HV<bool>& _ij, uint32_t op){
		uint32_t _ij_size = k_preCom*k_curCom * n;
		_ij.resize(_ij_size, 0);

		// for each row
		for (uint32_t ind = 0; ind < k_preCom; ++ind){

			thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(0) + k_curCom,
				h_ij(n, RAWD(h_preMatrix), k_preCom, ind,
				RAWD(h_curMatrix), k_curCom,
				RAWD(_ij), op));
		}

		return 1;
	}
	/*
	*/

	/* Asurs Event Extraction
	*/
	bool algorithm_event_extraction(comevo::Source &source, float k, bool display, bool create_file){
		// number of snaps and communities
		vector<community_t> communitySizes = source.get_scom();
        std::vector<snapshot_t> snaps = source.get_snaps();
        display_snapshots(snaps, "snaps");

		snapshot_t preSnap;
		snapshot_t curSnap;
		uint32_t preId = 0, curId, nRelevant;
		T_HV<uint32_t> preSizes, curSizes;
		T_HV<uint32_t> h_preSizes, h_curSizes;
		T_HV<uint32_t> h_preSnapVec, h_curSnapVec;
		T_HV<uint32_t> h_preSnapVecSort, h_curSnapVecSort;
		T_HV<uint32_t> h_preSnapVecUnique, h_curSnapVecUnique;
		T_HV<bool> h_preMatrix, h_curMatrix;
		uint32_t k_preCom, k_curCom;
		if (communitySizes.size() < 2)
			return 1;

		preSnap = source.get_snap(preId);
		preSizes = communitySizes[preId];
		h_preSizes.assign(preSizes.begin(), preSizes.end());
		comevohost::translate_snapshot_to_vector(preSnap, h_preSnapVec);
		h_preSnapVecUnique.assign(h_curSnapVec.begin(), h_curSnapVec.end());
		thrust::sort(h_preSnapVecUnique.begin(), h_preSnapVecUnique.end());
		h_preSnapVecUnique.erase(thrust::unique(h_preSnapVecUnique.begin(), h_preSnapVecUnique.end()), h_preSnapVecUnique.end());
		k_preCom = h_preSizes.size();

		// for each two Snapshots
		for (uint32_t snapId = 1; snapId < communitySizes.size(); ++snapId){
			cout << "current snaps: " << preId << " and " << snapId << endl;

			// extract snaps, prepare values
			preId = snapId - 1;
			curId = snapId;
			curSnap = source.get_snap(curId);
			/*display_snapshot(preSnap, "ps");
			display_snapshot(curSnap, "cs");*/
			curSizes = communitySizes[curId];
			h_curSizes.assign(curSizes.begin(), curSizes.end());

			k_curCom = h_curSizes.size();
			// translate Snapshot into Vector
			comevohost::translate_snapshot_to_vector(curSnap, h_curSnapVec);
			h_curSnapVecUnique.assign(h_curSnapVec.begin(), h_curSnapVec.end());
			thrust::sort(h_curSnapVecUnique.begin(), h_curSnapVecUnique.end());
			h_curSnapVecUnique.erase(thrust::unique(h_curSnapVecUnique.begin(), h_curSnapVecUnique.end()), h_curSnapVecUnique.end());

			// get relevant nodes
			T_HV<uint32_t> h_relevant(h_preSnapVecUnique.size() + h_curSnapVecUnique.size());
			h_relevant.erase(thrust::set_union(h_preSnapVecUnique.begin(), h_preSnapVecUnique.end(), h_curSnapVecUnique.begin(), h_curSnapVecUnique.end(), h_relevant.begin()), h_relevant.end());
			nRelevant = h_relevant.size();

			if (nRelevant > 0 && h_preSnapVec.size() > 0 && h_curSnapVec.size() > 0
				&& preSnap.size() > 0 && curSnap.size() > 0){

				// create Matrices of two Snapshots
				comevohost::translate_snapshot_to_matrix(h_preSnapVec, h_preSizes, h_relevant, h_preMatrix);
				comevohost::translate_snapshot_to_matrix(h_curSnapVec, h_curSizes, h_relevant, h_curMatrix);
				/*display_vector(h_preMatrix, "hpreM");
				display_vector(h_curMatrix, "hcurM");*/
				//display_vector<bool>(h_preMatrix, 0, nRelevant, "h_preMatrix");
				//display_vector<bool>(h_curMatrix, 0, nRelevant, "h_curMatrix");

				T_HV<uint32_t> _i_occurences;
				T_HV<uint32_t> _j_occurences;
				/*
				T_HV<bool> h_preMatrix_t(h_preMatrix.size(), 0);
				transpose(nRelevant, k_preCom, h_preMatrix, h_preMatrix_t);
				sum_row(h_preMatrix_t, k_preCom, _i_occurences);

				T_HV<bool> h_curMatrix_t(h_curMatrix.size(), 0);
				transpose(nRelevant, k_curCom, h_curMatrix, h_curMatrix_t);
				sum_row(h_curMatrix_t, k_curCom, _j_occurences);
				*/
				// start calculation: 
				// create iterator
				// A, A*: or_ii
				uint32_t _or_ii_n = 0.5*(k_preCom - 1)*k_preCom;
				T_HV<bool> _or_ii(_or_ii_n * nRelevant);
				ii(nRelevant, h_preMatrix, k_preCom, _or_ii, _OR);
				T_HV<uint32_t> _or_ii_sum;
				sum_row(_or_ii, nRelevant, _or_ii_sum);



				// or_jj
				uint32_t _or_jj_n = 0.5*(k_curCom - 1)*k_curCom;
				T_HV<bool> _or_jj(_or_jj_n * nRelevant);
				ii(nRelevant, h_curMatrix, k_curCom, _or_jj, _OR);
				T_HV<uint32_t> _or_jj_sum;
				sum_row(_or_jj, nRelevant, _or_jj_sum);

				// E
				T_HV<uint32_t> _i_sum;
				sum_row(h_preMatrix, nRelevant, _i_sum);
				T_HV<float> _i_f_sum(_i_sum.size());
				thrust::transform(_i_sum.begin(), _i_sum.end(), _i_f_sum.begin(), set_multiply<float>(0.5));

				// F
				T_HV<uint32_t> _j_sum;
				sum_row(h_curMatrix, nRelevant, _j_sum);
				T_HV<float> _j_f_sum(_j_sum.size());
				thrust::transform(_j_sum.begin(), _j_sum.end(), _j_f_sum.begin(), set_multiply<float>(0.5));

				// AF
				T_HV<bool> _anh_iij;
				ij(nRelevant, _or_ii, _or_ii_n, h_curMatrix, k_curCom, _anh_iij, _AND);
				T_HV<uint32_t> _or_iij_sum;
				sum_row(_anh_iij, nRelevant, _or_iij_sum);

				T_HV<bool> _anh_jji;
				ij(nRelevant, _or_jj, _or_jj_n, h_preMatrix, k_preCom, _anh_jji, _AND);
				T_HV<uint32_t> _or_jji_sum;
				sum_row(_anh_jji, nRelevant, _or_jji_sum);

				// B: or_ij
				T_HV<bool> _or_ij;
				ij(nRelevant, h_preMatrix, k_preCom, h_curMatrix, k_curCom, _or_ij, _OR);

				T_HV<bool> _or_ji;
				ij(nRelevant, h_curMatrix, k_curCom, h_preMatrix, k_preCom, _or_ji, _OR);

				// D, D*: anh_ij
				T_HV<bool> _anh_ij;
				ij(nRelevant, h_preMatrix, k_preCom, h_curMatrix, k_curCom, _anh_ij, _AND);
				T_HV<uint32_t> _anh_ij_sum;
				sum_row(_anh_ij, nRelevant, _anh_ij_sum);

				T_HV<bool> _anh_ji;
				ij(nRelevant, h_curMatrix, k_curCom, h_preMatrix, k_preCom, _anh_ji, _AND);
				T_HV<uint32_t> _anh_ji_sum;
				sum_row(_anh_ji, nRelevant, _anh_ji_sum);

				// G
				if (k_preCom > 0){
					T_HV<bool> h_preMatrix_t(h_preMatrix.size(), 0);
					transpose(nRelevant, k_preCom, h_preMatrix, h_preMatrix_t);
					//T_HV<uint32_t> _i_occurences;
					sum_row(h_preMatrix_t, k_preCom, _i_occurences);
				}

				// H
				if (k_curCom > 0){
					T_HV<bool> h_curMatrix_t(h_curMatrix.size(), 0);
					transpose(nRelevant, k_curCom, h_curMatrix, h_curMatrix_t);
					//T_HV<uint32_t> _j_occurences;
					sum_row(h_curMatrix_t, k_curCom, _j_occurences);
				}
				//*/

				// do the real work
				//displayVector(h_preMatrix, "h_preMatrix", nRelevant);
				//displayVector(h_curMatrix, "h_curMatrix", nRelevant);
				T_HV<bool> _dissolve;
				T_HV<bool> _form;
				T_HV<tuple_triple> _merge;
				T_HV<tuple_triple> _split;
				T_HV<pair_ti> _continue;
				T_HV<bool> _appear;
				T_HV<bool> _disappear;
				T_HV<pair_ti> _join;
				T_HV<pair_ti> _leve;
				// dissolve
				if (k_preCom > 0 && k_curCom > 0){
					_dissolve.assign(k_preCom, 0);

					for (uint32_t i = 0; i < k_preCom; ++i){
						T_HV<uint32_t>::iterator it = thrust::max_element(_anh_ij_sum.begin() + i * k_curCom, _anh_ij_sum.begin() + i * k_curCom + k_curCom);
						if (*it < 1)
							_dissolve[i] = 1;
					}

					if (display)display_vector<bool>(_dissolve, "dissolve");
				}

				// form
				if (k_preCom > 0 && k_curCom > 0){
					_form.assign(k_curCom, 0);

					for (uint32_t i = 0; i < k_curCom; ++i){
						T_HV<uint32_t>::iterator it = thrust::max_element(_anh_ji_sum.begin() + i * k_preCom, _anh_ji_sum.begin() + i * k_preCom + k_preCom);
						if (*it < 1)
							_form[i] = 1;
					}

					if (display)display_vector<bool>(_form, "_form");
				}

				// merge
				if (k_preCom > 1 && k_curCom > 0){
					float kappa = 0.5;
					_merge.assign(k_preCom*k_preCom, -1);
					thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_preCom - 1), h_mergesplit(
						k_preCom, k_curCom, kappa, RAWD(_merge), RAWD(_or_ii_sum), RAWD(_anh_ij_sum),
						RAWD(_or_iij_sum), RAWD(_i_f_sum), RAWD(_j_sum), false
						));

					T_HV<tuple_triple>::iterator tend = thrust::remove_if(_merge.begin(), _merge.end(), is_negative_triple());
					_merge.resize(tend - _merge.begin());
					//displayTripleVector(_merge, "_merge", false);
				}


				// split
				if (k_preCom > 0 && k_curCom > 1){
					float kappa = 0.5;
					_split.assign(k_curCom*k_curCom, -1);

					thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_curCom - 1), h_mergesplit(
						k_curCom, k_preCom, kappa, RAWD(_split), RAWD(_or_jj_sum), RAWD(_anh_ji_sum),
						RAWD(_or_jji_sum), RAWD(_j_f_sum), RAWD(_i_sum), true
						));

					T_HV<tuple_triple>::iterator tend = thrust::remove_if(_split.begin(), _split.end(), is_negative_triple());
					_split.resize(tend - _split.begin());
					if (display)displayTripleVector(_split, "_split", true);
				}

				// continue
				if (k_preCom > 0 && k_curCom > 0){
					_continue.assign(k_preCom*k_curCom, pair_ti(-1, -1));

					thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_preCom), h_continue(
						nRelevant, k_curCom, RAWD(_anh_ij), RAWD(_or_ij), RAWD(_continue)
						));

					T_HV<pair_ti>::iterator tend = thrust::remove_if(_continue.begin(), _continue.end(), is_negative_pair());
					_continue.resize(tend - _continue.begin());

					if (display)display_vector<int, int>(_continue, "_continue");
				}

				// appear
				if (k_preCom > 0){
					_appear.assign(nRelevant, 0);
					for (uint32_t i = 0; i < nRelevant; ++i){
						if (_i_occurences[i] == 0)
							if (_j_occurences[i] == 1)
								_appear[i] = 1;
					}
					if (display)display_vector<bool>(_appear, "_appear");
				}

				// disappear
				if (k_curCom > 0){
					_disappear.assign(nRelevant, 0);
					for (uint32_t i = 0; i < nRelevant; ++i){
						if (_i_occurences[i] == 1)
							if (_j_occurences[i] == 0)
								_disappear[i] = 1;
					}
					if (display)display_vector<bool>(_disappear, "_disappear");
				}

				display_single<uint32_t>(k_curCom, "k_curCom");
				display_single<uint32_t>(nRelevant, "nRelevant");
				display_single<uint32_t>(k_preCom, "k_preCom");
				cout << "_anh_ij_sum" << endl; 
				display_vector<uint32_t>(_anh_ij_sum);
				cout << "h_preMatrix" << endl;
				display_vector<bool>(h_preMatrix);
				cout << "h_curMatrix" << endl;
				display_vector<bool>(h_curMatrix);
				cout << "_i_f_sum" << endl;
				display_vector<float>(_i_f_sum);

				// join
				if (k_curCom > 0){
					_join.assign(k_curCom*nRelevant, pair_ti(-1, -1));

					thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_curCom), h_join(
						nRelevant, k_preCom, k_curCom, RAWD(_anh_ij_sum), RAWD(h_preMatrix),
						RAWD(h_curMatrix), RAWD(_i_f_sum), RAWD(_join)
						));

					T_HV<pair_ti>::iterator tend = thrust::remove_if(_join.begin(), _join.end(), is_negative_pair());
					_join.resize(tend - _join.begin());

					if (display)displayPairVector(_join, "_join", "to");
				}
				displayPairVector(_join, "_join", "to");

				T_HV<bool> h_joinVector(nRelevant, 0);
				cout << "h_joinVector" << endl;
				display_vector<bool>(h_joinVector);
				thrust::transform(T_MPI(h_joinVector.begin(), T_MTI(_join.begin(), first_element())),
					T_MPI(h_joinVector.begin(), T_MTI(_join.end(), first_element())),
					T_MPI(h_joinVector.begin(), T_MTI(_join.begin(), first_element())),
					set_one<bool>());
				cout << "h_joinVector" << endl;
				display_vector<bool>(h_joinVector);

				// leave
				if (k_preCom > 0){
					_leve.assign(k_preCom*nRelevant, pair_ti(-1, -1));

					thrust::for_each(thrust::host, T_MCI<uint32_t>(0), T_MCI<uint32_t>(k_preCom), h_leve(
						nRelevant, k_preCom, k_curCom, RAWD(_anh_ij_sum), RAWD(h_preMatrix),
						RAWD(h_curMatrix), RAWD(_i_f_sum), RAWD(_leve)
						));

					T_HV<pair_ti>::iterator tend = thrust::remove_if(_leve.begin(), _leve.end(), is_negative_pair());
					_leve.resize(tend - _leve.begin());

					if (display)displayPairVector(_leve, "_leve", "from");
				}

				if (create_file){
					time_t t = time(0);
					stringstream ss;
					ss << "events_";
					ss << t;
					string filename = ss.str();
                    std::vector<bool> dissolve(_dissolve.begin(), _dissolve.end());
                    std::vector<bool> form(_form.begin(), _form.end());
                    std::vector<tuple_triple> merge(_merge.begin(), _merge.end());
                    std::vector<tuple_triple> split(_split.begin(), _split.end());
                    std::vector<pair_ti> continue_(_continue.begin(), _continue.end());
                    std::vector<pair_ti> join(_join.begin(), _join.end());
                    std::vector<pair_ti> leve(_leve.begin(), _leve.end());
					comevo::save_to_file(filename,
                        dissolve,
                        form,
                        merge,
                        split,
                        continue_,
						thrust::count(_appear.begin(), _appear.end(), 1),
						thrust::count(_disappear.begin(), _disappear.end(), 1),
                        join,
                        leve);
				}

				printf("sizes: \n dissolve: %lu, form: %lu, merge: %lu, split: %lu, continue: %lu \n appear: %lu, disappear: %lu, join: %lu, leave: %lu \n\n",
					thrust::count(_dissolve.begin(), _dissolve.end(), 1),
					thrust::count(_form.begin(), _form.end(), 1),
					_merge.size(),
					_split.size(),
					_continue.size(),
					thrust::count(_appear.begin(), _appear.end(), 1),
					thrust::count(_disappear.begin(), _disappear.end(), 1),
					_join.size(),
					_leve.size()
					);


			}

			preSnap = curSnap;
			preSizes = curSizes;
			h_preSizes = h_curSizes;
			h_preSnapVec = h_curSnapVec;
			h_preSnapVecSort = h_curSnapVecSort;
			h_preSnapVecUnique = h_curSnapVecUnique;
			h_preMatrix = h_curMatrix;
			k_preCom = k_curCom;
			preId = curId;

		}


		return 1;
	}
}
//*/
