#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/matrix_calculation.h"
#include "include/device_analytic.h"

using namespace std;

bool matrices_create(uint32_t rowSize, uint32_t amount, uint32_t value, vector<T_HV<uint32_t> >& matrices){
	uint32_t matrixSize = rowSize*rowSize;
	for (uint32_t i = 0; i < amount; ++i){
		T_HV<uint32_t> matrix(matrixSize, value);
		matrices.push_back(matrix);
	}
	return 1;
}

bool create_adjacency(pairs_t pairs, T_DV<uint32_t>& d_matrix){
	T_DV<pair_t> d_pairs(pairs.begin(), pairs.end());
	//pairs_to
	//max node
	//to relevant nodes
	//for each pair 


	return 1;
}