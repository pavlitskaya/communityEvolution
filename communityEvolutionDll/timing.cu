#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "include/timing.h"

std::map<uint32_t, tuple_event> times;
std::map<uint32_t, clock_t> times_cpu;

void Timing::create_time(uint32_t id){
	
	if (times.end() == times.find(id)){
		hipEvent_t start, stop;
		times.insert(std::pair<uint32_t, tuple_event>(id, tuple_event(start, stop)));
	}
}

void Timing::start_time(uint32_t id){
	if (times.end() != times.find(id)){
		hipEventCreate(&times[id].first);
		hipEventCreate(&times[id].second);
		hipEventRecord(times[id].first, 0);
	}
}

float Timing::stop_time(uint32_t id){
	if (times.end() != times.find(id)){
		hipEventRecord(times[id].second, 0);
		hipEventSynchronize(times[id].second);
		float timeAlgorithm;
		hipEventElapsedTime(&timeAlgorithm, times[id].first, times[id].second);
		printf("\nTime for the kernel: %3.1f ms \n", timeAlgorithm);
		return timeAlgorithm;
	}
	return 0;
}

void Timing::start_time_cpu(uint32_t id){
	times_cpu.insert(std::pair<uint32_t, clock_t>(id, clock()));
}

float Timing::stop_time_cpu(uint32_t id){
	if (times_cpu.end() != times_cpu.find(id)){
		float timeAlgorithm = float(clock() - times_cpu[id]) / CLOCKS_PER_SEC;
		printf("\nTime for the CPU: %3.1f ms \n", timeAlgorithm);
		times_cpu.erase(id);
		return timeAlgorithm;
	}
	return 0;
}